#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void j3d125pt (double * __restrict__ t_in, double * __restrict__ t_out, int N) {
	//Determing the block's indices
	int i0 = (int)(blockIdx.x)*(int)(blockDim.x) + 2;
	int i = max(i0,2) + (int)(threadIdx.x);
	int j0 = 4*(int)(blockIdx.y)*(int)(blockDim.y) + 2;
	int j = max(j0,2) + 4*(int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*(int)(blockDim.z) + 2;
	int k = max(k0,2) + (int)(threadIdx.z);

	double (*in)[516][516] = (double (*)[516][516])t_in;
	double (*out)[516][516] = (double (*)[516][516])t_out;
	if (i>=2 && i<=N-3 && j>=2 && j<=N-3 && k>=2 && k<=N-3) {
double _t_3_ = in[k][j-2][i-2];
_t_3_ += in[k][j-2][i+2];
_t_3_ += in[k-1][j-1][i];
double _t_11_ = in[k-1][j-1][i];
_t_3_ += in[k-1][j][i-1];
double _t_7_ = in[k-1][j][i-1];
double _t_19_ = in[k-1][j][i-1];
_t_3_ += in[k-1][j][i+1];
_t_7_ += in[k-1][j][i+1];
_t_19_ += in[k-1][j][i+1];
_t_3_ += in[k-1][j+1][i];
double _t_12_ = in[k-1][j+1][i];
double _t_17_ = in[k-1][j+1][i];
double _t_25_ = in[k-1][j+1][i];
_t_3_ += in[k][j+2][i-2];
_t_11_ += in[k][j+2][i-2];
double _t_16_ = in[k][j+2][i-2];
_t_25_ += in[k][j+2][i-2];
_t_3_ += in[k][j+2][i+2];
_t_11_ += in[k][j+2][i+2];
_t_16_ += in[k][j+2][i+2];
_t_25_ += in[k][j+2][i+2];
_t_3_ += in[k+1][j-1][i];
_t_11_ += in[k+1][j-1][i];
_t_3_ += in[k+1][j][i-1];
_t_7_ += in[k+1][j][i-1];
_t_19_ += in[k+1][j][i-1];
_t_3_ += in[k+1][j][i+1];
_t_7_ += in[k+1][j][i+1];
_t_19_ += in[k+1][j][i+1];
_t_3_ += in[k+1][j+1][i];
_t_12_ += in[k+1][j+1][i];
_t_17_ += in[k+1][j+1][i];
_t_25_ += in[k+1][j+1][i];
_t_3_ += in[k-2][j][i-2];
double _t_9_ = in[k-2][j][i-2];
_t_9_ += in[k-2][j][i-2];
double _t_14_ = in[k-2][j][i-2];
double _t_15_ = in[k-2][j][i-2];
_t_3_ += in[k-2][j-2][i];
_t_3_ += in[k-2][j+2][i];
_t_11_ += in[k-2][j+2][i];
_t_11_ += in[k-2][j+2][i];
_t_16_ += in[k-2][j+2][i];
_t_16_ += in[k-2][j+2][i];
_t_25_ += in[k-2][j+2][i];
_t_25_ += in[k-2][j+2][i];
_t_3_ += in[k-2][j][i-2];
_t_3_ += in[k-2][j][i+2];
_t_9_ += in[k-2][j][i+2];
_t_9_ += in[k-2][j][i+2];
_t_14_ += in[k-2][j][i+2];
_t_15_ += in[k-2][j][i+2];
_t_3_ += in[k-2][j+2][i];
double outkc0jc0ic0 = 0.217 * _t_3_;
double _t_2_ = in[k-1][j-2][i-2];
_t_2_ += in[k-1][j-2][i+2];
_t_2_ += in[k][j-2][i];
_t_2_ += in[k+1][j-2][i-2];
_t_2_ += in[k+1][j-2][i+2];
_t_2_ += in[k-1][j+2][i-2];
_t_12_ += in[k-1][j+2][i-2];
double _t_18_ = in[k-1][j+2][i-2];
double _t_26_ = in[k-1][j+2][i-2];
_t_2_ += in[k-1][j+2][i+2];
_t_12_ += in[k-1][j+2][i+2];
_t_18_ += in[k-1][j+2][i+2];
_t_26_ += in[k-1][j+2][i+2];
_t_2_ += in[k][j][i-2];
_t_11_ += in[k][j][i-2];
_t_17_ += in[k][j][i-2];
_t_2_ += in[k][j][i+2];
_t_11_ += in[k][j][i+2];
_t_17_ += in[k][j][i+2];
_t_2_ += in[k][j+2][i];
_t_7_ += in[k][j+2][i];
_t_18_ += in[k][j+2][i];
double _t_21_ = in[k][j+2][i];
_t_2_ += in[k+1][j+2][i-2];
_t_12_ += in[k+1][j+2][i-2];
_t_18_ += in[k+1][j+2][i-2];
_t_26_ += in[k+1][j+2][i-2];
_t_2_ += in[k+1][j+2][i+2];
_t_12_ += in[k+1][j+2][i+2];
_t_18_ += in[k+1][j+2][i+2];
_t_26_ += in[k+1][j+2][i+2];
_t_2_ += in[k-2][j][i];
_t_11_ += in[k-2][j][i];
_t_11_ += in[k-2][j][i];
_t_17_ += in[k-2][j][i];
_t_17_ += in[k-2][j][i];
_t_2_ += in[k-2][j-1][i+2];
_t_7_ += in[k-2][j-1][i+2];
double _t_8_ = in[k-2][j-1][i+2];
_t_2_ += in[k-2][j+2][i+1];
_t_12_ += in[k-2][j+2][i+1];
double _t_13_ = in[k-2][j+2][i+1];
_t_18_ += in[k-2][j+2][i+1];
_t_18_ += in[k-2][j+2][i+1];
_t_26_ += in[k-2][j+2][i+1];
double _t_27_ = in[k-2][j+2][i+1];
_t_2_ += in[k-2][j+2][i-1];
_t_12_ += in[k-2][j+2][i-1];
_t_13_ += in[k-2][j+2][i-1];
_t_18_ += in[k-2][j+2][i-1];
_t_18_ += in[k-2][j+2][i-1];
_t_26_ += in[k-2][j+2][i-1];
_t_27_ += in[k-2][j+2][i-1];
_t_2_ += in[k-2][j-1][i+2];
_t_2_ += in[k-2][j+1][i+2];
double _t_10_ = in[k-2][j+1][i+2];
_t_10_ += in[k-2][j+1][i+2];
_t_16_ += in[k-2][j+1][i+2];
_t_16_ += in[k-2][j+1][i+2];
_t_21_ += in[k-2][j+1][i+2];
double _t_22_ = in[k-2][j+1][i+2];
_t_2_ += in[k-2][j+1][i+2];
_t_2_ += in[k-2][j][i];
_t_2_ += in[k-2][j+1][i-2];
_t_10_ += in[k-2][j+1][i-2];
_t_10_ += in[k-2][j+1][i-2];
_t_16_ += in[k-2][j+1][i-2];
_t_16_ += in[k-2][j+1][i-2];
_t_21_ += in[k-2][j+1][i-2];
_t_22_ += in[k-2][j+1][i-2];
_t_2_ += in[k-2][j+2][i-1];
_t_2_ += in[k-2][j-1][i-2];
_t_7_ += in[k-2][j-1][i-2];
_t_8_ += in[k-2][j-1][i-2];
_t_2_ += in[k-2][j+1][i-2];
_t_2_ += in[k-2][j-1][i-2];
outkc0jc0ic0 += 1.132 * _t_2_;
double _t_4_ = in[k-1][j-2][i];
_t_4_ += in[k][j-2][i-1];
_t_4_ += in[k][j-2][i+1];
_t_4_ += in[k+1][j-2][i];
_t_4_ += in[k-1][j][i-2];
_t_12_ += in[k-1][j][i-2];
_t_16_ += in[k-1][j][i-2];
_t_4_ += in[k-1][j][i+2];
_t_12_ += in[k-1][j][i+2];
_t_16_ += in[k-1][j][i+2];
_t_4_ += in[k-1][j+2][i];
_t_10_ += in[k-1][j+2][i];
_t_19_ += in[k-1][j+2][i];
double _t_24_ = in[k-1][j+2][i];
_t_4_ += in[k][j-1][i-2];
_t_10_ += in[k][j-1][i-2];
_t_4_ += in[k][j-1][i+2];
_t_10_ += in[k][j-1][i+2];
_t_4_ += in[k][j][i];
_t_7_ += in[k][j][i];
_t_16_ += in[k][j][i];
_t_4_ += in[k][j+1][i-2];
_t_9_ += in[k][j+1][i-2];
_t_18_ += in[k][j+1][i-2];
_t_24_ += in[k][j+1][i-2];
_t_4_ += in[k][j+1][i+2];
_t_9_ += in[k][j+1][i+2];
_t_18_ += in[k][j+1][i+2];
_t_24_ += in[k][j+1][i+2];
_t_4_ += in[k][j+2][i-1];
_t_12_ += in[k][j+2][i-1];
_t_14_ += in[k][j+2][i-1];
_t_26_ += in[k][j+2][i-1];
_t_4_ += in[k][j+2][i+1];
_t_12_ += in[k][j+2][i+1];
_t_14_ += in[k][j+2][i+1];
_t_26_ += in[k][j+2][i+1];
_t_4_ += in[k+1][j][i-2];
_t_12_ += in[k+1][j][i-2];
_t_16_ += in[k+1][j][i-2];
_t_4_ += in[k+1][j][i+2];
_t_12_ += in[k+1][j][i+2];
_t_16_ += in[k+1][j][i+2];
_t_4_ += in[k+1][j+2][i];
_t_10_ += in[k+1][j+2][i];
_t_19_ += in[k+1][j+2][i];
_t_24_ += in[k+1][j+2][i];
_t_4_ += in[k-2][j-1][i];
_t_10_ += in[k-2][j-1][i];
_t_10_ += in[k-2][j-1][i];
_t_4_ += in[k-2][j][i-1];
_t_12_ += in[k-2][j][i-1];
_t_13_ += in[k-2][j][i-1];
_t_16_ += in[k-2][j][i-1];
_t_16_ += in[k-2][j][i-1];
_t_4_ += in[k-2][j][i+1];
_t_12_ += in[k-2][j][i+1];
_t_13_ += in[k-2][j][i+1];
_t_16_ += in[k-2][j][i+1];
_t_16_ += in[k-2][j][i+1];
_t_4_ += in[k-2][j+1][i];
_t_9_ += in[k-2][j+1][i];
_t_9_ += in[k-2][j+1][i];
_t_18_ += in[k-2][j+1][i];
_t_18_ += in[k-2][j+1][i];
_t_24_ += in[k-2][j+1][i];
_t_24_ += in[k-2][j+1][i];
_t_4_ += in[k-2][j-1][i];
_t_4_ += in[k-2][j][i-1];
_t_4_ += in[k-2][j][i+1];
_t_4_ += in[k-2][j+1][i];
outkc0jc0ic0 += 2.13 * _t_4_;
double _t_0_ = in[k-2][j-2][i-2];
_t_0_ += in[k-2][j-2][i+2];
double _t_1_ = in[k-2][j-2][i-2];
_t_1_ += in[k-2][j-2][i+2];
_t_0_ += in[k-2][j+2][i-2];
_t_9_ += in[k-2][j+2][i-2];
_t_9_ += in[k-2][j+2][i-2];
_t_17_ += in[k-2][j+2][i-2];
_t_17_ += in[k-2][j+2][i-2];
double _t_23_ = in[k-2][j+2][i-2];
_t_23_ += in[k-2][j+2][i-2];
_t_0_ += in[k-2][j+2][i+2];
_t_9_ += in[k-2][j+2][i+2];
_t_9_ += in[k-2][j+2][i+2];
_t_17_ += in[k-2][j+2][i+2];
_t_17_ += in[k-2][j+2][i+2];
_t_23_ += in[k-2][j+2][i+2];
_t_23_ += in[k-2][j+2][i+2];
_t_0_ += in[k-1][j-1][i-1];
_t_12_ += in[k-1][j-1][i-1];
_t_0_ += in[k-1][j-1][i+1];
_t_12_ += in[k-1][j-1][i+1];
_t_0_ += in[k-1][j+1][i-1];
_t_10_ += in[k-1][j+1][i-1];
_t_14_ += in[k-1][j+1][i-1];
_t_26_ += in[k-1][j+1][i-1];
_t_0_ += in[k-1][j+1][i+1];
_t_10_ += in[k-1][j+1][i+1];
_t_14_ += in[k-1][j+1][i+1];
_t_26_ += in[k-1][j+1][i+1];
_t_0_ += in[k][j-1][i];
_t_9_ += in[k][j-1][i];
_t_0_ += in[k][j][i-1];
_t_12_ += in[k][j][i-1];
_t_18_ += in[k][j][i-1];
_t_0_ += in[k][j][i+1];
_t_12_ += in[k][j][i+1];
_t_18_ += in[k][j][i+1];
_t_0_ += in[k][j+1][i];
_t_11_ += in[k][j+1][i];
_t_14_ += in[k][j+1][i];
_t_23_ += in[k][j+1][i];
_t_0_ += in[k+1][j-1][i-1];
_t_12_ += in[k+1][j-1][i-1];
_t_0_ += in[k+1][j-1][i+1];
_t_12_ += in[k+1][j-1][i+1];
_t_0_ += in[k+1][j+1][i-1];
_t_10_ += in[k+1][j+1][i-1];
_t_14_ += in[k+1][j+1][i-1];
_t_26_ += in[k+1][j+1][i-1];
_t_0_ += in[k+1][j+1][i+1];
_t_10_ += in[k+1][j+1][i+1];
_t_14_ += in[k+1][j+1][i+1];
_t_26_ += in[k+1][j+1][i+1];
outkc0jc0ic0 += 0.75 * _t_0_;
_t_1_ += in[k-2][j+2][i-2];
_t_1_ += in[k-2][j+2][i+2];
outkc0jc0ic0 += 0.76 * _t_1_;
double _t_5_ = in[k-1][j-2][i-1];
_t_5_ += in[k-1][j-2][i+1];
_t_5_ += in[k+1][j-2][i-1];
_t_5_ += in[k+1][j-2][i+1];
_t_5_ += in[k-2][j-1][i-1];
_t_9_ += in[k-2][j-1][i-1];
_t_9_ += in[k-2][j-1][i-1];
_t_5_ += in[k-2][j-1][i+1];
_t_9_ += in[k-2][j-1][i+1];
_t_9_ += in[k-2][j-1][i+1];
_t_5_ += in[k-2][j+1][i-1];
_t_11_ += in[k-2][j+1][i-1];
_t_11_ += in[k-2][j+1][i-1];
_t_19_ += in[k-2][j+1][i-1];
double _t_20_ = in[k-2][j+1][i-1];
_t_23_ += in[k-2][j+1][i-1];
_t_23_ += in[k-2][j+1][i-1];
_t_5_ += in[k-2][j+1][i+1];
_t_11_ += in[k-2][j+1][i+1];
_t_11_ += in[k-2][j+1][i+1];
_t_19_ += in[k-2][j+1][i+1];
_t_20_ += in[k-2][j+1][i+1];
_t_23_ += in[k-2][j+1][i+1];
_t_23_ += in[k-2][j+1][i+1];
_t_5_ += in[k-1][j-1][i-2];
_t_9_ += in[k-1][j-1][i-2];
_t_5_ += in[k-1][j-1][i+2];
_t_9_ += in[k-1][j-1][i+2];
_t_5_ += in[k-1][j][i];
_t_10_ += in[k-1][j][i];
_t_18_ += in[k-1][j][i];
_t_5_ += in[k-1][j+1][i-2];
_t_11_ += in[k-1][j+1][i-2];
_t_19_ += in[k-1][j+1][i-2];
_t_23_ += in[k-1][j+1][i-2];
_t_5_ += in[k-1][j+1][i+2];
_t_11_ += in[k-1][j+1][i+2];
_t_19_ += in[k-1][j+1][i+2];
_t_23_ += in[k-1][j+1][i+2];
_t_5_ += in[k-1][j+2][i-1];
_t_7_ += in[k-1][j+2][i-1];
_t_17_ += in[k-1][j+2][i-1];
_t_21_ += in[k-1][j+2][i-1];
_t_5_ += in[k-1][j+2][i+1];
_t_7_ += in[k-1][j+2][i+1];
_t_17_ += in[k-1][j+2][i+1];
_t_21_ += in[k-1][j+2][i+1];
_t_5_ += in[k][j-1][i-1];
_t_11_ += in[k][j-1][i-1];
_t_5_ += in[k][j-1][i+1];
_t_11_ += in[k][j-1][i+1];
_t_5_ += in[k][j+1][i-1];
_t_7_ += in[k][j+1][i-1];
_t_19_ += in[k][j+1][i-1];
_t_25_ += in[k][j+1][i-1];
_t_5_ += in[k][j+1][i+1];
_t_7_ += in[k][j+1][i+1];
_t_19_ += in[k][j+1][i+1];
_t_25_ += in[k][j+1][i+1];
_t_5_ += in[k+1][j-1][i-2];
_t_9_ += in[k+1][j-1][i-2];
_t_5_ += in[k+1][j-1][i+2];
_t_9_ += in[k+1][j-1][i+2];
_t_5_ += in[k+1][j][i];
_t_10_ += in[k+1][j][i];
_t_18_ += in[k+1][j][i];
_t_5_ += in[k+1][j+1][i-2];
_t_11_ += in[k+1][j+1][i-2];
_t_19_ += in[k+1][j+1][i-2];
_t_23_ += in[k+1][j+1][i-2];
_t_5_ += in[k+1][j+1][i+2];
_t_11_ += in[k+1][j+1][i+2];
_t_19_ += in[k+1][j+1][i+2];
_t_23_ += in[k+1][j+1][i+2];
_t_5_ += in[k+1][j+2][i-1];
_t_7_ += in[k+1][j+2][i-1];
_t_17_ += in[k+1][j+2][i-1];
_t_21_ += in[k+1][j+2][i-1];
_t_5_ += in[k+1][j+2][i+1];
_t_7_ += in[k+1][j+2][i+1];
_t_17_ += in[k+1][j+2][i+1];
_t_21_ += in[k+1][j+2][i+1];
outkc0jc0ic0 += 0.331 * _t_5_;
double _t_6_ = in[k-2][j-1][i-1];
_t_6_ += in[k-2][j-1][i+1];
_t_6_ += in[k-2][j+1][i-1];
_t_6_ += in[k-2][j+1][i+1];
outkc0jc0ic0 += 0.332 * _t_6_;

_t_7_ += in[k-2][j+3][i-2];
_t_16_ += in[k-2][j+3][i-2];
_t_16_ += in[k-2][j+3][i-2];
_t_24_ += in[k-2][j+3][i-2];
_t_24_ += in[k-2][j+3][i-2];
_t_7_ += in[k-2][j+3][i+2];
_t_16_ += in[k-2][j+3][i+2];
_t_16_ += in[k-2][j+3][i+2];
_t_24_ += in[k-2][j+3][i+2];
_t_24_ += in[k-2][j+3][i+2];
double outkc0jp1ic0 = 0.75 * _t_7_;
_t_8_ += in[k-2][j+3][i-2];
_t_8_ += in[k-2][j+3][i+2];
outkc0jp1ic0 += 0.76 * _t_8_;
_t_9_ += in[k-2][j+3][i-1];
_t_19_ += in[k-2][j+3][i-1];
_t_25_ += in[k-2][j+3][i-1];
_t_25_ += in[k-2][j+3][i-1];
_t_9_ += in[k-2][j+3][i+1];
_t_19_ += in[k-2][j+3][i+1];
_t_25_ += in[k-2][j+3][i+1];
_t_25_ += in[k-2][j+3][i+1];
_t_9_ += in[k-2][j+3][i-1];
_t_9_ += in[k-2][j+3][i+1];
_t_9_ += in[k-1][j+3][i-2];
_t_19_ += in[k-1][j+3][i-2];
_t_25_ += in[k-1][j+3][i-2];
_t_9_ += in[k-1][j+3][i+2];
_t_19_ += in[k-1][j+3][i+2];
_t_25_ += in[k-1][j+3][i+2];
_t_9_ += in[k][j+3][i];
_t_14_ += in[k][j+3][i];
_t_25_ += in[k][j+3][i];
_t_9_ += in[k+1][j+3][i-2];
_t_19_ += in[k+1][j+3][i-2];
_t_25_ += in[k+1][j+3][i-2];
_t_9_ += in[k+1][j+3][i+2];
_t_19_ += in[k+1][j+3][i+2];
_t_25_ += in[k+1][j+3][i+2];
outkc0jp1ic0 += 1.132 * _t_9_;
_t_20_ += in[k-2][j+3][i-1];
_t_20_ += in[k-2][j+3][i+1];
double outkc0jp2ic0 = 0.332 * _t_20_;
_t_10_ += in[k-2][j+3][i];
_t_18_ += in[k-2][j+3][i];
_t_18_ += in[k-2][j+3][i];
_t_23_ += in[k-2][j+3][i];
_t_23_ += in[k-2][j+3][i];
_t_10_ += in[k-2][j+3][i];
_t_10_ += in[k][j+3][i-2];
_t_18_ += in[k][j+3][i-2];
_t_23_ += in[k][j+3][i-2];
_t_10_ += in[k][j+3][i+2];
_t_18_ += in[k][j+3][i+2];
_t_23_ += in[k][j+3][i+2];
outkc0jp1ic0 += 0.217 * _t_10_;
_t_11_ += in[k-1][j+3][i];
_t_17_ += in[k-1][j+3][i];
_t_26_ += in[k-1][j+3][i];
_t_11_ += in[k][j+3][i-1];
_t_19_ += in[k][j+3][i-1];
_t_21_ += in[k][j+3][i-1];
_t_11_ += in[k][j+3][i+1];
_t_19_ += in[k][j+3][i+1];
_t_21_ += in[k][j+3][i+1];
_t_11_ += in[k+1][j+3][i];
_t_17_ += in[k+1][j+3][i];
_t_26_ += in[k+1][j+3][i];
outkc0jp1ic0 += 2.13 * _t_11_;
_t_12_ += in[k-1][j+3][i-1];
_t_14_ += in[k-1][j+3][i-1];
_t_24_ += in[k-1][j+3][i-1];
_t_12_ += in[k-1][j+3][i+1];
_t_14_ += in[k-1][j+3][i+1];
_t_24_ += in[k-1][j+3][i+1];
_t_12_ += in[k+1][j+3][i-1];
_t_14_ += in[k+1][j+3][i-1];
_t_24_ += in[k+1][j+3][i-1];
_t_12_ += in[k+1][j+3][i+1];
_t_14_ += in[k+1][j+3][i+1];
_t_24_ += in[k+1][j+3][i+1];
outkc0jp1ic0 += 0.331 * _t_12_;
outkc0jp1ic0 += 0.332 * _t_13_;

_t_14_ += in[k-2][j+4][i-2];
_t_23_ += in[k-2][j+4][i-2];
_t_23_ += in[k-2][j+4][i-2];
_t_14_ += in[k-2][j+4][i+2];
_t_23_ += in[k-2][j+4][i+2];
_t_23_ += in[k-2][j+4][i+2];
outkc0jp2ic0 += 0.75 * _t_14_;
_t_15_ += in[k-2][j+4][i-2];
_t_15_ += in[k-2][j+4][i+2];
outkc0jp2ic0 += 0.76 * _t_15_;
_t_16_ += in[k-2][j+4][i-1];
_t_16_ += in[k-2][j+4][i+1];
_t_16_ += in[k-2][j+4][i-1];
_t_16_ += in[k-2][j+4][i+1];
_t_16_ += in[k-1][j+4][i-2];
_t_16_ += in[k-1][j+4][i+2];
_t_16_ += in[k][j+4][i];
_t_21_ += in[k][j+4][i];
_t_16_ += in[k+1][j+4][i-2];
_t_16_ += in[k+1][j+4][i+2];
outkc0jp2ic0 += 1.132 * _t_16_;
_t_27_ += in[k-2][j+4][i-1];
_t_27_ += in[k-2][j+4][i+1];
double outkc0jp3ic0 = 0.332 * _t_27_;
_t_17_ += in[k-2][j+4][i];
_t_17_ += in[k-2][j+4][i];
_t_17_ += in[k][j+4][i-2];
_t_17_ += in[k][j+4][i+2];
outkc0jp2ic0 += 0.217 * _t_17_;
_t_25_ += in[k-2][j+4][i];
_t_25_ += in[k-2][j+4][i];
_t_25_ += in[k][j+4][i-2];
_t_25_ += in[k][j+4][i+2];
_t_25_ += in[k-1][j+5][i];
_t_25_ += in[k][j+5][i-1];
_t_25_ += in[k][j+5][i+1];
_t_25_ += in[k+1][j+5][i];
outkc0jp3ic0 += 2.13 * _t_25_;
_t_18_ += in[k-1][j+4][i];
_t_24_ += in[k-1][j+4][i];
_t_18_ += in[k][j+4][i-1];
_t_18_ += in[k][j+4][i+1];
_t_18_ += in[k+1][j+4][i];
_t_24_ += in[k+1][j+4][i];
outkc0jp2ic0 += 2.13 * _t_18_;
_t_19_ += in[k-1][j+4][i-1];
_t_21_ += in[k-1][j+4][i-1];
_t_19_ += in[k-1][j+4][i+1];
_t_21_ += in[k-1][j+4][i+1];
_t_19_ += in[k+1][j+4][i-1];
_t_21_ += in[k+1][j+4][i-1];
_t_19_ += in[k+1][j+4][i+1];
_t_21_ += in[k+1][j+4][i+1];
outkc0jp2ic0 += 0.331 * _t_19_;
_t_26_ += in[k-1][j+4][i-2];
_t_26_ += in[k-1][j+4][i+2];
_t_26_ += in[k][j+4][i-1];
_t_26_ += in[k][j+4][i+1];
_t_26_ += in[k+1][j+4][i-2];
_t_26_ += in[k+1][j+4][i+2];
_t_26_ += in[k-1][j+5][i-1];
_t_26_ += in[k-1][j+5][i+1];
_t_26_ += in[k+1][j+5][i-1];
_t_26_ += in[k+1][j+5][i+1];
_t_26_ += in[k-2][j+4][i-1];
_t_26_ += in[k-2][j+4][i+1];
outkc0jp3ic0 += 0.331 * _t_26_;

_t_21_ += in[k-2][j+5][i-2];
_t_21_ += in[k-2][j+5][i+2];
_t_22_ += in[k-2][j+5][i-2];
_t_22_ += in[k-2][j+5][i+2];
outkc0jp3ic0 += 0.75 * _t_21_;
outkc0jp3ic0 += 0.76 * _t_22_;
_t_23_ += in[k-2][j+5][i+1];
outkc0jp3ic0 += 1.132 * _t_23_;
_t_24_ += in[k-2][j+5][i];
outkc0jp3ic0 += 0.217 * _t_24_;

out[k][j][i] = outkc0jc0ic0;
out[k][j+1][i] = outkc0jp1ic0;
out[k][j+2][i] = outkc0jp2ic0;
out[k][j+3][i] = outkc0jp3ic0;
	}
}

extern "C" void host_code (double *h_in, double *h_out, int N) {
	double *in;
	hipMalloc (&in, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for in\n");
	hipMemcpy (in, h_in, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *out;
	hipMalloc (&out, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for out\n");

	dim3 blockconfig (16, 4, 4);
	dim3 gridconfig (ceil(N-4, blockconfig.x), ceil(N-4, 4*blockconfig.y), ceil(N-4, blockconfig.z));

	j3d125pt<<<gridconfig, blockconfig>>> (in, out, N);

	hipMemcpy (h_out, out, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);

	hipFree (in); 
	hipFree (out);
}
