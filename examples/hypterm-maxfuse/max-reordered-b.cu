#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void hypterm (double * __restrict__ flux_in_0, double * __restrict__ flux_in_1, double * __restrict__ flux_in_2, double * __restrict__ flux_in_3, double * __restrict__ flux_in_4, double * __restrict__ cons_in_1, double * __restrict__ cons_in_2, double * __restrict__ cons_in_3, double * __restrict__ cons_in_4, double * __restrict__ q_in_1, double * __restrict__ q_in_2, double * __restrict__ q_in_3, double * __restrict__ q_in_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	int blockdim_k= (int)(blockDim.z);
	int k0 = (int)(blockIdx.z)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.z);

	double (*flux_0)[308][308] = (double (*)[308][308])flux_in_0;
	double (*flux_1)[308][308] = (double (*)[308][308])flux_in_1;
	double (*flux_2)[308][308] = (double (*)[308][308])flux_in_2;
	double (*flux_3)[308][308] = (double (*)[308][308])flux_in_3;
	double (*flux_4)[308][308] = (double (*)[308][308])flux_in_4;
	double (*q_1)[308][308] = (double (*)[308][308])q_in_1;
	double (*q_2)[308][308] = (double (*)[308][308])q_in_2;
	double (*q_3)[308][308] = (double (*)[308][308])q_in_3;
	double (*q_4)[308][308] = (double (*)[308][308])q_in_4;
	double (*cons_1)[308][308] = (double (*)[308][308])cons_in_1;
	double (*cons_2)[308][308] = (double (*)[308][308])cons_in_2;
	double (*cons_3)[308][308] = (double (*)[308][308])cons_in_3;
	double (*cons_4)[308][308] = (double (*)[308][308])cons_in_4;

	if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
double _t_1_ = cons_1[k][j][i+1];
_t_1_ -= cons_1[k][j][i-1];
double flux_0kc0jc0ic0 = dxinv0 * 0.8 * _t_1_;
double _t_2_ = cons_1[k][j][i+2];
_t_2_ -= cons_1[k][j][i-2];
flux_0kc0jc0ic0 -= dxinv0 * 0.2 * _t_2_;
double _t_3_ = cons_1[k][j][i+3];
_t_3_ -= cons_1[k][j][i-3];
flux_0kc0jc0ic0 += dxinv0 * 0.038 * _t_3_;
double _t_4_ = cons_1[k][j][i+4];
_t_4_ -= cons_1[k][j][i-4];
flux_0kc0jc0ic0 -= dxinv0 * 0.0035 * _t_4_;

double _t_6_ = cons_1[k][j][i+1] * q_1[k][j][i+1];
_t_6_ -= cons_1[k][j][i-1] * q_1[k][j][i-1];
_t_6_ += q_4[k][j][i+1];
_t_6_ -= q_4[k][j][i-1];
double flux_1kc0jc0ic0 = dxinv0 * 0.8 * _t_6_;
double _t_7_ = cons_1[k][j][i+2] * q_1[k][j][i+2];
_t_7_ -= cons_1[k][j][i-2] * q_1[k][j][i-2];
_t_7_ += q_4[k][j][i+2];
_t_7_ -= q_4[k][j][i-2];
flux_1kc0jc0ic0 -= dxinv0 * 0.2 * _t_7_;
double _t_8_ = cons_1[k][j][i+3] * q_1[k][j][i+3];
_t_8_ -= cons_1[k][j][i-3] * q_1[k][j][i-3];
_t_8_ += q_4[k][j][i+3];
_t_8_ -= q_4[k][j][i-3];
flux_1kc0jc0ic0 += dxinv0 * 0.038 * _t_8_;
double _t_9_ = cons_1[k][j][i+4] * q_1[k][j][i+4];
_t_9_ -= cons_1[k][j][i-4] * q_1[k][j][i-4];
_t_9_ += q_4[k][j][i+4];
_t_9_ -= q_4[k][j][i-4];
flux_1kc0jc0ic0 -= dxinv0 * 0.0035 * _t_9_;

double _t_11_ = cons_2[k][j][i+1] * q_1[k][j][i+1];
_t_11_ -= cons_2[k][j][i-1] * q_1[k][j][i-1];
double flux_2kc0jc0ic0 = dxinv0 * 0.8 * _t_11_;
double _t_12_ = cons_2[k][j][i+2] * q_1[k][j][i+2];
_t_12_ -= cons_2[k][j][i-2] * q_1[k][j][i-2];
flux_2kc0jc0ic0 -= dxinv0 * 0.2 * _t_12_;
double _t_13_ = cons_2[k][j][i+3] * q_1[k][j][i+3];
_t_13_ -= cons_2[k][j][i-3] * q_1[k][j][i-3];
flux_2kc0jc0ic0 += dxinv0 * 0.038 * _t_13_;
double _t_14_ = cons_2[k][j][i+4] * q_1[k][j][i+4];
_t_14_ -= cons_2[k][j][i-4] * q_1[k][j][i-4];
flux_2kc0jc0ic0 -= dxinv0 * 0.0035 * _t_14_;

double _t_16_ = cons_3[k][j][i+1] * q_1[k][j][i+1];
_t_16_ -= cons_3[k][j][i-1] * q_1[k][j][i-1];
double flux_3kc0jc0ic0 = dxinv0 * 0.8 * _t_16_;
double _t_17_ = cons_3[k][j][i+2] * q_1[k][j][i+2];
_t_17_ -= cons_3[k][j][i-2] * q_1[k][j][i-2];
flux_3kc0jc0ic0 -= dxinv0 * 0.2 * _t_17_;
double _t_18_ = cons_3[k][j][i+3] * q_1[k][j][i+3];
_t_18_ -= cons_3[k][j][i-3] * q_1[k][j][i-3];
flux_3kc0jc0ic0 += dxinv0 * 0.038 * _t_18_;
double _t_19_ = cons_3[k][j][i+4] * q_1[k][j][i+4];
_t_19_ -= cons_3[k][j][i-4] * q_1[k][j][i-4];
flux_3kc0jc0ic0 -= dxinv0 * 0.0035 * _t_19_;

double _t_21_ = q_4[k][j][i+1] * q_1[k][j][i+1];
double _v_24_ = cons_4[k][j][i+1] * q_1[k][j][i+1];
_t_21_ += _v_24_;
_t_21_ -= cons_4[k][j][i-1] * q_1[k][j][i-1];
double _v_27_ = q_4[k][j][i-1] * q_1[k][j][i-1];
_t_21_ -= _v_27_;
double flux_4kc0jc0ic0 = dxinv0 * 0.8 * _t_21_;
double _t_22_ = q_4[k][j][i+2] * q_1[k][j][i+2];
double _v_28_ = cons_4[k][j][i+2] * q_1[k][j][i+2];
_t_22_ += _v_28_;
_t_22_ -= cons_4[k][j][i-2] * q_1[k][j][i-2];
double _v_31_ = q_4[k][j][i-2] * q_1[k][j][i-2];
_t_22_ -= _v_31_;
flux_4kc0jc0ic0 -= dxinv0 * 0.2 * _t_22_;
double _t_23_ = q_4[k][j][i+3] * q_1[k][j][i+3];
double _v_32_ = cons_4[k][j][i+3] * q_1[k][j][i+3];
_t_23_ += _v_32_;
_t_23_ -= cons_4[k][j][i-3] * q_1[k][j][i-3];
double _v_35_ = q_4[k][j][i-3] * q_1[k][j][i-3];
_t_23_ -= _v_35_;
flux_4kc0jc0ic0 += dxinv0 * 0.038 * _t_23_;
double _t_24_ = q_4[k][j][i+4] * q_1[k][j][i+4];
double _v_36_ = cons_4[k][j][i+4] * q_1[k][j][i+4];
_t_24_ += _v_36_;
_t_24_ -= cons_4[k][j][i-4] * q_1[k][j][i-4];
double _v_39_ = q_4[k][j][i-4] * q_1[k][j][i-4];
_t_24_ -= _v_39_;
flux_4kc0jc0ic0 -= dxinv0 * 0.0035 * _t_24_;

double _t_27_ = cons_2[k][j+1][i];
_t_27_ -= cons_2[k][j-1][i];
double _t_25_ = dxinv1 * 0.8 * _t_27_;
double _t_28_ = cons_2[k][j+2][i];
_t_28_ -= cons_2[k][j-2][i];
_t_25_ -= dxinv1 * 0.2 * _t_28_;
double _t_29_ = cons_2[k][j+3][i];
_t_29_ -= cons_2[k][j-3][i];
_t_25_ += dxinv1 * 0.038 * _t_29_;
double _t_30_ = cons_2[k][j+4][i];
_t_30_ -= cons_2[k][j-4][i];
_t_25_ -= dxinv1 * 0.0035 * _t_30_;
flux_0kc0jc0ic0 -= _t_25_;

double _t_33_ = cons_1[k][j+1][i] * q_2[k][j+1][i];
_t_33_ -= cons_1[k][j-1][i] * q_2[k][j-1][i];
double _t_31_ = dxinv1 * 0.8 * _t_33_;
double _t_34_ = cons_1[k][j+2][i] * q_2[k][j+2][i];
_t_34_ -= cons_1[k][j-2][i] * q_2[k][j-2][i];
_t_31_ -= dxinv1 * 0.2 * _t_34_;
double _t_35_ = cons_1[k][j+3][i] * q_2[k][j+3][i];
_t_35_ -= cons_1[k][j-3][i] * q_2[k][j-3][i];
_t_31_ += dxinv1 * 0.038 * _t_35_;
double _t_36_ = cons_1[k][j+4][i] * q_2[k][j+4][i];
_t_36_ -= cons_1[k][j-4][i] * q_2[k][j-4][i];
_t_31_ -= dxinv1 * 0.0035 * _t_36_;
flux_1kc0jc0ic0 -= _t_31_;

double _t_39_ = cons_2[k][j+1][i] * q_2[k][j+1][i];
_t_39_ -= cons_2[k][j-1][i] * q_2[k][j-1][i];
_t_39_ += q_4[k][j+1][i];
_t_39_ -= q_4[k][j-1][i];
double _t_37_ = dxinv1 * 0.8 * _t_39_;
double _t_40_ = cons_2[k][j+2][i] * q_2[k][j+2][i];
_t_40_ -= cons_2[k][j-2][i] * q_2[k][j-2][i];
_t_40_ += q_4[k][j+2][i];
_t_40_ -= q_4[k][j-2][i];
_t_37_ -= dxinv1 * 0.2 * _t_40_;
double _t_41_ = cons_2[k][j+3][i] * q_2[k][j+3][i];
_t_41_ -= cons_2[k][j-3][i] * q_2[k][j-3][i];
_t_41_ += q_4[k][j+3][i];
_t_41_ -= q_4[k][j-3][i];
_t_37_ += dxinv1 * 0.038 * _t_41_;
double _t_42_ = cons_2[k][j+4][i] * q_2[k][j+4][i];
_t_42_ -= cons_2[k][j-4][i] * q_2[k][j-4][i];
_t_42_ += q_4[k][j+4][i];
_t_42_ -= q_4[k][j-4][i];
_t_37_ -= dxinv1 * 0.0035 * _t_42_;
flux_2kc0jc0ic0 -= _t_37_;

double _t_45_ = cons_3[k][j+1][i] * q_2[k][j+1][i];
_t_45_ -= cons_3[k][j-1][i] * q_2[k][j-1][i];
double _t_43_ = dxinv1 * 0.8 * _t_45_;
double _t_46_ = cons_3[k][j+2][i] * q_2[k][j+2][i];
_t_46_ -= cons_3[k][j-2][i] * q_2[k][j-2][i];
_t_43_ -= dxinv1 * 0.2 * _t_46_;
double _t_47_ = cons_3[k][j+3][i] * q_2[k][j+3][i];
_t_47_ -= cons_3[k][j-3][i] * q_2[k][j-3][i];
_t_43_ += dxinv1 * 0.038 * _t_47_;
double _t_48_ = cons_3[k][j+4][i] * q_2[k][j+4][i];
_t_48_ -= cons_3[k][j-4][i] * q_2[k][j-4][i];
_t_43_ -= dxinv1 * 0.0035 * _t_48_;
flux_3kc0jc0ic0 -= _t_43_;

double _t_51_ = q_4[k][j+1][i] * q_2[k][j+1][i];
double _v_64_ = cons_4[k][j+1][i] * q_2[k][j+1][i];
_t_51_ += _v_64_;
_t_51_ -= cons_4[k][j-1][i] * q_2[k][j-1][i];
double _v_67_ = q_4[k][j-1][i] * q_2[k][j-1][i];
_t_51_ -= _v_67_;
double _t_49_ = dxinv1 * 0.8 * _t_51_;
double _t_52_ = q_4[k][j+2][i] * q_2[k][j+2][i];
double _v_68_ = cons_4[k][j+2][i] * q_2[k][j+2][i];
_t_52_ += _v_68_;
_t_52_ -= cons_4[k][j-2][i] * q_2[k][j-2][i];
double _v_71_ = q_4[k][j-2][i] * q_2[k][j-2][i];
_t_52_ -= _v_71_;
_t_49_ -= dxinv1 * 0.2 * _t_52_;
double _t_53_ = q_4[k][j+3][i] * q_2[k][j+3][i];
double _v_72_ = cons_4[k][j+3][i] * q_2[k][j+3][i];
_t_53_ += _v_72_;
_t_53_ -= cons_4[k][j-3][i] * q_2[k][j-3][i];
double _v_75_ = q_4[k][j-3][i] * q_2[k][j-3][i];
_t_53_ -= _v_75_;
_t_49_ += dxinv1 * 0.038 * _t_53_;
double _t_54_ = q_4[k][j+4][i] * q_2[k][j+4][i];
double _v_76_ = cons_4[k][j+4][i] * q_2[k][j+4][i];
_t_54_ += _v_76_;
_t_54_ -= cons_4[k][j-4][i] * q_2[k][j-4][i];
double _v_79_ = q_4[k][j-4][i] * q_2[k][j-4][i];
_t_54_ -= _v_79_;
_t_49_ -= dxinv1 * 0.0035 * _t_54_;
flux_4kc0jc0ic0 -= _t_49_;

double _t_57_ = cons_3[k+1][j][i];
_t_57_ -= cons_3[k-1][j][i];
double _t_55_ = dxinv2 * 0.8 * _t_57_;
double _t_58_ = cons_3[k+2][j][i];
_t_58_ -= cons_3[k-2][j][i];
_t_55_ -= dxinv2 * 0.2 * _t_58_;
double _t_59_ = cons_3[k+3][j][i];
_t_59_ -= cons_3[k-3][j][i];
_t_55_ += dxinv2 * 0.038 * _t_59_;
double _t_60_ = cons_3[k+4][j][i];
_t_60_ -= cons_3[k-4][j][i];
_t_55_ -= dxinv2 * 0.0035 * _t_60_;
flux_0kc0jc0ic0 -= _t_55_;

double _t_63_ = cons_1[k+1][j][i] * q_3[k+1][j][i];
_t_63_ -= cons_1[k-1][j][i] * q_3[k-1][j][i];
double _t_61_ = dxinv2 * 0.8 * _t_63_;
double _t_64_ = cons_1[k+2][j][i] * q_3[k+2][j][i];
_t_64_ -= cons_1[k-2][j][i] * q_3[k-2][j][i];
_t_61_ -= dxinv2 * 0.2 * _t_64_;
double _t_65_ = cons_1[k+3][j][i] * q_3[k+3][j][i];
_t_65_ -= cons_1[k-3][j][i] * q_3[k-3][j][i];
_t_61_ += dxinv2 * 0.038 * _t_65_;
double _t_66_ = cons_1[k+4][j][i] * q_3[k+4][j][i];
_t_66_ -= cons_1[k-4][j][i] * q_3[k-4][j][i];
_t_61_ -= dxinv2 * 0.0035 * _t_66_;
flux_1kc0jc0ic0 -= _t_61_;

double _t_69_ = cons_2[k+1][j][i] * q_3[k+1][j][i];
_t_69_ -= cons_2[k-1][j][i] * q_3[k-1][j][i];
double _t_67_ = dxinv2 * 0.8 * _t_69_;
double _t_70_ = cons_2[k+2][j][i] * q_3[k+2][j][i];
_t_70_ -= cons_2[k-2][j][i] * q_3[k-2][j][i];
_t_67_ -= dxinv2 * 0.2 * _t_70_;
double _t_71_ = cons_2[k+3][j][i] * q_3[k+3][j][i];
_t_71_ -= cons_2[k-3][j][i] * q_3[k-3][j][i];
_t_67_ += dxinv2 * 0.038 * _t_71_;
double _t_72_ = cons_2[k+4][j][i] * q_3[k+4][j][i];
_t_72_ -= cons_2[k-4][j][i] * q_3[k-4][j][i];
_t_67_ -= dxinv2 * 0.0035 * _t_72_;
flux_2kc0jc0ic0 -= _t_67_;

double _t_75_ = cons_3[k+1][j][i] * q_3[k+1][j][i];
_t_75_ -= cons_3[k-1][j][i] * q_3[k-1][j][i];
_t_75_ += q_4[k+1][j][i];
_t_75_ -= q_4[k-1][j][i];
double _t_73_ = dxinv2 * 0.8 * _t_75_;
double _t_76_ = cons_3[k+2][j][i] * q_3[k+2][j][i];
_t_76_ -= cons_3[k-2][j][i] * q_3[k-2][j][i];
_t_76_ += q_4[k+2][j][i];
_t_76_ -= q_4[k-2][j][i];
_t_73_ -= dxinv2 * 0.2 * _t_76_;
double _t_77_ = cons_3[k+3][j][i] * q_3[k+3][j][i];
_t_77_ -= cons_3[k-3][j][i] * q_3[k-3][j][i];
_t_77_ += q_4[k+3][j][i];
_t_77_ -= q_4[k-3][j][i];
_t_73_ += dxinv2 * 0.038 * _t_77_;
double _t_78_ = cons_3[k+4][j][i] * q_3[k+4][j][i];
_t_78_ -= cons_3[k-4][j][i] * q_3[k-4][j][i];
_t_78_ += q_4[k+4][j][i];
_t_78_ -= q_4[k-4][j][i];
_t_73_ -= dxinv2 * 0.0035 * _t_78_;
flux_3kc0jc0ic0 -= _t_73_;

double _t_81_ = q_4[k+1][j][i] * q_3[k+1][j][i];
double _v_104_ = cons_4[k+1][j][i] * q_3[k+1][j][i];
_t_81_ += _v_104_;
_t_81_ -= cons_4[k-1][j][i] * q_3[k-1][j][i];
double _v_107_ = q_4[k-1][j][i] * q_3[k-1][j][i];
_t_81_ -= _v_107_;
double _t_79_ = dxinv2 * 0.8 * _t_81_;
double _t_82_ = q_4[k+2][j][i] * q_3[k+2][j][i];
double _v_108_ = cons_4[k+2][j][i] * q_3[k+2][j][i];
_t_82_ += _v_108_;
_t_82_ -= cons_4[k-2][j][i] * q_3[k-2][j][i];
double _v_111_ = q_4[k-2][j][i] * q_3[k-2][j][i];
_t_82_ -= _v_111_;
_t_79_ -= dxinv2 * 0.2 * _t_82_;
double _t_83_ = q_4[k+3][j][i] * q_3[k+3][j][i];
double _v_112_ = cons_4[k+3][j][i] * q_3[k+3][j][i];
_t_83_ += _v_112_;
_t_83_ -= cons_4[k-3][j][i] * q_3[k-3][j][i];
double _v_115_ = q_4[k-3][j][i] * q_3[k-3][j][i];
_t_83_ -= _v_115_;
_t_79_ += dxinv2 * 0.038 * _t_83_;
double _t_84_ = q_4[k+4][j][i] * q_3[k+4][j][i];
double _v_116_ = cons_4[k+4][j][i] * q_3[k+4][j][i];
_t_84_ += _v_116_;
_t_84_ -= cons_4[k-4][j][i] * q_3[k-4][j][i];
double _v_119_ = q_4[k-4][j][i] * q_3[k-4][j][i];
_t_84_ -= _v_119_;
_t_79_ -= dxinv2 * 0.0035 * _t_84_;
flux_4kc0jc0ic0 -= _t_79_;

flux_0[k][j][i] = flux_0kc0jc0ic0;
flux_1[k][j][i] = flux_1kc0jc0ic0;
flux_2[k][j][i] = flux_2kc0jc0ic0;
flux_3[k][j][i] = flux_3kc0jc0ic0;
flux_4[k][j][i] = flux_4kc0jc0ic0;
	} 
}

extern "C" void host_code (double *h_flux_0, double *h_flux_1, double *h_flux_2, double *h_flux_3, double *h_flux_4, double *h_cons_1, double *h_cons_2, double *h_cons_3, double *h_cons_4, double *h_q_1, double *h_q_2, double *h_q_3, double *h_q_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	double *flux_0;
	hipMalloc (&flux_0, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_0\n");
	hipMemcpy (flux_0, h_flux_0, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_1;
	hipMalloc (&flux_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_1\n");
	hipMemcpy (flux_1, h_flux_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_2;
	hipMalloc (&flux_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_2\n");
	hipMemcpy (flux_2, h_flux_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_3;
	hipMalloc (&flux_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_3\n");
	hipMemcpy (flux_3, h_flux_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_4;
	hipMalloc (&flux_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_4\n");
	hipMemcpy (flux_4, h_flux_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_1;
	hipMalloc (&cons_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_1\n");
	hipMemcpy (cons_1, h_cons_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_2;
	hipMalloc (&cons_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_2\n");
	hipMemcpy (cons_2, h_cons_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_3;
	hipMalloc (&cons_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_3\n");
	hipMemcpy (cons_3, h_cons_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_4;
	hipMalloc (&cons_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_4\n");
	hipMemcpy (cons_4, h_cons_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_1;
	hipMalloc (&q_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_1\n");
	hipMemcpy (q_1, h_q_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_2;
	hipMalloc (&q_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_2\n");
	hipMemcpy (q_2, h_q_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_3;
	hipMalloc (&q_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_3\n");
	hipMemcpy (q_3, h_q_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_4;
	hipMalloc (&q_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_4\n");
	hipMemcpy (q_4, h_q_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 4, 4);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(M, blockconfig.y), ceil(L, blockconfig.z));
	hypterm <<<gridconfig, blockconfig>>> (flux_0, flux_1, flux_2, flux_3, flux_4, cons_1, cons_2, cons_3, cons_4, q_1, q_2, q_3, q_4, -dxinv0, dxinv1, dxinv2, L, M, N);

	hipMemcpy (h_flux_0, flux_0, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_1, flux_1, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_3, flux_3, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_4, flux_4, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_2, flux_2, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
}
