#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void hypterm (double * __restrict__ flux_in_0, double * __restrict__ flux_in_1, double * __restrict__ flux_in_2, double * __restrict__ flux_in_3, double * __restrict__ flux_in_4, double * __restrict__ cons_in_1, double * __restrict__ cons_in_2, double * __restrict__ cons_in_3, double * __restrict__ cons_in_4, double * __restrict__ q_in_1, double * __restrict__ q_in_2, double * __restrict__ q_in_3, double * __restrict__ q_in_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	int blockdim_k= (int)(blockDim.z);
	int k0 = (int)(blockIdx.z)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.z);

	double (*flux_0)[308][308] = (double (*)[308][308])flux_in_0;
	double (*flux_1)[308][308] = (double (*)[308][308])flux_in_1;
	double (*flux_2)[308][308] = (double (*)[308][308])flux_in_2;
	double (*flux_3)[308][308] = (double (*)[308][308])flux_in_3;
	double (*flux_4)[308][308] = (double (*)[308][308])flux_in_4;
	double (*q_1)[308][308] = (double (*)[308][308])q_in_1;
	double (*q_2)[308][308] = (double (*)[308][308])q_in_2;
	double (*q_3)[308][308] = (double (*)[308][308])q_in_3;
	double (*q_4)[308][308] = (double (*)[308][308])q_in_4;
	double (*cons_1)[308][308] = (double (*)[308][308])cons_in_1;
	double (*cons_2)[308][308] = (double (*)[308][308])cons_in_2;
	double (*cons_3)[308][308] = (double (*)[308][308])cons_in_3;
	double (*cons_4)[308][308] = (double (*)[308][308])cons_in_4;

	if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
#pragma begin stencil1 unroll k=1,j=1,i=1
		flux_0[k][j][i] = ((0.8f*(cons_1[k][j][i+1] - cons_1[k][j][i-1]) - 0.2f*(cons_1[k][j][i+2] - cons_1[k][j][i-2]) + 0.038f*(cons_1[k][j][i+3] - cons_1[k][j][i-3]) - 0.0035f*(cons_1[k][j][i+4] - cons_1[k][j][i-4]))*dxinv0); 
		flux_1[k][j][i] = ((0.8f*(cons_1[k][j][i+1]*q_1[k][j][i+1]-cons_1[k][j][i-1]*q_1[k][j][i-1]+(q_4[k][j][i+1]-q_4[k][j][i-1]))-0.2f*(cons_1[k][j][i+2]*q_1[k][j][i+2]-cons_1[k][j][i-2]*q_1[k][j][i-2]+(q_4[k][j][i+2]-q_4[k][j][i-2]))+0.038f*(cons_1[k][j][i+3]*q_1[k][j][i+3]-cons_1[k][j][i-3]*q_1[k][j][i-3]+(q_4[k][j][i+3]-q_4[k][j][i-3]))-0.0035f*(cons_1[k][j][i+4]*q_1[k][j][i+4]-cons_1[k][j][i-4]*q_1[k][j][i-4]+(q_4[k][j][i+4]-q_4[k][j][i-4])))*dxinv0); 
		flux_2[k][j][i] = ((0.8f*(cons_2[k][j][i+1]*q_1[k][j][i+1]-cons_2[k][j][i-1]*q_1[k][j][i-1])-0.2f*(cons_2[k][j][i+2]*q_1[k][j][i+2]-cons_2[k][j][i-2]*q_1[k][j][i-2])+0.038f*(cons_2[k][j][i+3]*q_1[k][j][i+3]-cons_2[k][j][i-3]*q_1[k][j][i-3])-0.0035f*(cons_2[k][j][i+4]*q_1[k][j][i+4]-cons_2[k][j][i-4]*q_1[k][j][i-4]))*dxinv0); 
		flux_3[k][j][i] = ((0.8f*(cons_3[k][j][i+1]*q_1[k][j][i+1]-cons_3[k][j][i-1]*q_1[k][j][i-1])-0.2f*(cons_3[k][j][i+2]*q_1[k][j][i+2]-cons_3[k][j][i-2]*q_1[k][j][i-2])+0.038f*(cons_3[k][j][i+3]*q_1[k][j][i+3]-cons_3[k][j][i-3]*q_1[k][j][i-3])-0.0035f*(cons_3[k][j][i+4]*q_1[k][j][i+4]-cons_3[k][j][i-4]*q_1[k][j][i-4]))*dxinv0); 
		flux_4[k][j][i] = ((0.8f*(cons_4[k][j][i+1]*q_1[k][j][i+1]-cons_4[k][j][i-1]*q_1[k][j][i-1]+(q_4[k][j][i+1]*q_1[k][j][i+1]-q_4[k][j][i-1]*q_1[k][j][i-1]))-0.2f*(cons_4[k][j][i+2]*q_1[k][j][i+2]-cons_4[k][j][i-2]*q_1[k][j][i-2]+(q_4[k][j][i+2]*q_1[k][j][i+2]-q_4[k][j][i-2]*q_1[k][j][i-2]))+0.038f*(cons_4[k][j][i+3]*q_1[k][j][i+3]-cons_4[k][j][i-3]*q_1[k][j][i-3]+(q_4[k][j][i+3]*q_1[k][j][i+3]-q_4[k][j][i-3]*q_1[k][j][i-3]))-0.0035f*(cons_4[k][j][i+4]*q_1[k][j][i+4]-cons_4[k][j][i-4]*q_1[k][j][i-4]+(q_4[k][j][i+4]*q_1[k][j][i+4]-q_4[k][j][i-4]*q_1[k][j][i-4])))*dxinv0); 

		flux_0[k][j][i] -= ((0.8f*(cons_2[k][j+1][i] - cons_2[k][j-1][i]) - 0.2f*(cons_2[k][j+2][i] - cons_2[k][j-2][i]) + 0.038f*(cons_2[k][j+3][i] - cons_2[k][j-3][i]) - 0.0035f*(cons_2[k][j+4][i] - cons_2[k][j-4][i]))*dxinv1); 
		flux_1[k][j][i] -= (0.8f*(cons_1[k][j+1][i]*q_2[k][j+1][i]-cons_1[k][j-1][i]*q_2[k][j-1][i])-0.2f*(cons_1[k][j+2][i]*q_2[k][j+2][i]-cons_1[k][j-2][i]*q_2[k][j-2][i])+0.038f*(cons_1[k][j+3][i]*q_2[k][j+3][i]-cons_1[k][j-3][i]*q_2[k][j-3][i])-0.0035f*(cons_1[k][j+4][i]*q_2[k][j+4][i]-cons_1[k][j-4][i]*q_2[k][j-4][i]))*dxinv1; 
		flux_2[k][j][i] -= (0.8f*(cons_2[k][j+1][i]*q_2[k][j+1][i]-cons_2[k][j-1][i]*q_2[k][j-1][i]+(q_4[k][j+1][i]-q_4[k][j-1][i]))-0.2f*(cons_2[k][j+2][i]*q_2[k][j+2][i]-cons_2[k][j-2][i]*q_2[k][j-2][i]+(q_4[k][j+2][i]-q_4[k][j-2][i]))+0.038f*(cons_2[k][j+3][i]*q_2[k][j+3][i]-cons_2[k][j-3][i]*q_2[k][j-3][i]+(q_4[k][j+3][i]-q_4[k][j-3][i]))-0.0035f*(cons_2[k][j+4][i]*q_2[k][j+4][i]-cons_2[k][j-4][i]*q_2[k][j-4][i]+(q_4[k][j+4][i]-q_4[k][j-4][i])))*dxinv1; 
		flux_3[k][j][i] -= (0.8f*(cons_3[k][j+1][i]*q_2[k][j+1][i]-cons_3[k][j-1][i]*q_2[k][j-1][i])-0.2f*(cons_3[k][j+2][i]*q_2[k][j+2][i]-cons_3[k][j-2][i]*q_2[k][j-2][i])+0.038f*(cons_3[k][j+3][i]*q_2[k][j+3][i]-cons_3[k][j-3][i]*q_2[k][j-3][i])-0.0035f*(cons_3[k][j+4][i]*q_2[k][j+4][i]-cons_3[k][j-4][i]*q_2[k][j-4][i]))*dxinv1; 
		flux_4[k][j][i] -= (0.8f*(cons_4[k][j+1][i]*q_2[k][j+1][i]-cons_4[k][j-1][i]*q_2[k][j-1][i]+(q_4[k][j+1][i]*q_2[k][j+1][i]-q_4[k][j-1][i]*q_2[k][j-1][i]))-0.2f*(cons_4[k][j+2][i]*q_2[k][j+2][i]-cons_4[k][j-2][i]*q_2[k][j-2][i]+(q_4[k][j+2][i]*q_2[k][j+2][i]-q_4[k][j-2][i]*q_2[k][j-2][i]))+0.038f*(cons_4[k][j+3][i]*q_2[k][j+3][i]-cons_4[k][j-3][i]*q_2[k][j-3][i]+(q_4[k][j+3][i]*q_2[k][j+3][i]-q_4[k][j-3][i]*q_2[k][j-3][i]))-0.0035f*(cons_4[k][j+4][i]*q_2[k][j+4][i]-cons_4[k][j-4][i]*q_2[k][j-4][i]+(q_4[k][j+4][i]*q_2[k][j+4][i]-q_4[k][j-4][i]*q_2[k][j-4][i])))*dxinv1; 
#pragma end stencil1

		flux_0[k][j][i] -= ((0.8f*(cons_3[k+1][j][i] - cons_3[k-1][j][i]) - 0.2f*(cons_3[k+2][j][i] - cons_3[k-2][j][i]) + 0.038f*(cons_3[k+3][j][i] - cons_3[k-3][j][i]) - 0.0035f*(cons_3[k+4][j][i] - cons_3[k-4][j][i]))*dxinv2); 
		flux_1[k][j][i] -= (0.8f*(cons_1[k+1][j][i]*q_3[k+1][j][i]-cons_1[k-1][j][i]*q_3[k-1][j][i])-0.2f*(cons_1[k+2][j][i]*q_3[k+2][j][i]-cons_1[k-2][j][i]*q_3[k-2][j][i])+0.038f*(cons_1[k+3][j][i]*q_3[k+3][j][i]-cons_1[k-3][j][i]*q_3[k-3][j][i])-0.0035f*(cons_1[k+4][j][i]*q_3[k+4][j][i]-cons_1[k-4][j][i]*q_3[k-4][j][i]))*dxinv2; 
		flux_2[k][j][i] -= (0.8f*(cons_2[k+1][j][i]*q_3[k+1][j][i]-cons_2[k-1][j][i]*q_3[k-1][j][i])-0.2f*(cons_2[k+2][j][i]*q_3[k+2][j][i]-cons_2[k-2][j][i]*q_3[k-2][j][i])+0.038f*(cons_2[k+3][j][i]*q_3[k+3][j][i]-cons_2[k-3][j][i]*q_3[k-3][j][i])-0.0035f*(cons_2[k+4][j][i]*q_3[k+4][j][i]-cons_2[k-4][j][i]*q_3[k-4][j][i]))*dxinv2; 
		flux_3[k][j][i] -= (0.8f*(cons_3[k+1][j][i]*q_3[k+1][j][i]-cons_3[k-1][j][i]*q_3[k-1][j][i]+(q_4[k+1][j][i]-q_4[k-1][j][i]))-0.2f*(cons_3[k+2][j][i]*q_3[k+2][j][i]-cons_3[k-2][j][i]*q_3[k-2][j][i]+(q_4[k+2][j][i]-q_4[k-2][j][i]))+0.038f*(cons_3[k+3][j][i]*q_3[k+3][j][i]-cons_3[k-3][j][i]*q_3[k-3][j][i]+(q_4[k+3][j][i]-q_4[k-3][j][i]))-0.0035f*(cons_3[k+4][j][i]*q_3[k+4][j][i]-cons_3[k-4][j][i]*q_3[k-4][j][i]+(q_4[k+4][j][i]-q_4[k-4][j][i])))*dxinv2; 
		flux_4[k][j][i] -= (0.8f*(cons_4[k+1][j][i]*q_3[k+1][j][i]-cons_4[k-1][j][i]*q_3[k-1][j][i]+(q_4[k+1][j][i]*q_3[k+1][j][i]-q_4[k-1][j][i]*q_3[k-1][j][i]))-0.2f*(cons_4[k+2][j][i]*q_3[k+2][j][i]-cons_4[k-2][j][i]*q_3[k-2][j][i]+(q_4[k+2][j][i]*q_3[k+2][j][i]-q_4[k-2][j][i]*q_3[k-2][j][i]))+0.038f*(cons_4[k+3][j][i]*q_3[k+3][j][i]-cons_4[k-3][j][i]*q_3[k-3][j][i]+(q_4[k+3][j][i]*q_3[k+3][j][i]-q_4[k-3][j][i]*q_3[k-3][j][i]))-0.0035f*(cons_4[k+4][j][i]*q_3[k+4][j][i]-cons_4[k-4][j][i]*q_3[k-4][j][i]+(q_4[k+4][j][i]*q_3[k+4][j][i]-q_4[k-4][j][i]*q_3[k-4][j][i])))*dxinv2; 
	} 
}

extern "C" void host_code (double *h_flux_0, double *h_flux_1, double *h_flux_2, double *h_flux_3, double *h_flux_4, double *h_cons_1, double *h_cons_2, double *h_cons_3, double *h_cons_4, double *h_q_1, double *h_q_2, double *h_q_3, double *h_q_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	double *flux_0;
	hipMalloc (&flux_0, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_0\n");
	hipMemcpy (flux_0, h_flux_0, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_1;
	hipMalloc (&flux_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_1\n");
	hipMemcpy (flux_1, h_flux_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_2;
	hipMalloc (&flux_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_2\n");
	hipMemcpy (flux_2, h_flux_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_3;
	hipMalloc (&flux_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_3\n");
	hipMemcpy (flux_3, h_flux_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_4;
	hipMalloc (&flux_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_4\n");
	hipMemcpy (flux_4, h_flux_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_1;
	hipMalloc (&cons_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_1\n");
	hipMemcpy (cons_1, h_cons_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_2;
	hipMalloc (&cons_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_2\n");
	hipMemcpy (cons_2, h_cons_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_3;
	hipMalloc (&cons_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_3\n");
	hipMemcpy (cons_3, h_cons_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_4;
	hipMalloc (&cons_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_4\n");
	hipMemcpy (cons_4, h_cons_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_1;
	hipMalloc (&q_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_1\n");
	hipMemcpy (q_1, h_q_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_2;
	hipMalloc (&q_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_2\n");
	hipMemcpy (q_2, h_q_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_3;
	hipMalloc (&q_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_3\n");
	hipMemcpy (q_3, h_q_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_4;
	hipMalloc (&q_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_4\n");
	hipMemcpy (q_4, h_q_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 4, 4);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(M, blockconfig.y), ceil(L, blockconfig.z));
	hypterm <<<gridconfig, blockconfig>>> (flux_0, flux_1, flux_2, flux_3, flux_4, cons_1, cons_2, cons_3, cons_4, q_1, q_2, q_3, q_4, -dxinv0, dxinv1, dxinv2, L, M, N);

	hipMemcpy (h_flux_0, flux_0, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_1, flux_1, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_3, flux_3, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_4, flux_4, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_2, flux_2, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
}
