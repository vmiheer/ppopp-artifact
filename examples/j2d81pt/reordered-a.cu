#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void j2d81pt (double * __restrict__ l_in, double * __restrict__ l_out, int N) {
	//Determing the block's indices
	int i0 = (int)(blockIdx.x)*(int)(blockDim.x);
	int i = max(i0,0) + (int)(threadIdx.x);
	int j0 = 4*(int)(blockIdx.y)*(int)(blockDim.y);
	int j = max(j0,0) + 4*(int)(threadIdx.y);

	double (*in)[8200] = (double (*)[8200]) l_in;
	double (*out)[8200] = (double (*)[8200]) l_out;

	if (i>=0 & j>=0 & i<=N-9 & j<=N-9) {
double outjc0ic0 = 3.18622 * in[j][i];
outjc0ic0 += 3.18622 * in[j][i+8];
outjc0ic0 += 4.5339 * in[j][i+1];
outjc0ic0 += 4.5339 * in[j][i+7];
outjc0ic0 += -0.000357 * in[j][i+2];
outjc0ic0 += -0.000357 * in[j][i+6];
outjc0ic0 += 0.002856 * in[j][i+3];
outjc0ic0 += 0.002856 * in[j][i+5];
outjc0ic0 += -0.00508225 * in[j][i+4];
outjc0ic0 += 3.18622 * in[j+8][i];
double outjp1ic0 = 4.5339 * in[j+8][i];
double outjp2ic0 = -0.000357 * in[j+8][i];
double outjp3ic0 = 0.002856 * in[j+8][i];
outjc0ic0 += 3.18622 * in[j+8][i+8];
outjp1ic0 += 4.5339 * in[j+8][i+8];
outjp2ic0 += -0.000357 * in[j+8][i+8];
outjp3ic0 += 0.002856 * in[j+8][i+8];
outjc0ic0 += 4.5339 * in[j+1][i];
outjp1ic0 += 3.18622 * in[j+1][i];
outjc0ic0 += 4.5339 * in[j+1][i+8];
outjp1ic0 += 3.18622 * in[j+1][i+8];
outjc0ic0 += 4.5339 * in[j+7][i];
outjp1ic0 += -0.000357 * in[j+7][i];
outjp2ic0 += 0.002856 * in[j+7][i];
outjp3ic0 += -0.00508225 * in[j+7][i];
outjc0ic0 += 4.5339 * in[j+7][i+8];
outjp1ic0 += -0.000357 * in[j+7][i+8];
outjp2ic0 += 0.002856 * in[j+7][i+8];
outjp3ic0 += -0.00508225 * in[j+7][i+8];
outjc0ic0 += 4.5339 * in[j+8][i+1];
outjp1ic0 += 0.00064516 * in[j+8][i+1];
outjp2ic0 += -0.00508 * in[j+8][i+1];
outjp3ic0 += 0.04064 * in[j+8][i+1];
outjc0ic0 += 4.5339 * in[j+8][i+7];
outjp1ic0 += 0.00064516 * in[j+8][i+7];
outjp2ic0 += -0.00508 * in[j+8][i+7];
outjp3ic0 += 0.04064 * in[j+8][i+7];
outjc0ic0 += -0.000357 * in[j+2][i];
outjp1ic0 += 4.5339 * in[j+2][i];
outjp2ic0 += 3.18622 * in[j+2][i];
outjc0ic0 += -0.000357 * in[j+2][i+8];
outjp1ic0 += 4.5339 * in[j+2][i+8];
outjp2ic0 += 3.18622 * in[j+2][i+8];
outjc0ic0 += -0.000357 * in[j+6][i];
outjp1ic0 += 0.002856 * in[j+6][i];
outjp2ic0 += -0.00508225 * in[j+6][i];
outjp3ic0 += 0.002856 * in[j+6][i];
outjc0ic0 += -0.000357 * in[j+6][i+8];
outjp1ic0 += 0.002856 * in[j+6][i+8];
outjp2ic0 += -0.00508225 * in[j+6][i+8];
outjp3ic0 += 0.002856 * in[j+6][i+8];
outjc0ic0 += -0.000357 * in[j+8][i+2];
outjp1ic0 += -0.00508 * in[j+8][i+2];
outjp2ic0 += 0.04 * in[j+8][i+2];
outjp3ic0 += -0.32 * in[j+8][i+2];
outjc0ic0 += -0.000357 * in[j+8][i+6];
outjp1ic0 += -0.00508 * in[j+8][i+6];
outjp2ic0 += 0.04 * in[j+8][i+6];
outjp3ic0 += -0.32 * in[j+8][i+6];
outjc0ic0 += 0.002856 * in[j+3][i];
outjp1ic0 += -0.000357 * in[j+3][i];
outjp2ic0 += 4.5339 * in[j+3][i];
outjp3ic0 += 3.18622 * in[j+3][i];
outjc0ic0 += 0.002856 * in[j+3][i+8];
outjp1ic0 += -0.000357 * in[j+3][i+8];
outjp2ic0 += 4.5339 * in[j+3][i+8];
outjp3ic0 += 3.18622 * in[j+3][i+8];
outjc0ic0 += 0.002856 * in[j+5][i];
outjp1ic0 += -0.00508225 * in[j+5][i];
outjp2ic0 += 0.002856 * in[j+5][i];
outjp3ic0 += -0.000357 * in[j+5][i];
outjc0ic0 += 0.002856 * in[j+5][i+8];
outjp1ic0 += -0.00508225 * in[j+5][i+8];
outjp2ic0 += 0.002856 * in[j+5][i+8];
outjp3ic0 += -0.000357 * in[j+5][i+8];
outjc0ic0 += 0.002856 * in[j+8][i+3];
outjp1ic0 += 0.04064 * in[j+8][i+3];
outjp2ic0 += -0.32 * in[j+8][i+3];
outjp3ic0 += 2.56 * in[j+8][i+3];
outjc0ic0 += 0.002856 * in[j+8][i+5];
outjp1ic0 += 0.04064 * in[j+8][i+5];
outjp2ic0 += -0.32 * in[j+8][i+5];
outjp3ic0 += 2.56 * in[j+8][i+5];
outjc0ic0 += -0.00508225 * in[j+4][i+8];
outjp1ic0 += 0.002856 * in[j+4][i+8];
outjp2ic0 += -0.000357 * in[j+4][i+8];
outjp3ic0 += 4.5339 * in[j+4][i+8];
outjc0ic0 += -0.00508225 * in[j+4][i];
outjp1ic0 += 0.002856 * in[j+4][i];
outjp2ic0 += -0.000357 * in[j+4][i];
outjp3ic0 += 4.5339 * in[j+4][i];
outjc0ic0 += -0.00508225 * in[j+8][i+4];
outjp1ic0 += -0.0723189 * in[j+8][i+4];
outjp2ic0 += 0.56944 * in[j+8][i+4];
outjp3ic0 += -4.55552 * in[j+8][i+4];
outjc0ic0 += 0.00064516 * in[j+1][i+1];
outjp1ic0 += 4.5339 * in[j+1][i+1];
outjc0ic0 += 0.00064516 * in[j+1][i+7];
outjp1ic0 += 4.5339 * in[j+1][i+7];
outjc0ic0 += 0.00064516 * in[j+7][i+1];
outjp1ic0 += -0.00508 * in[j+7][i+1];
outjp2ic0 += 0.04064 * in[j+7][i+1];
outjp3ic0 += -0.0723189 * in[j+7][i+1];
outjc0ic0 += 0.00064516 * in[j+7][i+7];
outjp1ic0 += -0.00508 * in[j+7][i+7];
outjp2ic0 += 0.04064 * in[j+7][i+7];
outjp3ic0 += -0.0723189 * in[j+7][i+7];
outjc0ic0 += -0.00508 * in[j+1][i+2];
outjp1ic0 += -0.000357 * in[j+1][i+2];
outjc0ic0 += -0.00508 * in[j+1][i+6];
outjp1ic0 += -0.000357 * in[j+1][i+6];
outjc0ic0 += -0.00508 * in[j+2][i+1];
outjp1ic0 += 0.00064516 * in[j+2][i+1];
outjp2ic0 += 4.5339 * in[j+2][i+1];
outjc0ic0 += -0.00508 * in[j+2][i+7];
outjp1ic0 += 0.00064516 * in[j+2][i+7];
outjp2ic0 += 4.5339 * in[j+2][i+7];
outjc0ic0 += -0.00508 * in[j+6][i+1];
outjp1ic0 += 0.04064 * in[j+6][i+1];
outjp2ic0 += -0.0723189 * in[j+6][i+1];
outjp3ic0 += 0.04064 * in[j+6][i+1];
outjc0ic0 += -0.00508 * in[j+6][i+7];
outjp1ic0 += 0.04064 * in[j+6][i+7];
outjp2ic0 += -0.0723189 * in[j+6][i+7];
outjp3ic0 += 0.04064 * in[j+6][i+7];
outjc0ic0 += -0.00508 * in[j+7][i+2];
outjp1ic0 += 0.04 * in[j+7][i+2];
outjp2ic0 += -0.32 * in[j+7][i+2];
outjp3ic0 += 0.56944 * in[j+7][i+2];
outjc0ic0 += -0.00508 * in[j+7][i+6];
outjp1ic0 += 0.04 * in[j+7][i+6];
outjp2ic0 += -0.32 * in[j+7][i+6];
outjp3ic0 += 0.56944 * in[j+7][i+6];
outjc0ic0 += 0.04064 * in[j+1][i+3];
outjp1ic0 += 0.002856 * in[j+1][i+3];
outjc0ic0 += 0.04064 * in[j+1][i+5];
outjp1ic0 += 0.002856 * in[j+1][i+5];
outjc0ic0 += 0.04064 * in[j+3][i+1];
outjp1ic0 += -0.00508 * in[j+3][i+1];
outjp2ic0 += 0.00064516 * in[j+3][i+1];
outjp3ic0 += 4.5339 * in[j+3][i+1];
outjc0ic0 += 0.04064 * in[j+3][i+7];
outjp1ic0 += -0.00508 * in[j+3][i+7];
outjp2ic0 += 0.00064516 * in[j+3][i+7];
outjp3ic0 += 4.5339 * in[j+3][i+7];
outjc0ic0 += 0.04064 * in[j+5][i+1];
outjp1ic0 += -0.0723189 * in[j+5][i+1];
outjp2ic0 += 0.04064 * in[j+5][i+1];
outjp3ic0 += -0.00508 * in[j+5][i+1];
outjc0ic0 += 0.04064 * in[j+5][i+7];
outjp1ic0 += -0.0723189 * in[j+5][i+7];
outjp2ic0 += 0.04064 * in[j+5][i+7];
outjp3ic0 += -0.00508 * in[j+5][i+7];
outjc0ic0 += 0.04064 * in[j+7][i+3];
outjp1ic0 += -0.32 * in[j+7][i+3];
outjp2ic0 += 2.56 * in[j+7][i+3];
outjp3ic0 += -4.55552 * in[j+7][i+3];
outjc0ic0 += 0.04064 * in[j+7][i+5];
outjp1ic0 += -0.32 * in[j+7][i+5];
outjp2ic0 += 2.56 * in[j+7][i+5];
outjp3ic0 += -4.55552 * in[j+7][i+5];
outjc0ic0 += -0.0723189 * in[j+1][i+4];
outjp1ic0 += -0.00508225 * in[j+1][i+4];
outjc0ic0 += -0.0723189 * in[j+4][i+1];
outjp1ic0 += 0.04064 * in[j+4][i+1];
outjp2ic0 += -0.00508 * in[j+4][i+1];
outjp3ic0 += 0.00064516 * in[j+4][i+1];
outjc0ic0 += -0.0723189 * in[j+4][i+7];
outjp1ic0 += 0.04064 * in[j+4][i+7];
outjp2ic0 += -0.00508 * in[j+4][i+7];
outjp3ic0 += 0.00064516 * in[j+4][i+7];
outjc0ic0 += -0.0723189 * in[j+7][i+4];
outjp1ic0 += 0.56944 * in[j+7][i+4];
outjp2ic0 += -4.55552 * in[j+7][i+4];
outjp3ic0 += in[j+7][i+4] * 8.10655;
outjc0ic0 += 0.04 * in[j+2][i+2];
outjp1ic0 += -0.00508 * in[j+2][i+2];
outjp2ic0 += -0.000357 * in[j+2][i+2];
outjc0ic0 += 0.04 * in[j+2][i+6];
outjp1ic0 += -0.00508 * in[j+2][i+6];
outjp2ic0 += -0.000357 * in[j+2][i+6];
outjc0ic0 += 0.04 * in[j+6][i+2];
outjp1ic0 += -0.32 * in[j+6][i+2];
outjp2ic0 += 0.56944 * in[j+6][i+2];
outjp3ic0 += -0.32 * in[j+6][i+2];
outjc0ic0 += 0.04 * in[j+6][i+6];
outjp1ic0 += -0.32 * in[j+6][i+6];
outjp2ic0 += 0.56944 * in[j+6][i+6];
outjp3ic0 += -0.32 * in[j+6][i+6];
outjc0ic0 += -0.32 * in[j+2][i+3];
outjp1ic0 += 0.04064 * in[j+2][i+3];
outjp2ic0 += 0.002856 * in[j+2][i+3];
outjc0ic0 += -0.32 * in[j+2][i+5];
outjp1ic0 += 0.04064 * in[j+2][i+5];
outjp2ic0 += 0.002856 * in[j+2][i+5];
outjc0ic0 += -0.32 * in[j+3][i+2];
outjp1ic0 += 0.04 * in[j+3][i+2];
outjp2ic0 += -0.00508 * in[j+3][i+2];
outjp3ic0 += -0.000357 * in[j+3][i+2];
outjc0ic0 += -0.32 * in[j+3][i+6];
outjp1ic0 += 0.04 * in[j+3][i+6];
outjp2ic0 += -0.00508 * in[j+3][i+6];
outjp3ic0 += -0.000357 * in[j+3][i+6];
outjc0ic0 += -0.32 * in[j+5][i+2];
outjp1ic0 += 0.56944 * in[j+5][i+2];
outjp2ic0 += -0.32 * in[j+5][i+2];
outjp3ic0 += 0.04 * in[j+5][i+2];
outjc0ic0 += -0.32 * in[j+5][i+6];
outjp1ic0 += 0.56944 * in[j+5][i+6];
outjp2ic0 += -0.32 * in[j+5][i+6];
outjp3ic0 += 0.04 * in[j+5][i+6];
outjc0ic0 += -0.32 * in[j+6][i+3];
outjp1ic0 += 2.56 * in[j+6][i+3];
outjp2ic0 += -4.55552 * in[j+6][i+3];
outjp3ic0 += 2.56 * in[j+6][i+3];
outjc0ic0 += -0.32 * in[j+6][i+5];
outjp1ic0 += 2.56 * in[j+6][i+5];
outjp2ic0 += -4.55552 * in[j+6][i+5];
outjp3ic0 += 2.56 * in[j+6][i+5];
outjc0ic0 += 0.56944 * in[j+2][i+4];
outjp1ic0 += -0.0723189 * in[j+2][i+4];
outjp2ic0 += -0.00508225 * in[j+2][i+4];
outjc0ic0 += 0.56944 * in[j+4][i+2];
outjp1ic0 += -0.32 * in[j+4][i+2];
outjp2ic0 += 0.04 * in[j+4][i+2];
outjp3ic0 += -0.00508 * in[j+4][i+2];
outjc0ic0 += 0.56944 * in[j+4][i+6];
outjp1ic0 += -0.32 * in[j+4][i+6];
outjp2ic0 += 0.04 * in[j+4][i+6];
outjp3ic0 += -0.00508 * in[j+4][i+6];
outjc0ic0 += 0.56944 * in[j+6][i+4];
outjp1ic0 += -4.55552 * in[j+6][i+4];
outjp2ic0 += in[j+6][i+4] * 8.10655;
outjp3ic0 += -4.55552 * in[j+6][i+4];
outjc0ic0 += 2.56 * in[j+3][i+3];
outjp1ic0 += -0.32 * in[j+3][i+3];
outjp2ic0 += 0.04064 * in[j+3][i+3];
outjp3ic0 += 0.002856 * in[j+3][i+3];
outjc0ic0 += 2.56 * in[j+3][i+5];
outjp1ic0 += -0.32 * in[j+3][i+5];
outjp2ic0 += 0.04064 * in[j+3][i+5];
outjp3ic0 += 0.002856 * in[j+3][i+5];
outjc0ic0 += 2.56 * in[j+5][i+3];
outjp1ic0 += -4.55552 * in[j+5][i+3];
outjp2ic0 += 2.56 * in[j+5][i+3];
outjp3ic0 += -0.32 * in[j+5][i+3];
outjc0ic0 += 2.56 * in[j+5][i+5];
outjp1ic0 += -4.55552 * in[j+5][i+5];
outjp2ic0 += 2.56 * in[j+5][i+5];
outjp3ic0 += -0.32 * in[j+5][i+5];
outjc0ic0 += -4.55552 * in[j+3][i+4];
outjp1ic0 += 0.56944 * in[j+3][i+4];
outjp2ic0 += -0.0723189 * in[j+3][i+4];
outjp3ic0 += -0.00508225 * in[j+3][i+4];
outjc0ic0 += -4.55552 * in[j+4][i+3];
outjp1ic0 += 2.56 * in[j+4][i+3];
outjp2ic0 += -0.32 * in[j+4][i+3];
outjp3ic0 += 0.04064 * in[j+4][i+3];
outjc0ic0 += -4.55552 * in[j+4][i+5];
outjp1ic0 += 2.56 * in[j+4][i+5];
outjp2ic0 += -0.32 * in[j+4][i+5];
outjp3ic0 += 0.04064 * in[j+4][i+5];
outjc0ic0 += -4.55552 * in[j+5][i+4];
outjp1ic0 += in[j+5][i+4] * 8.10655;
outjp2ic0 += -4.55552 * in[j+5][i+4];
outjp3ic0 += 0.56944 * in[j+5][i+4];
outjc0ic0 += in[j+4][i+4] * 8.10655;
outjp1ic0 += -4.55552 * in[j+4][i+4];
outjp2ic0 += 0.56944 * in[j+4][i+4];
outjp3ic0 += -0.0723189 * in[j+4][i+4];

outjp1ic0 += 3.18622 * in[j+9][i];
outjp2ic0 += 4.5339 * in[j+9][i];
outjp3ic0 += -0.000357 * in[j+9][i];
outjp1ic0 += 3.18622 * in[j+9][i+8];
outjp2ic0 += 4.5339 * in[j+9][i+8];
outjp3ic0 += -0.000357 * in[j+9][i+8];
outjp1ic0 += 4.5339 * in[j+9][i+1];
outjp2ic0 += 0.00064516 * in[j+9][i+1];
outjp3ic0 += -0.00508 * in[j+9][i+1];
outjp1ic0 += 4.5339 * in[j+9][i+7];
outjp2ic0 += 0.00064516 * in[j+9][i+7];
outjp3ic0 += -0.00508 * in[j+9][i+7];
outjp1ic0 += -0.000357 * in[j+9][i+2];
outjp2ic0 += -0.00508 * in[j+9][i+2];
outjp3ic0 += 0.04 * in[j+9][i+2];
outjp1ic0 += -0.000357 * in[j+9][i+6];
outjp2ic0 += -0.00508 * in[j+9][i+6];
outjp3ic0 += 0.04 * in[j+9][i+6];
outjp1ic0 += 0.002856 * in[j+9][i+3];
outjp2ic0 += 0.04064 * in[j+9][i+3];
outjp3ic0 += -0.32 * in[j+9][i+3];
outjp1ic0 += 0.002856 * in[j+9][i+5];
outjp2ic0 += 0.04064 * in[j+9][i+5];
outjp3ic0 += -0.32 * in[j+9][i+5];
outjp1ic0 += -0.00508225 * in[j+9][i+4];
outjp2ic0 += -0.0723189 * in[j+9][i+4];
outjp3ic0 += 0.56944 * in[j+9][i+4];

outjp2ic0 += 3.18622 * in[j+10][i];
outjp3ic0 += 4.5339 * in[j+10][i];
outjp2ic0 += 3.18622 * in[j+10][i+8];
outjp3ic0 += 4.5339 * in[j+10][i+8];
outjp2ic0 += 4.5339 * in[j+10][i+1];
outjp3ic0 += 0.00064516 * in[j+10][i+1];
outjp2ic0 += 4.5339 * in[j+10][i+7];
outjp3ic0 += 0.00064516 * in[j+10][i+7];
outjp2ic0 += -0.000357 * in[j+10][i+2];
outjp3ic0 += -0.00508 * in[j+10][i+2];
outjp2ic0 += -0.000357 * in[j+10][i+6];
outjp3ic0 += -0.00508 * in[j+10][i+6];
outjp2ic0 += 0.002856 * in[j+10][i+3];
outjp3ic0 += 0.04064 * in[j+10][i+3];
outjp2ic0 += 0.002856 * in[j+10][i+5];
outjp3ic0 += 0.04064 * in[j+10][i+5];
outjp2ic0 += -0.00508225 * in[j+10][i+4];
outjp3ic0 += -0.0723189 * in[j+10][i+4];

outjp3ic0 += 3.18622 * in[j+11][i];
outjp3ic0 += 3.18622 * in[j+11][i+8];
outjp3ic0 += 4.5339 * in[j+11][i+1];
outjp3ic0 += 4.5339 * in[j+11][i+7];
outjp3ic0 += -0.000357 * in[j+11][i+2];
outjp3ic0 += -0.000357 * in[j+11][i+6];
outjp3ic0 += 0.002856 * in[j+11][i+3];
outjp3ic0 += 0.002856 * in[j+11][i+5];
outjp3ic0 += -0.00508225 * in[j+11][i+4];

out[j][i] = outjc0ic0;
out[j+1][i] = outjp1ic0;
out[j+2][i] = outjp2ic0;
out[j+3][i] = outjp3ic0;
	} 
}

extern "C" void host_code (double *h_in, double *h_out, int N) {
	double *in;
	hipMalloc (&in, sizeof(double)*N*N);
	check_error ("Failed to allocate device memory for in\n");
	hipMemcpy (in, h_in, sizeof(double)*N*N, hipMemcpyHostToDevice);
	double *out;
	hipMalloc (&out, sizeof(double)*N*N);
	check_error ("Failed to allocate device memory for out\n");

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, 4*blockconfig.y));

	j2d81pt<<<gridconfig, blockconfig>>> (in, out, N);

	hipMemcpy (h_out, out, sizeof(double)*N*N, hipMemcpyDeviceToHost);
	hipFree (in); 
	hipFree (out);
}
