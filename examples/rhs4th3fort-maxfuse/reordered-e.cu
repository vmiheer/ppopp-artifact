#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void sw4 (double * uacc_in_0, double * uacc_in_1, double * uacc_in_2, double * __restrict__ u_in_0, double * __restrict__ u_in_1, double * __restrict__ u_in_2, double * __restrict__ mu_in, double * __restrict__ la_in, double * strx, double * stry, double * strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	int blockdim_k= (int)(blockDim.z);
	int k0 = (int)(blockIdx.z)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.z);

	// Assumptions 
	int a1 = 1;
	double h = 3.7;
	double cof = 1e0 / ( h *  h);

	double (*uacc_0)[304][304] = (double (*)[304][304])uacc_in_0;
	double (*uacc_1)[304][304] = (double (*)[304][304])uacc_in_1;
	double (*uacc_2)[304][304] = (double (*)[304][304])uacc_in_2;
	double (*u_0)[304][304] = (double (*)[304][304])u_in_0;
	double (*u_1)[304][304] = (double (*)[304][304])u_in_1;
	double (*u_2)[304][304] = (double (*)[304][304])u_in_2;
	double (*mu)[304][304] = (double (*)[304][304])mu_in;
	double (*la)[304][304] = (double (*)[304][304])la_in;

	double mux1, mux2, mux3, mux4, muy1, muy2, muy3, muy4, muz1, muz2, muz3, muz4;
	double r1, r2, r3;
	if (i>=2 & j>=2 & k>=2 & i<=N-3 & j<=N-3 & k<=N-3) {
double mux1;
double mux2;
double mux3;
double mux4;
double muy1;
double muy2;
double muy3;
double muy4;
double muz1;
double muz2;
double muz3;
double muz4;
double _t_10_;
double r1;
double _t_15_;
double _t_5_;
double _t_7_;
double _t_3_;
double _t_9_;
double _t_4_;
double _t_2_;
double _t_6_;
double _t_1_;
double _t_8_;
double _t_21_;
double _t_35_;
double _t_30_;
double _t_32_;
double _t_28_;
double _t_34_;
double r2;
double _t_54_;
double _t_52_;
double _t_56_;
double _t_58_;
double _t_29_;
double _t_27_;
double _t_26_;
double _t_31_;
double _t_33_;
double _t_41_;
double r3;
double _t_46_;
double _t_53_;
double _t_51_;
double _t_55_;
double _t_57_;
double _t_59_;
double _t_100_;
double _t_74_;
double _t_61_;
double _t_87_;
double _t_75_;
double _t_101_;
double _t_62_;
double _t_88_;
double _t_106_;
double _t_78_;
double _t_80_;
double _t_109_;
double _t_83_;
double _t_86_;
double _t_104_;
double _t_112_;
double _t_60_;
double _t_67_;
double _t_91_;
double _t_70_;
double _t_93_;
double _t_96_;
double _t_99_;
double _t_65_;
double _t_73_;
double _t_127_;
double _t_153_;
double _t_140_;
double _t_114_;
double _t_115_;
double _t_141_;
double _t_128_;
double _t_154_;
double _t_120_;
double _t_144_;
double _t_123_;
double _t_146_;
double _t_149_;
double _t_152_;
double _t_118_;
double _t_126_;
double _t_113_;
double _t_133_;
double _t_157_;
double _t_136_;
double _t_159_;
double _t_162_;
double _t_165_;
double _t_131_;
double _t_139_;
double _t_167_;
double _t_180_;
double _t_206_;
double _t_193_;
double _t_168_;
double _t_181_;
double _t_194_;
double _t_207_;
double _t_173_;
double _t_184_;
double _t_176_;
double _t_186_;
double _t_189_;
double _t_192_;
double _t_171_;
double _t_179_;
double _t_166_;
double _t_199_;
double _t_210_;
double _t_202_;
double _t_212_;
double _t_215_;
double _t_218_;
double _t_197_;
double _t_205_;
double uacc_0kc0jc0ic0;
double uacc_1kc0jc0ic0;
double uacc_2kc0jc0ic0;

mux1 = -3.0 / 4.0 * mu[k][j][i-2] * strx[i-2];
mux1 += mu[k][j][i-1] * strx[i-1];
mux1 -= 3.0 / 4.0 * mu[k][j][i] * strx[i];
mux2 = mu[k][j][i-2] * strx[i-2];
mux2 += 3.0 * mu[k][j][i] * strx[i];
mux2 += 3.0 * mu[k][j][i-1] * strx[i-1];
mux2 += mu[k][j][i+1] * strx[i+1];
mux3 = mu[k][j][i-1] * strx[i-1];
mux3 += 3.0 * mu[k][j][i+1] * strx[i+1];
mux3 += 3.0 * mu[k][j][i] * strx[i];
mux3 += mu[k][j][i+2] * strx[i+2];
mux4 = mu[k][j][i+1] * strx[i+1];
mux4 -= 3.0 / 4.0 * mu[k][j][i] * strx[i];
mux4 -= 3.0 / 4.0 * mu[k][j][i+2] * strx[i+2];
muy1 = -3.0 / 4.0 * mu[k][j][i] * stry[j];
muy1 += mu[k][j-1][i] * stry[j-1];
muy1 -= 3.0 / 4.0 * mu[k][j-2][i] * stry[j-2];
muy2 = mu[k][j-2][i] * stry[j-2];
muy2 += 3.0 * mu[k][j][i] * stry[j];
muy2 += 3.0 * mu[k][j-1][i] * stry[j-1];
muy2 += mu[k][j+1][i] * stry[j+1];
muy3 = mu[k][j-1][i] * stry[j-1];
muy3 += 3.0 * mu[k][j+1][i] * stry[j+1];
muy3 += 3.0 * mu[k][j][i] * stry[j];
muy3 += mu[k][j+2][i] * stry[j+2];
muy4 = mu[k][j+1][i] * stry[j+1];
muy4 -= 3.0 / 4.0 * mu[k][j][i] * stry[j];
muy4 -= 3.0 / 4.0 * mu[k][j+2][i] * stry[j+2];
muz1 = -3.0 / 4.0 * mu[k][j][i] * strz[k];
muz1 += mu[k-1][j][i] * strz[k-1];
muz1 -= 3.0 / 4.0 * mu[k-2][j][i] * strz[k-2];
muz2 = mu[k-2][j][i] * strz[k-2];
muz2 += 3.0 * mu[k][j][i] * strz[k];
muz2 += 3.0 * mu[k-1][j][i] * strz[k-1];
muz2 += mu[k+1][j][i] * strz[k+1];
muz3 = mu[k-1][j][i] * strz[k-1];
muz3 += 3.0 * mu[k+1][j][i] * strz[k+1];
muz3 += 3.0 * mu[k][j][i] * strz[k];
muz4 = mu[k+1][j][i] * strz[k+1];
muz4 -= 3.0 / 4.0 * mu[k][j][i] * strz[k];
muz3 += mu[k+2][j][i] * strz[k+2];
muz4 -= 3.0 / 4.0 * mu[k+2][j][i] * strz[k+2];
_t_10_ = muy1 * u_0[k][j-2][i];
_t_10_ += muy2 * u_0[k][j-1][i];
_t_10_ += muy3 * u_0[k][j+1][i];
_t_10_ += muy4 * u_0[k][j+2][i];
_t_10_ -= muy1 * u_0[k][j][i];
_t_10_ -= muy2 * u_0[k][j][i];
_t_10_ -= muy3 * u_0[k][j][i];
_t_10_ -= muy4 * u_0[k][j][i];
r1 = 1.0 / 6.0 * stry[j] * _t_10_;
_t_15_ = -muz1 * u_0[k][j][i];
_t_15_ -= muz2 * u_0[k][j][i];
_t_15_ -= muz3 * u_0[k][j][i];
_t_15_ -= muz4 * u_0[k][j][i];
_t_15_ += muz1 * u_0[k-2][j][i];
_t_15_ += muz2 * u_0[k-1][j][i];
_t_15_ += muz3 * u_0[k+1][j][i];
_t_15_ += muz4 * u_0[k+2][j][i];
r1 += 1.0 / 6.0 * strz[k] * _t_15_;
_t_5_ = -u_0[k][j][i];
_t_5_ += u_0[k][j][i-1];
_t_7_ = -u_0[k][j][i];
_t_7_ += u_0[k][j][i+1];
_t_3_ = -u_0[k][j][i];
_t_9_ = -u_0[k][j][i];
_t_3_ += u_0[k][j][i-2];
_t_9_ += u_0[k][j][i+2];
_t_4_ = 2.0 * mux2;
_t_4_ += la[k][j][i-2] * strx[i-2];
_t_2_ = -3.0 / 4.0 * la[k][j][i-2] * strx[i-2];
_t_2_ += 2.0 * mux1;
_t_2_ += la[k][j][i-1] * strx[i-1];
_t_4_ += 3.0 * la[k][j][i-1] * strx[i-1];
_t_6_ = la[k][j][i-1] * strx[i-1];
_t_6_ += 2.0 * mux3;
_t_2_ -= 3.0 / 4.0 * la[k][j][i] * strx[i];
_t_4_ += 3.0 * la[k][j][i] * strx[i];
_t_6_ += 3.0 * la[k][j][i] * strx[i];
_t_1_ = _t_2_ * _t_3_;
_t_8_ = -3.0 / 4.0 * la[k][j][i] * strx[i];
_t_8_ += 2.0 * mux4;
_t_4_ += la[k][j][i+1] * strx[i+1];
_t_1_ += _t_4_ * _t_5_;
_t_6_ += 3.0 * la[k][j][i+1] * strx[i+1];
_t_8_ += la[k][j][i+1] * strx[i+1];
_t_6_ += la[k][j][i+2] * strx[i+2];
_t_1_ += _t_6_ * _t_7_;
_t_8_ -= 3.0 / 4.0 * la[k][j][i+2] * strx[i+2];
_t_1_ += _t_8_ * _t_9_;
r1 += 1.0 / 6.0 * strx[i] * _t_1_;
_t_21_ = mux1 * u_1[k][j][i-2];
_t_21_ += mux2 * u_1[k][j][i-1];
_t_21_ += mux3 * u_1[k][j][i+1];
_t_21_ += mux4 * u_1[k][j][i+2];
_t_35_ = muz1 * u_1[k-2][j][i];
_t_35_ += muz2 * u_1[k-1][j][i];
_t_35_ += muz3 * u_1[k+1][j][i];
_t_35_ += muz4 * u_1[k+2][j][i];
_t_30_ = u_1[k][j-1][i];
_t_32_ = u_1[k][j+1][i];
_t_28_ = u_1[k][j-2][i];
_t_28_ -= u_1[k][j][i];
_t_30_ -= u_1[k][j][i];
_t_32_ -= u_1[k][j][i];
_t_34_ = -u_1[k][j][i];
_t_21_ -= mux1 * u_1[k][j][i];
_t_21_ -= mux2 * u_1[k][j][i];
_t_21_ -= mux3 * u_1[k][j][i];
_t_21_ -= mux4 * u_1[k][j][i];
_t_35_ -= muz1 * u_1[k][j][i];
_t_35_ -= muz2 * u_1[k][j][i];
_t_35_ -= muz3 * u_1[k][j][i];
_t_35_ -= muz4 * u_1[k][j][i];
r2 = 1.0 / 6.0 * strx[i] * _t_21_;
r2 += 1.0 / 6.0 * strz[k] * _t_35_;
_t_34_ += u_1[k][j+2][i];
_t_54_ = 3.0 * la[k][j][i] * strz[k];
_t_54_ += 2.0 * muz2;
_t_52_ = -3.0 / 4.0 * la[k][j][i] * strz[k];
_t_52_ += 2.0 * muz1;
_t_56_ = 3.0 * la[k][j][i] * strz[k];
_t_56_ += 2.0 * muz3;
_t_58_ = -3.0 / 4.0 * la[k][j][i] * strz[k];
_t_58_ += 2.0 * muz4;
_t_52_ -= 3.0 / 4.0 * la[k-2][j][i] * strz[k-2];
_t_54_ += la[k-2][j][i] * strz[k-2];
_t_56_ += la[k+2][j][i] * strz[k+2];
_t_58_ -= 3.0 / 4.0 * la[k+2][j][i] * strz[k+2];
_t_52_ += la[k-1][j][i] * strz[k-1];
_t_54_ += 3.0 * la[k-1][j][i] * strz[k-1];
_t_56_ += la[k-1][j][i] * strz[k-1];
_t_54_ += la[k+1][j][i] * strz[k+1];
_t_56_ += 3.0 * la[k+1][j][i] * strz[k+1];
_t_58_ += la[k+1][j][i] * strz[k+1];
_t_29_ = 3.0 * la[k][j][i] * stry[j];
_t_29_ += 2.0 * muy2;
_t_27_ = -3.0 / 4.0 * la[k][j][i] * stry[j];
_t_27_ += 2.0 * muy1;
_t_27_ -= 3.0 / 4.0 * la[k][j-2][i] * stry[j-2];
_t_29_ += la[k][j-2][i] * stry[j-2];
_t_27_ += la[k][j-1][i] * stry[j-1];
_t_29_ += 3.0 * la[k][j-1][i] * stry[j-1];
_t_26_ = _t_27_ * _t_28_;
_t_31_ = la[k][j-1][i] * stry[j-1];
_t_31_ += 3.0 * la[k][j][i] * stry[j];
_t_31_ += 2.0 * muy3;
_t_33_ = -3.0 / 4.0 * la[k][j][i] * stry[j];
_t_33_ += 2.0 * muy4;
_t_31_ += la[k][j+2][i] * stry[j+2];
_t_33_ -= 3.0 / 4.0 * la[k][j+2][i] * stry[j+2];
_t_29_ += la[k][j+1][i] * stry[j+1];
_t_26_ += _t_29_ * _t_30_;
_t_31_ += 3.0 * la[k][j+1][i] * stry[j+1];
_t_26_ += _t_31_ * _t_32_;
_t_33_ += la[k][j+1][i] * stry[j+1];
_t_26_ += _t_33_ * _t_34_;
r2 += 1.0 / 6.0 * stry[j] * _t_26_;
_t_41_ = mux1 * u_2[k][j][i-2];
_t_41_ -= mux1 * u_2[k][j][i];
_t_41_ -= mux2 * u_2[k][j][i];
_t_41_ -= mux3 * u_2[k][j][i];
_t_41_ -= mux4 * u_2[k][j][i];
_t_41_ += mux2 * u_2[k][j][i-1];
_t_41_ += mux3 * u_2[k][j][i+1];
_t_41_ += mux4 * u_2[k][j][i+2];
r3 = 1.0 / 6.0 * strx[i] * _t_41_;
_t_46_ = -muy1 * u_2[k][j][i];
_t_46_ -= muy2 * u_2[k][j][i];
_t_46_ -= muy3 * u_2[k][j][i];
_t_46_ -= muy4 * u_2[k][j][i];
_t_46_ += muy1 * u_2[k][j-2][i];
_t_46_ += muy2 * u_2[k][j-1][i];
_t_46_ += muy3 * u_2[k][j+1][i];
_t_46_ += muy4 * u_2[k][j+2][i];
r3 += 1.0 / 6.0 * stry[j] * _t_46_;
_t_53_ = -u_2[k][j][i];
_t_53_ += u_2[k-2][j][i];
_t_51_ = _t_52_ * _t_53_;
_t_55_ = -u_2[k][j][i];
_t_55_ += u_2[k-1][j][i];
_t_51_ += _t_54_ * _t_55_;
_t_57_ = -u_2[k][j][i];
_t_59_ = -u_2[k][j][i];
_t_57_ += u_2[k+1][j][i];
_t_51_ += _t_56_ * _t_57_;
_t_59_ += u_2[k+2][j][i];
_t_51_ += _t_58_ * _t_59_;
r3 += 1.0 / 6.0 * strz[k] * _t_51_;
_t_100_ = 1.0 / 144.0 * stry[j] * strz[k];
_t_74_ = 1.0 / 144.0 * stry[j] * strz[k];
_t_61_ = 1.0 / 144.0 * strx[i] * strz[k];
_t_87_ = 1.0 / 144.0 * strx[i] * strz[k];
_t_75_ = mu[k][j-2][i] * u_1[k-2][j-2][i];
_t_101_ = la[k-2][j][i] * u_1[k-2][j-2][i];
_t_75_ -= mu[k][j+2][i] * u_1[k-2][j+2][i];
_t_101_ -= la[k-2][j][i] * u_1[k-2][j+2][i];
_t_75_ -= mu[k][j-2][i] * u_1[k+2][j-2][i];
_t_101_ -= la[k+2][j][i] * u_1[k+2][j-2][i];
_t_75_ += mu[k][j+2][i] * u_1[k+2][j+2][i];
_t_101_ += la[k+2][j][i] * u_1[k+2][j+2][i];
_t_62_ = mu[k][j][i-2] * u_0[k-2][j][i-2];
_t_88_ = la[k-2][j][i] * u_0[k-2][j][i-2];
_t_62_ -= mu[k][j][i+2] * u_0[k-2][j][i+2];
_t_88_ -= la[k-2][j][i] * u_0[k-2][j][i+2];
_t_62_ -= mu[k][j][i-2] * u_0[k+2][j][i-2];
_t_88_ -= la[k+2][j][i] * u_0[k+2][j][i-2];
_t_62_ += mu[k][j][i+2] * u_0[k+2][j][i+2];
_t_88_ += la[k+2][j][i] * u_0[k+2][j][i+2];
_t_106_ = u_1[k-1][j-2][i];
_t_78_ = -u_1[k-1][j-2][i];
_t_106_ += 8.0 * -u_1[k-1][j-1][i];
_t_80_ = 8.0 * -u_1[k-1][j-1][i];
_t_78_ += u_1[k+1][j-2][i];
_t_75_ += mu[k][j-2][i] * 8.0 * _t_78_;
_t_109_ = u_1[k+1][j-2][i];
_t_80_ += 8.0 * u_1[k+1][j-1][i];
_t_109_ += 8.0 * -u_1[k+1][j-1][i];
_t_83_ = 8.0 * -u_1[k-1][j+1][i];
_t_106_ += 8.0 * u_1[k-1][j+1][i];
_t_83_ += 8.0 * u_1[k+1][j+1][i];
_t_109_ += 8.0 * u_1[k+1][j+1][i];
_t_86_ = -u_1[k-1][j+2][i];
_t_106_ -= u_1[k-1][j+2][i];
_t_101_ -= 8.0 * la[k-1][j][i] * _t_106_;
_t_86_ += u_1[k+1][j+2][i];
_t_75_ -= mu[k][j+2][i] * 8.0 * _t_86_;
_t_109_ -= u_1[k+1][j+2][i];
_t_101_ += 8.0 * la[k+1][j][i] * _t_109_;
_t_80_ += u_1[k-2][j-1][i];
_t_104_ = -u_1[k-2][j-1][i];
_t_83_ += u_1[k-2][j+1][i];
_t_104_ += u_1[k-2][j+1][i];
_t_101_ += la[k-2][j][i] * 8.0 * _t_104_;
_t_80_ -= u_1[k+2][j-1][i];
_t_75_ -= 8.0 * mu[k][j-1][i] * _t_80_;
_t_112_ = -u_1[k+2][j-1][i];
_t_83_ -= u_1[k+2][j+1][i];
_t_75_ += 8.0 * mu[k][j+1][i] * _t_83_;
_t_112_ += u_1[k+2][j+1][i];
_t_101_ -= la[k+2][j][i] * 8.0 * _t_112_;
_t_60_ = _t_74_ * _t_75_;
_t_60_ += _t_100_ * _t_101_;
_t_67_ = u_0[k-2][j][i-1];
_t_91_ = -u_0[k-2][j][i-1];
_t_91_ += u_0[k-2][j][i+1];
_t_88_ += la[k-2][j][i] * 8.0 * _t_91_;
_t_70_ = u_0[k-2][j][i+1];
_t_67_ += 8.0 * -u_0[k-1][j][i-1];
_t_93_ = 8.0 * -u_0[k-1][j][i-1];
_t_70_ += 8.0 * -u_0[k-1][j][i+1];
_t_93_ += 8.0 * u_0[k-1][j][i+1];
_t_67_ += 8.0 * u_0[k+1][j][i-1];
_t_96_ = 8.0 * -u_0[k+1][j][i-1];
_t_70_ += 8.0 * u_0[k+1][j][i+1];
_t_96_ += 8.0 * u_0[k+1][j][i+1];
_t_67_ -= u_0[k+2][j][i-1];
_t_62_ -= 8.0 * mu[k][j][i-1] * _t_67_;
_t_99_ = -u_0[k+2][j][i-1];
_t_70_ -= u_0[k+2][j][i+1];
_t_62_ += 8.0 * mu[k][j][i+1] * _t_70_;
_t_99_ += u_0[k+2][j][i+1];
_t_88_ -= la[k+2][j][i] * 8.0 * _t_99_;
_t_93_ += u_0[k-1][j][i-2];
_t_65_ = -u_0[k-1][j][i-2];
_t_65_ += u_0[k+1][j][i-2];
_t_62_ += mu[k][j][i-2] * 8.0 * _t_65_;
_t_96_ += u_0[k+1][j][i-2];
_t_93_ -= u_0[k-1][j][i+2];
_t_88_ -= 8.0 * la[k-1][j][i] * _t_93_;
_t_73_ = -u_0[k-1][j][i+2];
_t_73_ += u_0[k+1][j][i+2];
_t_62_ -= mu[k][j][i+2] * 8.0 * _t_73_;
_t_60_ += _t_61_ * _t_62_;
_t_96_ -= u_0[k+1][j][i+2];
_t_88_ += 8.0 * la[k+1][j][i] * _t_96_;
_t_60_ += _t_87_ * _t_88_;
r3 += _t_60_;
_t_127_ = 1.0 / 144.0 * strx[i] * strz[k];
_t_153_ = 1.0 / 144.0 * strx[i] * strz[k];
_t_140_ = 1.0 / 144.0 * strx[i] * stry[j];
_t_114_ = 1.0 / 144.0 * strx[i] * stry[j];
_t_115_ = la[k][j][i-2] * u_1[k][j-2][i-2];
_t_141_ = mu[k][j-2][i] * u_1[k][j-2][i-2];
_t_115_ -= la[k][j][i+2] * u_1[k][j-2][i+2];
_t_141_ -= mu[k][j-2][i] * u_1[k][j-2][i+2];
_t_115_ -= la[k][j][i-2] * u_1[k][j+2][i-2];
_t_141_ -= mu[k][j+2][i] * u_1[k][j+2][i-2];
_t_115_ += la[k][j][i+2] * u_1[k][j+2][i+2];
_t_141_ += mu[k][j+2][i] * u_1[k][j+2][i+2];
_t_128_ = la[k][j][i-2] * u_2[k-2][j][i-2];
_t_154_ = mu[k-2][j][i] * u_2[k-2][j][i-2];
_t_128_ -= la[k][j][i+2] * u_2[k-2][j][i+2];
_t_154_ -= mu[k-2][j][i] * u_2[k-2][j][i+2];
_t_128_ -= la[k][j][i-2] * u_2[k+2][j][i-2];
_t_154_ -= mu[k+2][j][i] * u_2[k+2][j][i-2];
_t_128_ += la[k][j][i+2] * u_2[k+2][j][i+2];
_t_154_ += mu[k+2][j][i] * u_2[k+2][j][i+2];
_t_120_ = u_1[k][j-2][i-1];
_t_144_ = -u_1[k][j-2][i-1];
_t_144_ += u_1[k][j-2][i+1];
_t_141_ += mu[k][j-2][i] * 8.0 * _t_144_;
_t_123_ = u_1[k][j-2][i+1];
_t_120_ += 8.0 * -u_1[k][j-1][i-1];
_t_146_ = 8.0 * -u_1[k][j-1][i-1];
_t_123_ += 8.0 * -u_1[k][j-1][i+1];
_t_146_ += 8.0 * u_1[k][j-1][i+1];
_t_120_ += 8.0 * u_1[k][j+1][i-1];
_t_149_ = 8.0 * -u_1[k][j+1][i-1];
_t_123_ += 8.0 * u_1[k][j+1][i+1];
_t_149_ += 8.0 * u_1[k][j+1][i+1];
_t_120_ -= u_1[k][j+2][i-1];
_t_115_ -= 8.0 * la[k][j][i-1] * _t_120_;
_t_152_ = -u_1[k][j+2][i-1];
_t_123_ -= u_1[k][j+2][i+1];
_t_115_ += 8.0 * la[k][j][i+1] * _t_123_;
_t_152_ += u_1[k][j+2][i+1];
_t_141_ -= mu[k][j+2][i] * 8.0 * _t_152_;
_t_118_ = -u_1[k][j-1][i-2];
_t_146_ += u_1[k][j-1][i-2];
_t_118_ += u_1[k][j+1][i-2];
_t_115_ += la[k][j][i-2] * 8.0 * _t_118_;
_t_149_ += u_1[k][j+1][i-2];
_t_126_ = -u_1[k][j-1][i+2];
_t_146_ -= u_1[k][j-1][i+2];
_t_141_ -= 8.0 * mu[k][j-1][i] * _t_146_;
_t_126_ += u_1[k][j+1][i+2];
_t_115_ -= la[k][j][i+2] * 8.0 * _t_126_;
_t_149_ -= u_1[k][j+1][i+2];
_t_141_ += 8.0 * mu[k][j+1][i] * _t_149_;
_t_113_ = _t_114_ * _t_115_;
_t_113_ += _t_140_ * _t_141_;
_t_133_ = u_2[k-2][j][i-1];
_t_157_ = -u_2[k-2][j][i-1];
_t_157_ += u_2[k-2][j][i+1];
_t_154_ += mu[k-2][j][i] * 8.0 * _t_157_;
_t_136_ = u_2[k-2][j][i+1];
_t_133_ += 8.0 * -u_2[k-1][j][i-1];
_t_159_ = 8.0 * -u_2[k-1][j][i-1];
_t_136_ += 8.0 * -u_2[k-1][j][i+1];
_t_159_ += 8.0 * u_2[k-1][j][i+1];
_t_133_ += 8.0 * u_2[k+1][j][i-1];
_t_162_ = 8.0 * -u_2[k+1][j][i-1];
_t_136_ += 8.0 * u_2[k+1][j][i+1];
_t_162_ += 8.0 * u_2[k+1][j][i+1];
_t_133_ -= u_2[k+2][j][i-1];
_t_128_ -= 8.0 * la[k][j][i-1] * _t_133_;
_t_165_ = -u_2[k+2][j][i-1];
_t_136_ -= u_2[k+2][j][i+1];
_t_128_ += 8.0 * la[k][j][i+1] * _t_136_;
_t_165_ += u_2[k+2][j][i+1];
_t_154_ -= mu[k+2][j][i] * 8.0 * _t_165_;
_t_131_ = -u_2[k-1][j][i-2];
_t_159_ += u_2[k-1][j][i-2];
_t_131_ += u_2[k+1][j][i-2];
_t_128_ += la[k][j][i-2] * 8.0 * _t_131_;
_t_162_ += u_2[k+1][j][i-2];
_t_139_ = -u_2[k-1][j][i+2];
_t_159_ -= u_2[k-1][j][i+2];
_t_154_ -= 8.0 * mu[k-1][j][i] * _t_159_;
_t_139_ += u_2[k+1][j][i+2];
_t_128_ -= la[k][j][i+2] * 8.0 * _t_139_;
_t_113_ += _t_127_ * _t_128_;
_t_162_ -= u_2[k+1][j][i+2];
_t_154_ += 8.0 * mu[k+1][j][i] * _t_162_;
_t_113_ += _t_153_ * _t_154_;
r1 += _t_113_;
_t_167_ = 1.0 / 144.0 * strx[i] * stry[j];
_t_180_ = 1.0 / 144.0 * strx[i] * stry[j];
_t_206_ = 1.0 / 144.0 * stry[j] * strz[k];
_t_193_ = 1.0 / 144.0 * stry[j] * strz[k];
_t_168_ = mu[k][j][i-2] * u_0[k][j-2][i-2];
_t_181_ = la[k][j-2][i] * u_0[k][j-2][i-2];
_t_168_ -= mu[k][j][i+2] * u_0[k][j-2][i+2];
_t_181_ -= la[k][j-2][i] * u_0[k][j-2][i+2];
_t_168_ -= mu[k][j][i-2] * u_0[k][j+2][i-2];
_t_181_ -= la[k][j+2][i] * u_0[k][j+2][i-2];
_t_168_ += mu[k][j][i+2] * u_0[k][j+2][i+2];
_t_181_ += la[k][j+2][i] * u_0[k][j+2][i+2];
_t_194_ = la[k][j-2][i] * u_2[k-2][j-2][i];
_t_207_ = mu[k-2][j][i] * u_2[k-2][j-2][i];
_t_194_ -= la[k][j+2][i] * u_2[k-2][j+2][i];
_t_207_ -= mu[k-2][j][i] * u_2[k-2][j+2][i];
_t_194_ -= la[k][j-2][i] * u_2[k+2][j-2][i];
_t_207_ -= mu[k+2][j][i] * u_2[k+2][j-2][i];
_t_194_ += la[k][j+2][i] * u_2[k+2][j+2][i];
_t_207_ += mu[k+2][j][i] * u_2[k+2][j+2][i];
_t_173_ = u_0[k][j-2][i-1];
_t_184_ = -u_0[k][j-2][i-1];
_t_184_ += u_0[k][j-2][i+1];
_t_181_ += la[k][j-2][i] * 8.0 * _t_184_;
_t_176_ = u_0[k][j-2][i+1];
_t_173_ += 8.0 * -u_0[k][j-1][i-1];
_t_186_ = 8.0 * -u_0[k][j-1][i-1];
_t_176_ += 8.0 * -u_0[k][j-1][i+1];
_t_186_ += 8.0 * u_0[k][j-1][i+1];
_t_173_ += 8.0 * u_0[k][j+1][i-1];
_t_189_ = 8.0 * -u_0[k][j+1][i-1];
_t_176_ += 8.0 * u_0[k][j+1][i+1];
_t_189_ += 8.0 * u_0[k][j+1][i+1];
_t_173_ -= u_0[k][j+2][i-1];
_t_168_ -= 8.0 * mu[k][j][i-1] * _t_173_;
_t_192_ = -u_0[k][j+2][i-1];
_t_176_ -= u_0[k][j+2][i+1];
_t_168_ += 8.0 * mu[k][j][i+1] * _t_176_;
_t_192_ += u_0[k][j+2][i+1];
_t_181_ -= la[k][j+2][i] * 8.0 * _t_192_;
_t_171_ = -u_0[k][j-1][i-2];
_t_186_ += u_0[k][j-1][i-2];
_t_171_ += u_0[k][j+1][i-2];
_t_168_ += mu[k][j][i-2] * 8.0 * _t_171_;
_t_189_ += u_0[k][j+1][i-2];
_t_179_ = -u_0[k][j-1][i+2];
_t_186_ -= u_0[k][j-1][i+2];
_t_181_ -= 8.0 * la[k][j-1][i] * _t_186_;
_t_179_ += u_0[k][j+1][i+2];
_t_168_ -= mu[k][j][i+2] * 8.0 * _t_179_;
_t_189_ -= u_0[k][j+1][i+2];
_t_181_ += 8.0 * la[k][j+1][i] * _t_189_;
_t_166_ = _t_167_ * _t_168_;
_t_166_ += _t_180_ * _t_181_;
_t_199_ = u_2[k-2][j-1][i];
_t_210_ = -u_2[k-2][j-1][i];
_t_210_ += u_2[k-2][j+1][i];
_t_207_ += mu[k-2][j][i] * 8.0 * _t_210_;
_t_202_ = u_2[k-2][j+1][i];
_t_199_ += 8.0 * -u_2[k-1][j-1][i];
_t_212_ = 8.0 * -u_2[k-1][j-1][i];
_t_202_ += 8.0 * -u_2[k-1][j+1][i];
_t_212_ += 8.0 * u_2[k-1][j+1][i];
_t_199_ += 8.0 * u_2[k+1][j-1][i];
_t_215_ = 8.0 * -u_2[k+1][j-1][i];
_t_202_ += 8.0 * u_2[k+1][j+1][i];
_t_215_ += 8.0 * u_2[k+1][j+1][i];
_t_199_ -= u_2[k+2][j-1][i];
_t_194_ -= 8.0 * la[k][j-1][i] * _t_199_;
_t_218_ = -u_2[k+2][j-1][i];
_t_202_ -= u_2[k+2][j+1][i];
_t_194_ += 8.0 * la[k][j+1][i] * _t_202_;
_t_218_ += u_2[k+2][j+1][i];
_t_207_ -= mu[k+2][j][i] * 8.0 * _t_218_;
_t_197_ = -u_2[k-1][j-2][i];
_t_212_ += u_2[k-1][j-2][i];
_t_197_ += u_2[k+1][j-2][i];
_t_194_ += la[k][j-2][i] * 8.0 * _t_197_;
_t_215_ += u_2[k+1][j-2][i];
_t_205_ = -u_2[k-1][j+2][i];
_t_212_ -= u_2[k-1][j+2][i];
_t_207_ -= 8.0 * mu[k-1][j][i] * _t_212_;
_t_205_ += u_2[k+1][j+2][i];
_t_194_ -= la[k][j+2][i] * 8.0 * _t_205_;
_t_166_ += _t_193_ * _t_194_;
_t_215_ -= u_2[k+1][j+2][i];
_t_207_ += 8.0 * mu[k+1][j][i] * _t_215_;
_t_166_ += _t_206_ * _t_207_;
r2 += _t_166_;
uacc_0kc0jc0ic0 = a1 * uacc_0[k][j][i];
uacc_0kc0jc0ic0 += cof * r1;
uacc_0[k][j][i] = uacc_0kc0jc0ic0;
uacc_1kc0jc0ic0 = a1 * uacc_1[k][j][i];
uacc_1kc0jc0ic0 += cof * r2;
uacc_1[k][j][i] = uacc_1kc0jc0ic0;
uacc_2kc0jc0ic0 = a1 * uacc_2[k][j][i];
uacc_2kc0jc0ic0 += cof * r3;
uacc_2[k][j][i] = uacc_2kc0jc0ic0;
	} 
}

extern "C" void host_code (double *h_uacc_0, double *h_uacc_1, double *h_uacc_2, double *h_u_0, double *h_u_1, double *h_u_2, double *h_mu, double *h_la, double *h_strx, double *h_stry, double *h_strz, int N) {
	double *uacc_0;
	hipMalloc (&uacc_0, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_0\n");
	hipMemcpy (uacc_0, h_uacc_0, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *uacc_1;
	hipMalloc (&uacc_1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_1\n");
	hipMemcpy (uacc_1, h_uacc_1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *uacc_2;
	hipMalloc (&uacc_2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_2\n");
	hipMemcpy (uacc_2, h_uacc_2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_0;
	hipMalloc (&u_0, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_0\n");
	hipMemcpy (u_0, h_u_0, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_1;
	hipMalloc (&u_1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_1\n");
	hipMemcpy (u_1, h_u_1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_2;
	hipMalloc (&u_2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_2\n");
	hipMemcpy (u_2, h_u_2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *mu;
	hipMalloc (&mu, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for mu\n");
	hipMemcpy (mu, h_mu, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *la;
	hipMalloc (&la, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for la\n");
	hipMemcpy (la, h_la, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *strx;
	hipMalloc (&strx, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(double)*N, hipMemcpyHostToDevice);
	double *stry;
	hipMalloc (&stry, sizeof(double)*N);
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(double)*N, hipMemcpyHostToDevice);
	double *strz;
	hipMalloc (&strz, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strz\n");
	hipMemcpy (strz, h_strz, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 2, 2);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, blockconfig.y), ceil(N, blockconfig.z));

	sw4 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);

	hipMemcpy (h_uacc_0, uacc_0, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_uacc_1, uacc_1, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_uacc_2, uacc_2, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);

	hipFree (uacc_0); 
	hipFree (uacc_1);
	hipFree (uacc_2);
	hipFree (u_0);
	hipFree (u_1);
	hipFree (u_2);
	hipFree (mu);
	hipFree (la);
	hipFree (strx);
	hipFree (stry);
	hipFree (strz);
}
