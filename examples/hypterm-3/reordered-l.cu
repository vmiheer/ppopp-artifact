#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void hypterm_0 (double * __restrict__ flux_in_0, double * __restrict__ flux_in_1, double * __restrict__ flux_in_2, double * __restrict__ flux_in_3, double * __restrict__ cons_in_1, double * __restrict__ cons_in_2, double * __restrict__ cons_in_3, double * __restrict__ q_in_1, double * __restrict__ q_in_2, double * __restrict__ q_in_3, double * __restrict__ q_in_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	int blockdim_k= (int)(blockDim.z);
	int k0 = (int)(blockIdx.z)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.z);

	double (*flux_0)[308][308] = (double (*)[308][308])flux_in_0;
	double (*flux_1)[308][308] = (double (*)[308][308])flux_in_1;
	double (*flux_2)[308][308] = (double (*)[308][308])flux_in_2;
	double (*flux_3)[308][308] = (double (*)[308][308])flux_in_3;
	double (*cons_1)[308][308] = (double (*)[308][308])cons_in_1;
	double (*cons_2)[308][308] = (double (*)[308][308])cons_in_2;
	double (*cons_3)[308][308] = (double (*)[308][308])cons_in_3;
    double (*q_1)[308][308] = (double (*)[308][308])q_in_1;
    double (*q_2)[308][308] = (double (*)[308][308])q_in_2;
    double (*q_3)[308][308] = (double (*)[308][308])q_in_3;
    double (*q_4)[308][308] = (double (*)[308][308])q_in_4;

	if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
double _t_1_;
double _t_0_;
double _t_2_;
double _t_3_;
double _t_4_;
double flux_0kc0jc0ic0;
double _t_7_;
double _t_6_;
double _t_8_;
double _t_9_;
double _t_10_;
double _t_5_;
double _t_12_;
double _t_11_;
double _t_13_;
double _t_14_;
double _t_15_;
double flux_1kc0jc0ic0;
double _t_18_;
double _t_17_;
double _t_19_;
double _t_20_;
double _t_21_;
double _t_16_;
double _t_23_;
double _t_22_;
double _t_24_;
double _t_25_;
double _t_26_;
double flux_2kc0jc0ic0;
double _t_29_;
double _t_28_;
double _t_30_;
double _t_31_;
double _t_32_;
double _t_27_;
double _t_34_;
double _t_33_;
double _t_35_;
double _t_36_;
double _t_37_;
double flux_3kc0jc0ic0;
double _t_40_;
double _t_39_;
double _t_41_;
double _t_42_;
double _t_43_;
double _t_38_;

_t_1_ = cons_1[k][j][i+1];
_t_1_ -= cons_1[k][j][i-1];
_t_0_ = 0.8 * _t_1_;
_t_2_ = cons_1[k][j][i+2];
_t_2_ -= cons_1[k][j][i-2];
_t_0_ -= 0.2 * _t_2_;
_t_3_ = cons_1[k][j][i+3];
_t_3_ -= cons_1[k][j][i-3];
_t_0_ += 0.038 * _t_3_;
_t_4_ = cons_1[k][j][i+4];
_t_4_ -= cons_1[k][j][i-4];
_t_0_ -= 0.0035 * _t_4_;
flux_0kc0jc0ic0 = _t_0_ * dxinv0;
_t_7_ = cons_2[k][j+1][i];
_t_7_ -= cons_2[k][j-1][i];
_t_6_ = 0.8 * _t_7_;
_t_8_ = cons_2[k][j+2][i];
_t_8_ -= cons_2[k][j-2][i];
_t_6_ -= 0.2 * _t_8_;
_t_9_ = cons_2[k][j+3][i];
_t_9_ -= cons_2[k][j-3][i];
_t_6_ += 0.038 * _t_9_;
_t_10_ = cons_2[k][j+4][i];
_t_10_ -= cons_2[k][j-4][i];
_t_6_ -= 0.0035 * _t_10_;
_t_5_ = _t_6_ * dxinv1;
flux_0kc0jc0ic0 -= _t_5_;
flux_0[k][j][i] = flux_0kc0jc0ic0;
_t_12_ = cons_1[k][j][i+1] * q_1[k][j][i+1];
_t_12_ -= cons_1[k][j][i-1] * q_1[k][j][i-1];
_t_12_ += q_4[k][j][i+1];
_t_12_ -= q_4[k][j][i-1];
_t_11_ = 0.8 * _t_12_;
_t_13_ = cons_1[k][j][i+2] * q_1[k][j][i+2];
_t_13_ -= cons_1[k][j][i-2] * q_1[k][j][i-2];
_t_13_ += q_4[k][j][i+2];
_t_13_ -= q_4[k][j][i-2];
_t_11_ -= 0.2 * _t_13_;
_t_14_ = cons_1[k][j][i+3] * q_1[k][j][i+3];
_t_14_ -= cons_1[k][j][i-3] * q_1[k][j][i-3];
_t_14_ += q_4[k][j][i+3];
_t_14_ -= q_4[k][j][i-3];
_t_11_ += 0.038 * _t_14_;
_t_15_ = cons_1[k][j][i+4] * q_1[k][j][i+4];
_t_15_ -= cons_1[k][j][i-4] * q_1[k][j][i-4];
_t_15_ += q_4[k][j][i+4];
_t_15_ -= q_4[k][j][i-4];
_t_11_ -= 0.0035 * _t_15_;
flux_1kc0jc0ic0 = _t_11_ * dxinv0;
_t_18_ = cons_1[k][j+1][i] * q_2[k][j+1][i];
_t_18_ -= cons_1[k][j-1][i] * q_2[k][j-1][i];
_t_17_ = 0.8 * _t_18_;
_t_19_ = cons_1[k][j+2][i] * q_2[k][j+2][i];
_t_19_ -= cons_1[k][j-2][i] * q_2[k][j-2][i];
_t_17_ -= 0.2 * _t_19_;
_t_20_ = cons_1[k][j+3][i] * q_2[k][j+3][i];
_t_20_ -= cons_1[k][j-3][i] * q_2[k][j-3][i];
_t_17_ += 0.038 * _t_20_;
_t_21_ = cons_1[k][j+4][i] * q_2[k][j+4][i];
_t_21_ -= cons_1[k][j-4][i] * q_2[k][j-4][i];
_t_17_ -= 0.0035 * _t_21_;
_t_16_ = _t_17_ * dxinv1;
flux_1kc0jc0ic0 -= _t_16_;
flux_1[k][j][i] = flux_1kc0jc0ic0;
_t_23_ = cons_2[k][j][i+1] * q_1[k][j][i+1];
_t_23_ -= cons_2[k][j][i-1] * q_1[k][j][i-1];
_t_22_ = 0.8 * _t_23_;
_t_24_ = cons_2[k][j][i+2] * q_1[k][j][i+2];
_t_24_ -= cons_2[k][j][i-2] * q_1[k][j][i-2];
_t_22_ -= 0.2 * _t_24_;
_t_25_ = cons_2[k][j][i+3] * q_1[k][j][i+3];
_t_25_ -= cons_2[k][j][i-3] * q_1[k][j][i-3];
_t_22_ += 0.038 * _t_25_;
_t_26_ = cons_2[k][j][i+4] * q_1[k][j][i+4];
_t_26_ -= cons_2[k][j][i-4] * q_1[k][j][i-4];
_t_22_ -= 0.0035 * _t_26_;
flux_2kc0jc0ic0 = _t_22_ * dxinv0;
_t_29_ = cons_2[k][j+1][i] * q_2[k][j+1][i];
_t_29_ -= cons_2[k][j-1][i] * q_2[k][j-1][i];
_t_29_ += q_4[k][j+1][i];
_t_29_ -= q_4[k][j-1][i];
_t_28_ = 0.8 * _t_29_;
_t_30_ = cons_2[k][j+2][i] * q_2[k][j+2][i];
_t_30_ -= cons_2[k][j-2][i] * q_2[k][j-2][i];
_t_30_ += q_4[k][j+2][i];
_t_30_ -= q_4[k][j-2][i];
_t_28_ -= 0.2 * _t_30_;
_t_31_ = cons_2[k][j+3][i] * q_2[k][j+3][i];
_t_31_ -= cons_2[k][j-3][i] * q_2[k][j-3][i];
_t_31_ += q_4[k][j+3][i];
_t_31_ -= q_4[k][j-3][i];
_t_28_ += 0.038 * _t_31_;
_t_32_ = cons_2[k][j+4][i] * q_2[k][j+4][i];
_t_32_ -= cons_2[k][j-4][i] * q_2[k][j-4][i];
_t_32_ += q_4[k][j+4][i];
_t_32_ -= q_4[k][j-4][i];
_t_28_ -= 0.0035 * _t_32_;
_t_27_ = _t_28_ * dxinv1;
flux_2kc0jc0ic0 -= _t_27_;
flux_2[k][j][i] = flux_2kc0jc0ic0;
_t_34_ = cons_3[k][j][i+1] * q_1[k][j][i+1];
_t_34_ -= cons_3[k][j][i-1] * q_1[k][j][i-1];
_t_33_ = 0.8 * _t_34_;
_t_35_ = cons_3[k][j][i+2] * q_1[k][j][i+2];
_t_35_ -= cons_3[k][j][i-2] * q_1[k][j][i-2];
_t_33_ -= 0.2 * _t_35_;
_t_36_ = cons_3[k][j][i+3] * q_1[k][j][i+3];
_t_36_ -= cons_3[k][j][i-3] * q_1[k][j][i-3];
_t_33_ += 0.038 * _t_36_;
_t_37_ = cons_3[k][j][i+4] * q_1[k][j][i+4];
_t_37_ -= cons_3[k][j][i-4] * q_1[k][j][i-4];
_t_33_ -= 0.0035 * _t_37_;
flux_3kc0jc0ic0 = _t_33_ * dxinv0;
_t_40_ = cons_3[k][j+1][i] * q_2[k][j+1][i];
_t_40_ -= cons_3[k][j-1][i] * q_2[k][j-1][i];
_t_39_ = 0.8 * _t_40_;
_t_41_ = cons_3[k][j+2][i] * q_2[k][j+2][i];
_t_41_ -= cons_3[k][j-2][i] * q_2[k][j-2][i];
_t_39_ -= 0.2 * _t_41_;
_t_42_ = cons_3[k][j+3][i] * q_2[k][j+3][i];
_t_42_ -= cons_3[k][j-3][i] * q_2[k][j-3][i];
_t_39_ += 0.038 * _t_42_;
_t_43_ = cons_3[k][j+4][i] * q_2[k][j+4][i];
_t_43_ -= cons_3[k][j-4][i] * q_2[k][j-4][i];
_t_39_ -= 0.0035 * _t_43_;
_t_38_ = _t_39_ * dxinv1;
flux_3kc0jc0ic0 -= _t_38_;
flux_3[k][j][i] = flux_3kc0jc0ic0;
	} 
}

__global__ void hypterm_1 (double * __restrict__ flux_in_0, double * __restrict__ flux_in_1, double * __restrict__ flux_in_2, double * __restrict__ flux_in_3, double * __restrict__ cons_in_1, double * __restrict__ cons_in_2, double * __restrict__ cons_in_3, double * __restrict__ q_in_1, double * __restrict__ q_in_2, double * __restrict__ q_in_3, double * __restrict__ q_in_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
    //Determing the block's indices
    int blockdim_i= (int)(blockDim.x);
    int i0 = (int)(blockIdx.x)*(blockdim_i);
    int i = max (i0, 0) + (int)(threadIdx.x);
    int blockdim_j= (int)(blockDim.y);
    int j0 = (int)(blockIdx.y)*(blockdim_j);
    int j = max (j0, 0) + (int)(threadIdx.y);
    int blockdim_k= (int)(blockDim.z);
    int k0 = (int)(blockIdx.z)*(4*blockdim_k);
    int k = max (k0, 0) + (int)(4*threadIdx.z);

    double (*flux_0)[308][308] = (double (*)[308][308])flux_in_0;
    double (*flux_1)[308][308] = (double (*)[308][308])flux_in_1;
    double (*flux_2)[308][308] = (double (*)[308][308])flux_in_2;
    double (*flux_3)[308][308] = (double (*)[308][308])flux_in_3;
    double (*cons_1)[308][308] = (double (*)[308][308])cons_in_1;
    double (*cons_2)[308][308] = (double (*)[308][308])cons_in_2;
    double (*cons_3)[308][308] = (double (*)[308][308])cons_in_3;
    double (*q_1)[308][308] = (double (*)[308][308])q_in_1;
    double (*q_2)[308][308] = (double (*)[308][308])q_in_2;
    double (*q_3)[308][308] = (double (*)[308][308])q_in_3;
    double (*q_4)[308][308] = (double (*)[308][308])q_in_4;
	double flux0_a, flux1_a, flux2_a, flux3_a;
	double flux0_b, flux1_b, flux2_b, flux3_b;
	double flux0_c, flux1_c, flux2_c, flux3_c;
	double flux0_d, flux1_d, flux2_d, flux3_d;

	if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
double flux0_a;
double flux_0kc0jc0ic0;
double _t_1_;
double _t_0_;
double _t_2_;
double _t_3_;
double _t_4_;
double flux0_b;
double flux_0kp1jc0ic0;
double _t_6_;
double _t_5_;
double _t_7_;
double _t_8_;
double _t_9_;
double flux0_c;
double flux_0kp2jc0ic0;
double _t_11_;
double _t_10_;
double _t_12_;
double _t_13_;
double _t_14_;
double flux0_d;
double flux_0kp3jc0ic0;
double _t_16_;
double _t_15_;
double _t_17_;
double _t_18_;
double _t_19_;
double flux1_a;
double flux_1kc0jc0ic0;
double _t_21_;
double _t_20_;
double _t_22_;
double _t_23_;
double _t_24_;
double flux1_b;
double flux_1kp1jc0ic0;
double _t_26_;
double _t_25_;
double _t_27_;
double _t_28_;
double _t_29_;
double flux1_c;
double flux_1kp2jc0ic0;
double _t_31_;
double _t_30_;
double _t_32_;
double _t_33_;
double _t_34_;
double flux1_d;
double flux_1kp3jc0ic0;
double _t_36_;
double _t_35_;
double _t_37_;
double _t_38_;
double _t_39_;
double flux2_a;
double flux_2kc0jc0ic0;
double _t_41_;
double _t_40_;
double _t_42_;
double _t_43_;
double _t_44_;
double flux2_b;
double flux_2kp1jc0ic0;
double _t_46_;
double _t_45_;
double _t_47_;
double _t_48_;
double _t_49_;
double flux2_c;
double flux_2kp2jc0ic0;
double _t_51_;
double _t_50_;
double _t_52_;
double _t_53_;
double _t_54_;
double flux2_d;
double flux_2kp3jc0ic0;
double _t_56_;
double _t_55_;
double _t_57_;
double _t_58_;
double _t_59_;
double flux3_a;
double flux_3kc0jc0ic0;
double _t_61_;
double _t_60_;
double _t_62_;
double _t_63_;
double _t_64_;
double flux3_b;
double flux_3kp1jc0ic0;
double _t_66_;
double _t_65_;
double _t_67_;
double _t_68_;
double _t_69_;
double flux3_c;
double flux_3kp2jc0ic0;
double _t_71_;
double _t_70_;
double _t_72_;
double _t_73_;
double _t_74_;
double flux3_d;
double flux_3kp3jc0ic0;
double _t_76_;
double _t_75_;
double _t_77_;
double _t_78_;
double _t_79_;

flux0_a = flux_0[k][j][i];
flux_0kc0jc0ic0 = flux0_a;
_t_1_ = cons_3[k+1][j][i];
_t_1_ -= cons_3[k-1][j][i];
_t_0_ = 0.8 * _t_1_;
_t_2_ = cons_3[k+2][j][i];
_t_2_ -= cons_3[k-2][j][i];
_t_0_ -= 0.2 * _t_2_;
_t_3_ = cons_3[k+3][j][i];
_t_3_ -= cons_3[k-3][j][i];
_t_0_ += 0.038 * _t_3_;
_t_4_ = cons_3[k+4][j][i];
_t_4_ -= cons_3[k-4][j][i];
_t_0_ -= 0.0035 * _t_4_;
flux_0kc0jc0ic0 -= _t_0_ * dxinv2;
flux_0[k][j][i] = flux_0kc0jc0ic0;
flux0_b = flux_0[k+1][j][i];
flux_0kp1jc0ic0 = flux0_b;
_t_6_ = cons_3[k+2][j][i];
_t_6_ -= cons_3[k][j][i];
_t_5_ = 0.8 * _t_6_;
_t_7_ = cons_3[k+3][j][i];
_t_7_ -= cons_3[k-1][j][i];
_t_5_ -= 0.2 * _t_7_;
_t_8_ = cons_3[k+4][j][i];
_t_8_ -= cons_3[k-2][j][i];
_t_5_ += 0.038 * _t_8_;
_t_9_ = cons_3[k+5][j][i];
_t_9_ -= cons_3[k-3][j][i];
_t_5_ -= 0.0035 * _t_9_;
flux_0kp1jc0ic0 -= _t_5_ * dxinv2;
flux_0[k+1][j][i] = flux_0kp1jc0ic0;
flux0_c = flux_0[k+2][j][i];
flux_0kp2jc0ic0 = flux0_c;
_t_11_ = cons_3[k+3][j][i];
_t_11_ -= cons_3[k+1][j][i];
_t_10_ = 0.8 * _t_11_;
_t_12_ = cons_3[k+4][j][i];
_t_12_ -= cons_3[k][j][i];
_t_10_ -= 0.2 * _t_12_;
_t_13_ = cons_3[k+5][j][i];
_t_13_ -= cons_3[k-1][j][i];
_t_10_ += 0.038 * _t_13_;
_t_14_ = cons_3[k+6][j][i];
_t_14_ -= cons_3[k-2][j][i];
_t_10_ -= 0.0035 * _t_14_;
flux_0kp2jc0ic0 -= _t_10_ * dxinv2;
flux_0[k+2][j][i] = flux_0kp2jc0ic0;
flux0_d = flux_0[k+3][j][i];
flux_0kp3jc0ic0 = flux0_d;
_t_16_ = cons_3[k+4][j][i];
_t_16_ -= cons_3[k+2][j][i];
_t_15_ = 0.8 * _t_16_;
_t_17_ = cons_3[k+5][j][i];
_t_17_ -= cons_3[k+1][j][i];
_t_15_ -= 0.2 * _t_17_;
_t_18_ = cons_3[k+6][j][i];
_t_18_ -= cons_3[k][j][i];
_t_15_ += 0.038 * _t_18_;
_t_19_ = cons_3[k+7][j][i];
_t_19_ -= cons_3[k-1][j][i];
_t_15_ -= 0.0035 * _t_19_;
flux_0kp3jc0ic0 -= _t_15_ * dxinv2;
flux_0[k+3][j][i] = flux_0kp3jc0ic0;
flux1_a = flux_1[k][j][i];
flux_1kc0jc0ic0 = flux1_a;
_t_21_ = cons_1[k+1][j][i] * q_3[k+1][j][i];
_t_21_ -= cons_1[k-1][j][i] * q_3[k-1][j][i];
_t_20_ = 0.8 * _t_21_;
_t_22_ = cons_1[k+2][j][i] * q_3[k+2][j][i];
_t_22_ -= cons_1[k-2][j][i] * q_3[k-2][j][i];
_t_20_ -= 0.2 * _t_22_;
_t_23_ = cons_1[k+3][j][i] * q_3[k+3][j][i];
_t_23_ -= cons_1[k-3][j][i] * q_3[k-3][j][i];
_t_20_ += 0.038 * _t_23_;
_t_24_ = cons_1[k+4][j][i] * q_3[k+4][j][i];
_t_24_ -= cons_1[k-4][j][i] * q_3[k-4][j][i];
_t_20_ -= 0.0035 * _t_24_;
flux_1kc0jc0ic0 -= _t_20_ * dxinv2;
flux_1[k][j][i] = flux_1kc0jc0ic0;
flux1_b = flux_1[k+1][j][i];
flux_1kp1jc0ic0 = flux1_b;
_t_26_ = cons_1[k+2][j][i] * q_3[k+2][j][i];
_t_26_ -= cons_1[k][j][i] * q_3[k][j][i];
_t_25_ = 0.8 * _t_26_;
_t_27_ = cons_1[k+3][j][i] * q_3[k+3][j][i];
_t_27_ -= cons_1[k-1][j][i] * q_3[k-1][j][i];
_t_25_ -= 0.2 * _t_27_;
_t_28_ = cons_1[k+4][j][i] * q_3[k+4][j][i];
_t_28_ -= cons_1[k-2][j][i] * q_3[k-2][j][i];
_t_25_ += 0.038 * _t_28_;
_t_29_ = cons_1[k+5][j][i] * q_3[k+5][j][i];
_t_29_ -= cons_1[k-3][j][i] * q_3[k-3][j][i];
_t_25_ -= 0.0035 * _t_29_;
flux_1kp1jc0ic0 -= _t_25_ * dxinv2;
flux_1[k+1][j][i] = flux_1kp1jc0ic0;
flux1_c = flux_1[k+2][j][i];
flux_1kp2jc0ic0 = flux1_c;
_t_31_ = cons_1[k+3][j][i] * q_3[k+3][j][i];
_t_31_ -= cons_1[k+1][j][i] * q_3[k+1][j][i];
_t_30_ = 0.8 * _t_31_;
_t_32_ = cons_1[k+4][j][i] * q_3[k+4][j][i];
_t_32_ -= cons_1[k][j][i] * q_3[k][j][i];
_t_30_ -= 0.2 * _t_32_;
_t_33_ = cons_1[k+5][j][i] * q_3[k+5][j][i];
_t_33_ -= cons_1[k-1][j][i] * q_3[k-1][j][i];
_t_30_ += 0.038 * _t_33_;
_t_34_ = cons_1[k+6][j][i] * q_3[k+6][j][i];
_t_34_ -= cons_1[k-2][j][i] * q_3[k-2][j][i];
_t_30_ -= 0.0035 * _t_34_;
flux_1kp2jc0ic0 -= _t_30_ * dxinv2;
flux_1[k+2][j][i] = flux_1kp2jc0ic0;
flux1_d = flux_1[k+3][j][i];
flux_1kp3jc0ic0 = flux1_d;
_t_36_ = cons_1[k+4][j][i] * q_3[k+4][j][i];
_t_36_ -= cons_1[k+2][j][i] * q_3[k+2][j][i];
_t_35_ = 0.8 * _t_36_;
_t_37_ = cons_1[k+5][j][i] * q_3[k+5][j][i];
_t_37_ -= cons_1[k+1][j][i] * q_3[k+1][j][i];
_t_35_ -= 0.2 * _t_37_;
_t_38_ = cons_1[k+6][j][i] * q_3[k+6][j][i];
_t_38_ -= cons_1[k][j][i] * q_3[k][j][i];
_t_35_ += 0.038 * _t_38_;
_t_39_ = cons_1[k+7][j][i] * q_3[k+7][j][i];
_t_39_ -= cons_1[k-1][j][i] * q_3[k-1][j][i];
_t_35_ -= 0.0035 * _t_39_;
flux_1kp3jc0ic0 -= _t_35_ * dxinv2;
flux_1[k+3][j][i] = flux_1kp3jc0ic0;
flux2_a = flux_2[k][j][i];
flux_2kc0jc0ic0 = flux2_a;
_t_41_ = cons_2[k+1][j][i] * q_3[k+1][j][i];
_t_41_ -= cons_2[k-1][j][i] * q_3[k-1][j][i];
_t_40_ = 0.8 * _t_41_;
_t_42_ = cons_2[k+2][j][i] * q_3[k+2][j][i];
_t_42_ -= cons_2[k-2][j][i] * q_3[k-2][j][i];
_t_40_ -= 0.2 * _t_42_;
_t_43_ = cons_2[k+3][j][i] * q_3[k+3][j][i];
_t_43_ -= cons_2[k-3][j][i] * q_3[k-3][j][i];
_t_40_ += 0.038 * _t_43_;
_t_44_ = cons_2[k+4][j][i] * q_3[k+4][j][i];
_t_44_ -= cons_2[k-4][j][i] * q_3[k-4][j][i];
_t_40_ -= 0.0035 * _t_44_;
flux_2kc0jc0ic0 -= _t_40_ * dxinv2;
flux_2[k][j][i] = flux_2kc0jc0ic0;
flux2_b = flux_2[k+1][j][i];
flux_2kp1jc0ic0 = flux2_b;
_t_46_ = cons_2[k+2][j][i] * q_3[k+2][j][i];
_t_46_ -= cons_2[k][j][i] * q_3[k][j][i];
_t_45_ = 0.8 * _t_46_;
_t_47_ = cons_2[k+3][j][i] * q_3[k+3][j][i];
_t_47_ -= cons_2[k-1][j][i] * q_3[k-1][j][i];
_t_45_ -= 0.2 * _t_47_;
_t_48_ = cons_2[k+4][j][i] * q_3[k+4][j][i];
_t_48_ -= cons_2[k-2][j][i] * q_3[k-2][j][i];
_t_45_ += 0.038 * _t_48_;
_t_49_ = cons_2[k+5][j][i] * q_3[k+5][j][i];
_t_49_ -= cons_2[k-3][j][i] * q_3[k-3][j][i];
_t_45_ -= 0.0035 * _t_49_;
flux_2kp1jc0ic0 -= _t_45_ * dxinv2;
flux_2[k+1][j][i] = flux_2kp1jc0ic0;
flux2_c = flux_2[k+2][j][i];
flux_2kp2jc0ic0 = flux2_c;
_t_51_ = cons_2[k+3][j][i] * q_3[k+3][j][i];
_t_51_ -= cons_2[k+1][j][i] * q_3[k+1][j][i];
_t_50_ = 0.8 * _t_51_;
_t_52_ = cons_2[k+4][j][i] * q_3[k+4][j][i];
_t_52_ -= cons_2[k][j][i] * q_3[k][j][i];
_t_50_ -= 0.2 * _t_52_;
_t_53_ = cons_2[k+5][j][i] * q_3[k+5][j][i];
_t_53_ -= cons_2[k-1][j][i] * q_3[k-1][j][i];
_t_50_ += 0.038 * _t_53_;
_t_54_ = cons_2[k+6][j][i] * q_3[k+6][j][i];
_t_54_ -= cons_2[k-2][j][i] * q_3[k-2][j][i];
_t_50_ -= 0.0035 * _t_54_;
flux_2kp2jc0ic0 -= _t_50_ * dxinv2;
flux_2[k+2][j][i] = flux_2kp2jc0ic0;
flux2_d = flux_2[k+3][j][i];
flux_2kp3jc0ic0 = flux2_d;
_t_56_ = cons_2[k+4][j][i] * q_3[k+4][j][i];
_t_56_ -= cons_2[k+2][j][i] * q_3[k+2][j][i];
_t_55_ = 0.8 * _t_56_;
_t_57_ = cons_2[k+5][j][i] * q_3[k+5][j][i];
_t_57_ -= cons_2[k+1][j][i] * q_3[k+1][j][i];
_t_55_ -= 0.2 * _t_57_;
_t_58_ = cons_2[k+6][j][i] * q_3[k+6][j][i];
_t_58_ -= cons_2[k][j][i] * q_3[k][j][i];
_t_55_ += 0.038 * _t_58_;
_t_59_ = cons_2[k+7][j][i] * q_3[k+7][j][i];
_t_59_ -= cons_2[k-1][j][i] * q_3[k-1][j][i];
_t_55_ -= 0.0035 * _t_59_;
flux_2kp3jc0ic0 -= _t_55_ * dxinv2;
flux_2[k+3][j][i] = flux_2kp3jc0ic0;
flux3_a = flux_3[k][j][i];
flux_3kc0jc0ic0 = flux3_a;
_t_61_ = cons_3[k+1][j][i] * q_3[k+1][j][i];
_t_61_ -= cons_3[k-1][j][i] * q_3[k-1][j][i];
_t_61_ += q_4[k+1][j][i];
_t_61_ -= q_4[k-1][j][i];
_t_60_ = 0.8 * _t_61_;
_t_62_ = cons_3[k+2][j][i] * q_3[k+2][j][i];
_t_62_ -= cons_3[k-2][j][i] * q_3[k-2][j][i];
_t_62_ += q_4[k+2][j][i];
_t_62_ -= q_4[k-2][j][i];
_t_60_ -= 0.2 * _t_62_;
_t_63_ = cons_3[k+3][j][i] * q_3[k+3][j][i];
_t_63_ -= cons_3[k-3][j][i] * q_3[k-3][j][i];
_t_63_ += q_4[k+3][j][i];
_t_63_ -= q_4[k-3][j][i];
_t_60_ += 0.038 * _t_63_;
_t_64_ = cons_3[k+4][j][i] * q_3[k+4][j][i];
_t_64_ -= cons_3[k-4][j][i] * q_3[k-4][j][i];
_t_64_ += q_4[k+4][j][i];
_t_64_ -= q_4[k-4][j][i];
_t_60_ -= 0.0035 * _t_64_;
flux_3kc0jc0ic0 -= _t_60_ * dxinv2;
flux_3[k][j][i] = flux_3kc0jc0ic0;
flux3_b = flux_3[k+1][j][i];
flux_3kp1jc0ic0 = flux3_b;
_t_66_ = cons_3[k+2][j][i] * q_3[k+2][j][i];
_t_66_ -= cons_3[k][j][i] * q_3[k][j][i];
_t_66_ += q_4[k+2][j][i];
_t_66_ -= q_4[k][j][i];
_t_65_ = 0.8 * _t_66_;
_t_67_ = cons_3[k+3][j][i] * q_3[k+3][j][i];
_t_67_ -= cons_3[k-1][j][i] * q_3[k-1][j][i];
_t_67_ += q_4[k+3][j][i];
_t_67_ -= q_4[k-1][j][i];
_t_65_ -= 0.2 * _t_67_;
_t_68_ = cons_3[k+4][j][i] * q_3[k+4][j][i];
_t_68_ -= cons_3[k-2][j][i] * q_3[k-2][j][i];
_t_68_ += q_4[k+4][j][i];
_t_68_ -= q_4[k-2][j][i];
_t_65_ += 0.038 * _t_68_;
_t_69_ = cons_3[k+5][j][i] * q_3[k+5][j][i];
_t_69_ -= cons_3[k-3][j][i] * q_3[k-3][j][i];
_t_69_ += q_4[k+5][j][i];
_t_69_ -= q_4[k-3][j][i];
_t_65_ -= 0.0035 * _t_69_;
flux_3kp1jc0ic0 -= _t_65_ * dxinv2;
flux_3[k+1][j][i] = flux_3kp1jc0ic0;
flux3_c = flux_3[k+2][j][i];
flux_3kp2jc0ic0 = flux3_c;
_t_71_ = cons_3[k+3][j][i] * q_3[k+3][j][i];
_t_71_ -= cons_3[k+1][j][i] * q_3[k+1][j][i];
_t_71_ += q_4[k+3][j][i];
_t_71_ -= q_4[k+1][j][i];
_t_70_ = 0.8 * _t_71_;
_t_72_ = cons_3[k+4][j][i] * q_3[k+4][j][i];
_t_72_ -= cons_3[k][j][i] * q_3[k][j][i];
_t_72_ += q_4[k+4][j][i];
_t_72_ -= q_4[k][j][i];
_t_70_ -= 0.2 * _t_72_;
_t_73_ = cons_3[k+5][j][i] * q_3[k+5][j][i];
_t_73_ -= cons_3[k-1][j][i] * q_3[k-1][j][i];
_t_73_ += q_4[k+5][j][i];
_t_73_ -= q_4[k-1][j][i];
_t_70_ += 0.038 * _t_73_;
_t_74_ = cons_3[k+6][j][i] * q_3[k+6][j][i];
_t_74_ -= cons_3[k-2][j][i] * q_3[k-2][j][i];
_t_74_ += q_4[k+6][j][i];
_t_74_ -= q_4[k-2][j][i];
_t_70_ -= 0.0035 * _t_74_;
flux_3kp2jc0ic0 -= _t_70_ * dxinv2;
flux_3[k+2][j][i] = flux_3kp2jc0ic0;
flux3_d = flux_3[k+3][j][i];
flux_3kp3jc0ic0 = flux3_d;
_t_76_ = cons_3[k+4][j][i] * q_3[k+4][j][i];
_t_76_ -= cons_3[k+2][j][i] * q_3[k+2][j][i];
_t_76_ += q_4[k+4][j][i];
_t_76_ -= q_4[k+2][j][i];
_t_75_ = 0.8 * _t_76_;
_t_77_ = cons_3[k+5][j][i] * q_3[k+5][j][i];
_t_77_ -= cons_3[k+1][j][i] * q_3[k+1][j][i];
_t_77_ += q_4[k+5][j][i];
_t_77_ -= q_4[k+1][j][i];
_t_75_ -= 0.2 * _t_77_;
_t_78_ = cons_3[k+6][j][i] * q_3[k+6][j][i];
_t_78_ -= cons_3[k][j][i] * q_3[k][j][i];
_t_78_ += q_4[k+6][j][i];
_t_78_ -= q_4[k][j][i];
_t_75_ += 0.038 * _t_78_;
_t_79_ = cons_3[k+7][j][i] * q_3[k+7][j][i];
_t_79_ -= cons_3[k-1][j][i] * q_3[k-1][j][i];
_t_79_ += q_4[k+7][j][i];
_t_79_ -= q_4[k-1][j][i];
_t_75_ -= 0.0035 * _t_79_;
flux_3kp3jc0ic0 -= _t_75_ * dxinv2;
flux_3[k+3][j][i] = flux_3kp3jc0ic0;
	} 
}

__global__ void hypterm_2 (double * __restrict__ flux_in_4, double * __restrict__ cons_in_4, double * __restrict__ q_in_1, double * __restrict__ q_in_2, double * __restrict__ q_in_3, double * __restrict__ q_in_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	int blockdim_k= (int)(blockDim.z);
	int k0 = (int)(blockIdx.z)*(2*blockdim_k);
	int k = max (k0, 0) + (int)(2*threadIdx.z);

	double (*flux_4)[308][308] = (double (*)[308][308])flux_in_4;
	double (*q_1)[308][308] = (double (*)[308][308])q_in_1;
	double (*q_2)[308][308] = (double (*)[308][308])q_in_2;
	double (*q_3)[308][308] = (double (*)[308][308])q_in_3;
	double (*q_4)[308][308] = (double (*)[308][308])q_in_4;
	double (*cons_4)[308][308] = (double (*)[308][308])cons_in_4;

	if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
		flux_4[k][j][i] = ((0.8*(cons_4[k][j][i+1]*q_1[k][j][i+1]-cons_4[k][j][i-1]*q_1[k][j][i-1]+(q_4[k][j][i+1]*q_1[k][j][i+1]-q_4[k][j][i-1]*q_1[k][j][i-1]))-0.2*(cons_4[k][j][i+2]*q_1[k][j][i+2]-cons_4[k][j][i-2]*q_1[k][j][i-2]+(q_4[k][j][i+2]*q_1[k][j][i+2]-q_4[k][j][i-2]*q_1[k][j][i-2]))+0.038*(cons_4[k][j][i+3]*q_1[k][j][i+3]-cons_4[k][j][i-3]*q_1[k][j][i-3]+(q_4[k][j][i+3]*q_1[k][j][i+3]-q_4[k][j][i-3]*q_1[k][j][i-3]))-0.0035*(cons_4[k][j][i+4]*q_1[k][j][i+4]-cons_4[k][j][i-4]*q_1[k][j][i-4]+(q_4[k][j][i+4]*q_1[k][j][i+4]-q_4[k][j][i-4]*q_1[k][j][i-4])))*dxinv0);
		flux_4[k+1][j][i] = ((0.8*(cons_4[k+1][j][i+1]*q_1[k+1][j][i+1]-cons_4[k+1][j][i-1]*q_1[k+1][j][i-1]+(q_4[k+1][j][i+1]*q_1[k+1][j][i+1]-q_4[k+1][j][i-1]*q_1[k+1][j][i-1]))-0.2*(cons_4[k+1][j][i+2]*q_1[k+1][j][i+2]-cons_4[k+1][j][i-2]*q_1[k+1][j][i-2]+(q_4[k+1][j][i+2]*q_1[k+1][j][i+2]-q_4[k+1][j][i-2]*q_1[k+1][j][i-2]))+0.038*(cons_4[k+1][j][i+3]*q_1[k+1][j][i+3]-cons_4[k+1][j][i-3]*q_1[k+1][j][i-3]+(q_4[k+1][j][i+3]*q_1[k+1][j][i+3]-q_4[k+1][j][i-3]*q_1[k+1][j][i-3]))-0.0035*(cons_4[k+1][j][i+4]*q_1[k+1][j][i+4]-cons_4[k+1][j][i-4]*q_1[k+1][j][i-4]+(q_4[k+1][j][i+4]*q_1[k+1][j][i+4]-q_4[k+1][j][i-4]*q_1[k+1][j][i-4])))*dxinv0);
		flux_4[k][j][i] -= (0.8*(cons_4[k][j+1][i]*q_2[k][j+1][i]-cons_4[k][j-1][i]*q_2[k][j-1][i]+(q_4[k][j+1][i]*q_2[k][j+1][i]-q_4[k][j-1][i]*q_2[k][j-1][i]))-0.2*(cons_4[k][j+2][i]*q_2[k][j+2][i]-cons_4[k][j-2][i]*q_2[k][j-2][i]+(q_4[k][j+2][i]*q_2[k][j+2][i]-q_4[k][j-2][i]*q_2[k][j-2][i]))+0.038*(cons_4[k][j+3][i]*q_2[k][j+3][i]-cons_4[k][j-3][i]*q_2[k][j-3][i]+(q_4[k][j+3][i]*q_2[k][j+3][i]-q_4[k][j-3][i]*q_2[k][j-3][i]))-0.0035*(cons_4[k][j+4][i]*q_2[k][j+4][i]-cons_4[k][j-4][i]*q_2[k][j-4][i]+(q_4[k][j+4][i]*q_2[k][j+4][i]-q_4[k][j-4][i]*q_2[k][j-4][i])))*dxinv1;
		flux_4[k+1][j][i] -= (0.8*(cons_4[k+1][j+1][i]*q_2[k+1][j+1][i]-cons_4[k+1][j-1][i]*q_2[k+1][j-1][i]+(q_4[k+1][j+1][i]*q_2[k+1][j+1][i]-q_4[k+1][j-1][i]*q_2[k+1][j-1][i]))-0.2*(cons_4[k+1][j+2][i]*q_2[k+1][j+2][i]-cons_4[k+1][j-2][i]*q_2[k+1][j-2][i]+(q_4[k+1][j+2][i]*q_2[k+1][j+2][i]-q_4[k+1][j-2][i]*q_2[k+1][j-2][i]))+0.038*(cons_4[k+1][j+3][i]*q_2[k+1][j+3][i]-cons_4[k+1][j-3][i]*q_2[k+1][j-3][i]+(q_4[k+1][j+3][i]*q_2[k+1][j+3][i]-q_4[k+1][j-3][i]*q_2[k+1][j-3][i]))-0.0035*(cons_4[k+1][j+4][i]*q_2[k+1][j+4][i]-cons_4[k+1][j-4][i]*q_2[k+1][j-4][i]+(q_4[k+1][j+4][i]*q_2[k+1][j+4][i]-q_4[k+1][j-4][i]*q_2[k+1][j-4][i])))*dxinv1;
		flux_4[k][j][i] -= (0.8*(cons_4[k+1][j][i]*q_3[k+1][j][i]-cons_4[k-1][j][i]*q_3[k-1][j][i]+(q_4[k+1][j][i]*q_3[k+1][j][i]-q_4[k-1][j][i]*q_3[k-1][j][i]))-0.2*(cons_4[k+2][j][i]*q_3[k+2][j][i]-cons_4[k-2][j][i]*q_3[k-2][j][i]+(q_4[k+2][j][i]*q_3[k+2][j][i]-q_4[k-2][j][i]*q_3[k-2][j][i]))+0.038*(cons_4[k+3][j][i]*q_3[k+3][j][i]-cons_4[k-3][j][i]*q_3[k-3][j][i]+(q_4[k+3][j][i]*q_3[k+3][j][i]-q_4[k-3][j][i]*q_3[k-3][j][i]))-0.0035*(cons_4[k+4][j][i]*q_3[k+4][j][i]-cons_4[k-4][j][i]*q_3[k-4][j][i]+(q_4[k+4][j][i]*q_3[k+4][j][i]-q_4[k-4][j][i]*q_3[k-4][j][i])))*dxinv2;
		flux_4[k+1][j][i] -= (0.8*(cons_4[k+1+1][j][i]*q_3[k+1+1][j][i]-cons_4[k+1-1][j][i]*q_3[k+1-1][j][i]+(q_4[k+1+1][j][i]*q_3[k+1+1][j][i]-q_4[k+1-1][j][i]*q_3[k+1-1][j][i]))-0.2*(cons_4[k+1+2][j][i]*q_3[k+1+2][j][i]-cons_4[k+1-2][j][i]*q_3[k+1-2][j][i]+(q_4[k+1+2][j][i]*q_3[k+1+2][j][i]-q_4[k+1-2][j][i]*q_3[k+1-2][j][i]))+0.038*(cons_4[k+1+3][j][i]*q_3[k+1+3][j][i]-cons_4[k+1-3][j][i]*q_3[k+1-3][j][i]+(q_4[k+1+3][j][i]*q_3[k+1+3][j][i]-q_4[k+1-3][j][i]*q_3[k+1-3][j][i]))-0.0035*(cons_4[k+1+4][j][i]*q_3[k+1+4][j][i]-cons_4[k+1-4][j][i]*q_3[k+1-4][j][i]+(q_4[k+1+4][j][i]*q_3[k+1+4][j][i]-q_4[k+1-4][j][i]*q_3[k+1-4][j][i])))*dxinv2;
	} 
}

extern "C" void host_code (double *h_flux_0, double *h_flux_1, double *h_flux_2, double *h_flux_3, double *h_flux_4, double *h_cons_1, double *h_cons_2, double *h_cons_3, double *h_cons_4, double *h_q_1, double *h_q_2, double *h_q_3, double *h_q_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	double *flux_0;
	hipMalloc (&flux_0, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_0\n");
	hipMemcpy (flux_0, h_flux_0, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_1;
	hipMalloc (&flux_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_1\n");
	hipMemcpy (flux_1, h_flux_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_2;
	hipMalloc (&flux_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_2\n");
	hipMemcpy (flux_2, h_flux_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_3;
	hipMalloc (&flux_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_3\n");
	hipMemcpy (flux_3, h_flux_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_4;
	hipMalloc (&flux_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_4\n");
	hipMemcpy (flux_4, h_flux_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_1;
	hipMalloc (&cons_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_1\n");
	hipMemcpy (cons_1, h_cons_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_2;
	hipMalloc (&cons_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_2\n");
	hipMemcpy (cons_2, h_cons_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_3;
	hipMalloc (&cons_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_3\n");
	hipMemcpy (cons_3, h_cons_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_4;
	hipMalloc (&cons_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_4\n");
	hipMemcpy (cons_4, h_cons_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_1;
	hipMalloc (&q_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_1\n");
	hipMemcpy (q_1, h_q_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_2;
	hipMalloc (&q_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_2\n");
	hipMemcpy (q_2, h_q_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_3;
	hipMalloc (&q_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_3\n");
	hipMemcpy (q_3, h_q_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_4;
	hipMalloc (&q_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_4\n");
	hipMemcpy (q_4, h_q_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 4, 4);
	dim3 gridconfig_0 (ceil(N, blockconfig.x), ceil(M, blockconfig.y), ceil(L, blockconfig.z));
	hypterm_0 <<<gridconfig_0, blockconfig>>> (flux_0, flux_1, flux_2, flux_3, cons_1, cons_2, cons_3, q_1, q_2, q_3, q_4, -dxinv0, dxinv1, dxinv2, L, M, N);
	dim3 gridconfig_1 (ceil(N, blockconfig.x), ceil(M, blockconfig.y), ceil(L, 4*blockconfig.z));
	hypterm_1 <<<gridconfig_1, blockconfig>>> (flux_0, flux_1, flux_2, flux_3, cons_1, cons_2, cons_3, q_1, q_2, q_3, q_4, -dxinv0, dxinv1, dxinv2, L, M, N);
	dim3 gridconfig_2 (ceil(N, blockconfig.x), ceil(M, blockconfig.y), ceil(L, 2*blockconfig.z));
	hypterm_2 <<<gridconfig_2, blockconfig>>> (flux_4, cons_4, q_1, q_2, q_3, q_4, -dxinv0, dxinv1, dxinv2, L, M, N);

	hipMemcpy (h_flux_0, flux_0, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_1, flux_1, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_3, flux_3, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_4, flux_4, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_2, flux_2, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
}
