#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void curvi (double * __restrict__ in_r1, double *__restrict__ in_u1, double * __restrict__ in_u2, double *__restrict__ in_u3, double * __restrict__ in_mu, double * __restrict__ in_la, double * __restrict__ in_met1, double * __restrict__ in_met2, double * __restrict__ in_met3, double * __restrict__ in_met4, double * strx, double * stry, double c1, double c2, int N) {
	//Determing the block's indices
	int blockdim_k= (int)(blockDim.x);
	int k0 = (int)(blockIdx.x)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	double (*u1)[304][304] = (double (*)[304][304])in_u1;
	double (*u2)[304][304] = (double (*)[304][304])in_u2;
	double (*u3)[304][304] = (double (*)[304][304])in_u3;
	double (*mu)[304][304] = (double (*)[304][304])in_mu;
	double (*la)[304][304] = (double (*)[304][304])in_la;
	double (*r1)[304][304] = (double (*)[304][304])in_r1;
	double (*met1)[304][304] = (double (*)[304][304])in_met1;
	double (*met2)[304][304] = (double (*)[304][304])in_met2;
	double (*met3)[304][304] = (double (*)[304][304])in_met3;
	double (*met4)[304][304] = (double (*)[304][304])in_met4;

	if (j>=2 & k>=2 & j<=N-3 & k<=N-3) {
		for (int i=2; i<=N-3; i++) {
double _t_86_ = 2.0 * mu[i+2][j][k];
_t_86_ += la[i+2][j][k];
double _t_83_ = met1[i+2][j][k] * _t_86_ * met2[i+2][j][k];
double _v_38_ = c2 * u1[i+2][j][k+2];
double _v_0_ = c2 * u1[i+2][j][k+2];
double _v_76_ = c2 * u1[i][j+2][k+2];
_v_76_ -= c2 * u1[i][j-2][k+2];
double _v_79_ = c2 * u2[i][j+2][k+2];
_v_79_ -= c2 * u2[i][j-2][k+2];
double _v_82_ = c2 * u1[i][j+2][k-2];
_v_82_ -= c2 * u1[i][j-2][k-2];
double _v_85_ = c2 * u2[i][j+2][k-2];
_v_85_ -= c2 * u2[i][j-2][k-2];
double _v_89_ = c2 * u1[i][j+2][k+1];
_v_89_ -= c2 * u1[i][j-2][k+1];
double _v_92_ = c2 * u2[i][j+2][k+1];
_v_92_ -= c2 * u2[i][j-2][k+1];
double _v_95_ = c2 * u1[i][j+2][k-1];
_v_95_ -= c2 * u1[i][j-2][k-1];
double _v_98_ = c2 * u2[i][j+2][k-1];
_v_98_ -= c2 * u2[i][j-2][k-1];
_v_38_ -= c2 * u1[i+2][j][k-2];
double _v_9_ = c2 * u1[i+2][j][k-2];
double _t_84_ = _v_38_;
double _v_39_ = c1 * u1[i+2][j][k+1];
double _v_77_ = c1 * u1[i][j+1][k+2];
_v_77_ -= c1 * u1[i][j-1][k+2];
double _v_80_ = c1 * u2[i][j+1][k+2];
_v_80_ -= c1 * u2[i][j-1][k+2];
double _v_83_ = c1 * u1[i][j+1][k-2];
_v_83_ -= c1 * u1[i][j-1][k-2];
double _v_86_ = c1 * u2[i][j+1][k-2];
_v_86_ -= c1 * u2[i][j-1][k-2];
double _v_90_ = c1 * u1[i][j+1][k+1];
_v_90_ -= c1 * u1[i][j-1][k+1];
double _v_93_ = c1 * u2[i][j+1][k+1];
_v_93_ -= c1 * u2[i][j-1][k+1];
double _v_96_ = c1 * u1[i][j+1][k-1];
_v_96_ -= c1 * u1[i][j-1][k-1];
double _v_99_ = c1 * u2[i][j+1][k-1];
_v_99_ -= c1 * u2[i][j-1][k-1];
_v_39_ -= c1 * u1[i+2][j][k-1];
_t_84_ += _v_39_;
double _v_40_ = strx[i] * _t_83_ * _t_84_;
double _v_19_ = c2 * u1[i+2][j][k+1];
double _v_28_ = c2 * u1[i+2][j][k-1];
double _v_56_ = c2 * _v_40_;
double _v_41_ = c2 * u2[i+2][j][k+2];
double _v_3_ = c2 * u2[i+2][j][k+2];
_v_41_ -= c2 * u2[i+2][j][k-2];
double _v_12_ = c2 * u2[i+2][j][k-2];
double _t_91_ = _v_41_;
double _v_42_ = c1 * u2[i+2][j][k+1];
_v_42_ -= c1 * u2[i+2][j][k-1];
_t_91_ += _v_42_;
double _t_90_ = met1[i+2][j][k] * la[i+2][j][k] * met3[i+2][j][k];
double _v_43_ = stry[j] * _t_90_ * _t_91_;
double _v_22_ = c2 * u2[i+2][j][k+1];
double _v_31_ = c2 * u2[i+2][j][k-1];
_v_56_ += c2 * _v_43_;
double _t_95_ = met1[i+2][j][k] * la[i+2][j][k] * met4[i+2][j][k];
double _v_44_ = c2 * u3[i+2][j][k+2];
double _v_6_ = c2 * u3[i+2][j][k+2];
_v_44_ -= c2 * u3[i+2][j][k-2];
double _v_15_ = c2 * u3[i+2][j][k-2];
double _t_96_ = _v_44_;
double _v_45_ = c1 * u3[i+2][j][k+1];
_v_45_ -= c1 * u3[i+2][j][k-1];
_t_96_ += _v_45_;
double _v_46_ = _t_95_ * _t_96_;
double _v_25_ = c2 * u3[i+2][j][k+1];
double _v_34_ = c2 * u3[i+2][j][k-1];
_v_56_ += c2 * _v_46_;
double _t_104_ = 2.0 * mu[i-2][j][k];
_t_104_ += la[i-2][j][k];
double _t_101_ = met1[i-2][j][k] * _t_104_ * met2[i-2][j][k];
double _v_47_ = c2 * u1[i-2][j][k+2];
_v_0_ -= c2 * u1[i-2][j][k+2];
_v_47_ -= c2 * u1[i-2][j][k-2];
_v_9_ -= c2 * u1[i-2][j][k-2];
double _t_102_ = _v_47_;
double _v_48_ = c1 * u1[i-2][j][k+1];
_v_48_ -= c1 * u1[i-2][j][k-1];
_t_102_ += _v_48_;
double _v_49_ = strx[i] * _t_101_ * _t_102_;
_v_19_ -= c2 * u1[i-2][j][k+1];
_v_28_ -= c2 * u1[i-2][j][k-1];
_v_56_ += c2 * _v_49_;
double _v_50_ = c2 * u2[i-2][j][k+2];
_v_3_ -= c2 * u2[i-2][j][k+2];
_v_50_ -= c2 * u2[i-2][j][k-2];
_v_12_ -= c2 * u2[i-2][j][k-2];
double _t_109_ = _v_50_;
double _v_51_ = c1 * u2[i-2][j][k+1];
_v_51_ -= c1 * u2[i-2][j][k-1];
_t_109_ += _v_51_;
double _t_108_ = met1[i-2][j][k] * la[i-2][j][k] * met3[i-2][j][k];
double _v_52_ = stry[j] * _t_108_ * _t_109_;
_v_22_ -= c2 * u2[i-2][j][k+1];
_v_31_ -= c2 * u2[i-2][j][k-1];
_v_56_ += c2 * _v_52_;
double _t_113_ = met1[i-2][j][k] * la[i-2][j][k] * met4[i-2][j][k];
double _v_53_ = c2 * u3[i-2][j][k+2];
_v_6_ -= c2 * u3[i-2][j][k+2];
_v_53_ -= c2 * u3[i-2][j][k-2];
_v_15_ -= c2 * u3[i-2][j][k-2];
double _t_114_ = _v_53_;
double _v_54_ = c1 * u3[i-2][j][k+1];
_v_54_ -= c1 * u3[i-2][j][k-1];
_t_114_ += _v_54_;
double _v_55_ = _t_113_ * _t_114_;
_v_25_ -= c2 * u3[i-2][j][k+1];
_v_34_ -= c2 * u3[i-2][j][k-1];
_v_56_ += c2 * _v_55_;
double _t_79_ = stry[j] * _v_56_;
double _t_123_ = 2.0 * mu[i+1][j][k];
_t_123_ += la[i+1][j][k];
double _t_120_ = met1[i+1][j][k] * _t_123_ * met2[i+1][j][k];
double _v_57_ = c2 * u1[i+1][j][k+2];
_v_57_ -= c2 * u1[i+1][j][k-2];
double _t_121_ = _v_57_;
double _v_58_ = c1 * u1[i+1][j][k+1];
double _v_20_ = c1 * u1[i+1][j][k+1];
_v_58_ -= c1 * u1[i+1][j][k-1];
double _v_29_ = c1 * u1[i+1][j][k-1];
_t_121_ += _v_58_;
double _v_59_ = strx[i] * _t_120_ * _t_121_;
double _v_1_ = c1 * u1[i+1][j][k+2];
double _v_10_ = c1 * u1[i+1][j][k-2];
double _v_75_ = c1 * _v_59_;
double _v_60_ = c2 * u2[i+1][j][k+2];
_v_60_ -= c2 * u2[i+1][j][k-2];
double _t_128_ = _v_60_;
double _v_61_ = c1 * u2[i+1][j][k+1];
double _v_23_ = c1 * u2[i+1][j][k+1];
_v_61_ -= c1 * u2[i+1][j][k-1];
double _v_32_ = c1 * u2[i+1][j][k-1];
_t_128_ += _v_61_;
double _t_127_ = met1[i+1][j][k] * la[i+1][j][k] * met3[i+1][j][k];
double _v_62_ = stry[j] * _t_127_ * _t_128_;
double _v_4_ = c1 * u2[i+1][j][k+2];
double _v_13_ = c1 * u2[i+1][j][k-2];
_v_75_ += c1 * _v_62_;
double _t_132_ = met1[i+1][j][k] * la[i+1][j][k] * met4[i+1][j][k];
double _v_63_ = c2 * u3[i+1][j][k+2];
_v_63_ -= c2 * u3[i+1][j][k-2];
double _t_133_ = _v_63_;
double _v_64_ = c1 * u3[i+1][j][k+1];
double _v_26_ = c1 * u3[i+1][j][k+1];
_v_64_ -= c1 * u3[i+1][j][k-1];
double _v_35_ = c1 * u3[i+1][j][k-1];
_t_133_ += _v_64_;
double _v_65_ = _t_132_ * _t_133_;
double _v_7_ = c1 * u3[i+1][j][k+2];
double _v_16_ = c1 * u3[i+1][j][k-2];
_v_75_ += c1 * _v_65_;
double _t_141_ = 2.0 * mu[i-1][j][k];
_t_141_ += la[i-1][j][k];
double _t_138_ = met1[i-1][j][k] * _t_141_ * met2[i-1][j][k];
double _v_66_ = c2 * u1[i-1][j][k+2];
_v_66_ -= c2 * u1[i-1][j][k-2];
double _t_139_ = _v_66_;
double _v_67_ = c1 * u1[i-1][j][k+1];
_v_20_ -= c1 * u1[i-1][j][k+1];
_v_67_ -= c1 * u1[i-1][j][k-1];
_v_29_ -= c1 * u1[i-1][j][k-1];
_t_139_ += _v_67_;
double _v_68_ = strx[i] * _t_138_ * _t_139_;
_v_1_ -= c1 * u1[i-1][j][k+2];
_v_10_ -= c1 * u1[i-1][j][k-2];
_v_75_ += c1 * _v_68_;
double _v_69_ = c2 * u2[i-1][j][k+2];
_v_69_ -= c2 * u2[i-1][j][k-2];
double _t_146_ = _v_69_;
double _v_70_ = c1 * u2[i-1][j][k+1];
_v_23_ -= c1 * u2[i-1][j][k+1];
_v_70_ -= c1 * u2[i-1][j][k-1];
_v_32_ -= c1 * u2[i-1][j][k-1];
_t_146_ += _v_70_;
double _t_145_ = met1[i-1][j][k] * la[i-1][j][k] * met3[i-1][j][k];
double _v_71_ = stry[j] * _t_145_ * _t_146_;
_v_4_ -= c1 * u2[i-1][j][k+2];
_v_13_ -= c1 * u2[i-1][j][k-2];
_v_75_ += c1 * _v_71_;
double _t_150_ = met1[i-1][j][k] * la[i-1][j][k] * met4[i-1][j][k];
double _v_72_ = c2 * u3[i-1][j][k+2];
_v_72_ -= c2 * u3[i-1][j][k-2];
double _t_151_ = _v_72_;
double _v_73_ = c1 * u3[i-1][j][k+1];
_v_26_ -= c1 * u3[i-1][j][k+1];
_v_73_ -= c1 * u3[i-1][j][k-1];
_v_35_ -= c1 * u3[i-1][j][k-1];
_t_151_ += _v_73_;
double _v_74_ = _t_150_ * _t_151_;
_v_7_ -= c1 * u3[i-1][j][k+2];
_v_16_ -= c1 * u3[i-1][j][k-2];
_v_75_ += c1 * _v_74_;
_t_79_ += stry[j] * _v_75_;
double r1ic0jc0kc0 = r1[i][j][k];
r1ic0jc0kc0 += _t_79_;

double _t_17_ = _v_6_;
_t_17_ += _v_7_;
double _t_16_ = met1[i][j][k+2] * mu[i][j][k+2] * met4[i][j][k+2];
double _v_8_ = stry[j] * _t_16_ * _t_17_;
double _v_18_ = c2 * _v_8_;
double _t_5_ = _v_0_;
_t_5_ += _v_1_;
double _t_7_ = 2.0 * mu[i][j][k+2];
double _t_10_ = met1[i][j][k+2] * mu[i][j][k+2] * met3[i][j][k+2];
_t_7_ += la[i][j][k+2];
double _t_4_ = met1[i][j][k+2] * _t_7_ * met2[i][j][k+2];
double _t_164_ = met1[i][j][k+2] * la[i][j][k+2] * met2[i][j][k+2];
double _t_3_ = _t_4_ * _t_5_;
double _v_2_ = stry[j] * _t_3_ * strx[i];
_v_18_ += c2 * _v_2_;
double _t_11_ = _v_3_;
_t_11_ += _v_4_;
double _v_5_ = _t_10_ * _t_11_;
_v_18_ += c2 * _v_5_;
double _t_24_ = _v_9_;
_t_24_ += _v_10_;
double _t_26_ = 2.0 * mu[i][j][k-2];
_t_26_ += la[i][j][k-2];
double _t_23_ = met1[i][j][k-2] * _t_26_ * met2[i][j][k-2];
double _t_176_ = met1[i][j][k-2] * la[i][j][k-2] * met2[i][j][k-2];
double _t_22_ = _t_23_ * _t_24_;
double _v_11_ = stry[j] * _t_22_ * strx[i];
_v_18_ += c2 * _v_11_;
double _t_30_ = _v_12_;
_t_30_ += _v_13_;
double _t_29_ = met1[i][j][k-2] * mu[i][j][k-2] * met3[i][j][k-2];
double _t_35_ = met1[i][j][k-2] * mu[i][j][k-2] * met4[i][j][k-2];
double _v_14_ = _t_29_ * _t_30_;
_v_18_ += c2 * _v_14_;
double _t_36_ = _v_15_;
_t_36_ += _v_16_;
double _v_17_ = stry[j] * _t_35_ * _t_36_;
_v_18_ += c2 * _v_17_;
double _t_0_ = _v_18_;
double _t_56_ = _v_25_;
_t_56_ += _v_26_;
double _t_55_ = met1[i][j][k+1] * mu[i][j][k+1] * met4[i][j][k+1];
double _v_27_ = stry[j] * _t_55_ * _t_56_;
double _v_37_ = c1 * _v_27_;
double _t_44_ = _v_19_;
_t_44_ += _v_20_;
double _t_46_ = 2.0 * mu[i][j][k+1];
double _t_49_ = met1[i][j][k+1] * mu[i][j][k+1] * met3[i][j][k+1];
_t_46_ += la[i][j][k+1];
double _t_43_ = met1[i][j][k+1] * _t_46_ * met2[i][j][k+1];
double _t_189_ = met1[i][j][k+1] * la[i][j][k+1] * met2[i][j][k+1];
double _t_42_ = _t_43_ * _t_44_;
double _v_21_ = stry[j] * _t_42_ * strx[i+2];
_v_37_ += c1 * _v_21_;
double _t_50_ = _v_22_;
_t_50_ += _v_23_;
double _v_24_ = _t_49_ * _t_50_;
_v_37_ += c1 * _v_24_;
double _t_63_ = _v_28_;
_t_63_ += _v_29_;
double _t_65_ = 2.0 * mu[i][j][k-1];
_t_65_ += la[i][j][k-1];
double _t_62_ = met1[i][j][k-1] * _t_65_ * met2[i][j][k-1];
double _t_201_ = met1[i][j][k-1] * la[i][j][k-1] * met2[i][j][k-1];
double _t_61_ = _t_62_ * _t_63_;
double _v_30_ = stry[j] * _t_61_ * strx[i-2];
_v_37_ += c1 * _v_30_;
double _t_69_ = _v_31_;
_t_69_ += _v_32_;
double _t_68_ = met1[i][j][k-1] * mu[i][j][k-1] * met3[i][j][k-1];
double _t_74_ = met1[i][j][k-1] * mu[i][j][k-1] * met4[i][j][k-1];
double _v_33_ = _t_68_ * _t_69_;
_v_37_ += c1 * _v_33_;
double _t_75_ = _v_34_;
_t_75_ += _v_35_;
double _v_36_ = stry[j] * _t_74_ * _t_75_;
_v_37_ += c1 * _v_36_;
_t_0_ += _v_37_;
r1ic0jc0kc0 += _t_0_;

double _t_159_ = _t_10_;
double _t_160_ = _v_76_;
_t_160_ += _v_77_;
double _t_158_ = _t_159_ * _t_160_;
double _v_78_ = strx[i] * _t_158_ * stry[j+2];
double _v_88_ = c2 * _v_78_;
double _t_165_ = _v_79_;
_t_165_ += _v_80_;
double _v_81_ = _t_164_ * _t_165_;
_v_88_ += c2 * _v_81_;
double _t_171_ = _t_29_;
double _t_172_ = _v_82_;
_t_172_ += _v_83_;
double _t_170_ = _t_171_ * _t_172_;
double _v_84_ = strx[i] * _t_170_ * stry[j];
_v_88_ += c2 * _v_84_;
double _t_177_ = _v_85_;
_t_177_ += _v_86_;
double _v_87_ = _t_176_ * _t_177_;
_v_88_ += c2 * _v_87_;
double _t_155_ = _v_88_;
double _t_184_ = _t_49_;
double _t_185_ = _v_89_;
_t_185_ += _v_90_;
double _t_183_ = _t_184_ * _t_185_;
double _v_91_ = strx[i] * _t_183_ * stry[j-2];
double _v_101_ = c1 * _v_91_;
double _t_190_ = _v_92_;
_t_190_ += _v_93_;
double _v_94_ = _t_189_ * _t_190_;
_v_101_ += c1 * _v_94_;
double _t_196_ = _t_68_;
double _t_197_ = _v_95_;
_t_197_ += _v_96_;
double _t_195_ = _t_196_ * _t_197_;
double _v_97_ = strx[i] * _t_195_ * stry[j];
_v_101_ += c1 * _v_97_;
double _t_202_ = _v_98_;
_t_202_ += _v_99_;
double _v_100_ = _t_201_ * _t_202_;
_v_101_ += c1 * _v_100_;
_t_155_ += _v_101_;
r1ic0jc0kc0 += _t_155_;

r1[i][j][k] = r1ic0jc0kc0;

			r1[i][j][k] += c2*(
					mu[i][j+2][k]*met3[i][j+2][k]*met1[i][j+2][k]*(
						c2*(u1[i][j+2][k+2]-u1[i][j+2][k-2]) +
						c1*(u1[i][j+2][k+1]-u1[i][j+2][k-1])   )*stry[j+1]*strx[i]
					+ mu[i][j+2][k]*met2[i][j+2][k]*met1[i][j+2][k]*(
						c2*(u2[i][j+2][k+2]-u2[i][j+2][k-2]) +
						c1*(u2[i][j+2][k+1]-u2[i][j+2][k-1])  )
					+ ( mu[i][j-2][k]*met3[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u1[i][j-2][k+2]-u1[i][j-2][k-2]) +
							c1*(u1[i][j-2][k+1]-u1[i][j-2][k-1])  )*stry[j]*strx[i]
						+ mu[i][j-2][k]*met2[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u2[i][j-2][k+2]-u2[i][j-2][k-2]) +
							c1*(u2[i][j-2][k+1]-u2[i][j-2][k-1])   ) )
					) + c1*(
						mu[i][j+1][k]*met3[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u1[i][j+1][k+2]-u1[i][j+1][k-2]) +
							c1*(u1[i][j+1][k+1]-u1[i][j+1][k-1]) )*stry[j-1]*strx[i]
						+ mu[i][j+1][k]*met2[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u2[i][j+1][k+2]-u2[i][j+1][k-2]) +
							c1*(u2[i][j+1][k+1]-u2[i][j+1][k-1]) )
						+ ( mu[i][j-1][k]*met3[i][j-1][k]*met1[i][j-1][k]*(
								c2*(u1[i][j-1][k+2]-u1[i][j-1][k-2]) +
								c1*(u1[i][j-1][k+1]-u1[i][j-1][k-1]) )*stry[j]*strx[i]
							+ mu[i][j-1][k]*met2[i][j-1][k]*met1[i][j-1][k]*(
								c2*(u2[i][j-1][k+2]-u2[i][j-1][k-2]) +
								c1*(u2[i][j-1][k+1]-u2[i][j-1][k-1]) ) ) );

			r1[i][j][k] +=
				c2*(  mu[i][j+2][k]*met1[i][j+2][k]*met1[i][j+2][k]*(
							c2*(u2[i+2][j+2][k]-u2[i-2][j+2][k]) +
							c1*(u2[i+1][j+2][k]-u2[i-1][j+2][k])    )
						+  mu[i][j-2][k]*met1[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u2[i+2][j-2][k]-u2[i-2][j-2][k])+
							c1*(u2[i+1][j-2][k]-u2[i-1][j-2][k])     )
				   ) +
				c1*(  mu[i][j+1][k]*met1[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u2[i+2][j+1][k]-u2[i-2][j+1][k]) +
							c1*(u2[i+1][j+1][k]-u2[i-1][j+1][k])  )
						+ mu[i][j-1][k]*met1[i][j-1][k]*met1[i][j-1][k]*(
							c2*(u2[i+2][j-1][k]-u2[i-2][j-1][k]) +
							c1*(u2[i+1][j-1][k]-u2[i-1][j-1][k])))
				+
				c2*(  la[i+2][j][k]*met1[i+2][j][k]*met1[i+2][j][k]*(
							c2*(u2[i+2][j+2][k]-u2[i+2][j-2][k]) +
							c1*(u2[i+2][j+1][k]-u2[i+2][j-1][k])    )
						+ la[i-2][j][k]*met1[i-2][j][k]*met1[i-2][j][k]*(
							c2*(u2[i-2][j+2][k]-u2[i-2][j-2][k])+
							c1*(u2[i-2][j+1][k]-u2[i-2][j-1][k])     )
				   ) +
				c1*(  la[i+1][j][k]*met1[i+1][j][k]*met1[i+1][j][k]*(
							c2*(u2[i+1][j+2][k]-u2[i+1][j-2][k]) +
							c1*(u2[i+1][j+1][k]-u2[i+1][j-1][k])  )
						+ la[i-1][j][k]*met1[i-1][j][k]*met1[i-1][j][k]*(
							c2*(u2[i-1][j+2][k]-u2[i-1][j-2][k]) +
							c1*(u2[i-1][j+1][k]-u2[i-1][j-1][k])));

		} 
	}
}

extern "C" void host_code (double *h_r1, double *h_u1, double *h_u2, double *h_u3,  double *h_mu, double *h_la, double *h_met1, double *h_met2, double *h_met3, double *h_met4, double *h_strx, double *h_stry, double c1, double c2, int N) {
	double *r1;
	hipMalloc (&r1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for r1\n");
	hipMemcpy (r1, h_r1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u1;
	hipMalloc (&u1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u1\n");
	hipMemcpy (u1, h_u1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u2;
	hipMalloc (&u2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u2\n");
	hipMemcpy (u2, h_u2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u3;
	hipMalloc (&u3, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u3\n");
	hipMemcpy (u3, h_u3, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *mu;
	hipMalloc (&mu, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for mu\n");
	hipMemcpy (mu, h_mu, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *la;
	hipMalloc (&la, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for la\n");
	hipMemcpy (la, h_la, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met1;
	hipMalloc (&met1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met1\n");
	hipMemcpy (met1, h_met1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met2;
	hipMalloc (&met2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met2\n");
	hipMemcpy (met2, h_met2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met3;
	hipMalloc (&met3, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met3\n");
	hipMemcpy (met3, h_met3, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met4;
	hipMalloc (&met4, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met4\n");
	hipMemcpy (met4, h_met4, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *strx;
	hipMalloc (&strx, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(double)*N, hipMemcpyHostToDevice);
	double *stry;
	hipMalloc (&stry, sizeof(double)*N);
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, blockconfig.y), 1);

	curvi <<<gridconfig, blockconfig>>> (r1, u1, u2, u3, mu, la, met1, met2, met3, met4, strx, stry, c1, c2, N);
	hipMemcpy (h_r1, r1, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
}
