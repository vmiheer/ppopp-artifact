#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void j2d64pt (double * __restrict__ l_in, double * __restrict__ l_out, int N) {
	//Determing the block's indices
	int i0 = (int)(blockIdx.x)*(int)(blockDim.x) + 4;
	int i = max(i0,4) + (int)(threadIdx.x);
	int j0 = 4*(int)(blockIdx.y)*(int)(blockDim.y) + 4;
	int j = max(j0,4) + 4*(int)(threadIdx.y);

	double (*in)[8200] = (double (*)[8200]) l_in;
	double (*out)[8200] = (double (*)[8200]) l_out;

	if (i>=4 & j>=4 & i<=N-5 & j<=N-5) {
double _t_0_ = in[j-4][i-4];
_t_0_ -= in[j-4][i+4];
_t_0_ -= in[j+4][i-4];
double _t_11_ = in[j+4][i-4];
double _t_22_ = -(in[j+4][i-4]);
double _t_33_ = in[j+4][i-4];
_t_0_ += in[j+4][i+4];
_t_11_ -= in[j+4][i+4];
_t_22_ += in[j+4][i+4];
_t_33_ -= in[j+4][i+4];
double outjc0ic0 = _t_0_ * 1.27449;
double _t_4_ = in[j-3][i-3];
_t_11_ += -in[j-3][i-3];
_t_4_ -= in[j-3][i+3];
_t_11_ += in[j-3][i+3];
_t_4_ -= in[j+3][i-3];
double _t_15_ = in[j+3][i-3];
double _t_26_ = -(in[j+3][i-3]);
_t_4_ += in[j+3][i+3];
_t_15_ -= in[j+3][i+3];
_t_26_ += in[j+3][i+3];
outjc0ic0 += _t_4_ * 0.00145161;
double _t_7_ = in[j-2][i-2];
_t_15_ += -in[j-2][i-2];
_t_22_ += in[j-2][i-2];
_t_7_ -= in[j-2][i+2];
_t_15_ += in[j-2][i+2];
_t_22_ -= in[j-2][i+2];
_t_7_ -= in[j+2][i-2];
double _t_18_ = in[j+2][i-2];
double _t_38_ = -(in[j+2][i-2]);
_t_7_ += in[j+2][i+2];
_t_18_ -= in[j+2][i+2];
_t_38_ += in[j+2][i+2];
outjc0ic0 += _t_7_ * 0.04;
double _t_9_ = in[j-1][i-1];
_t_18_ += -in[j-1][i-1];
_t_26_ += in[j-1][i-1];
_t_33_ += -in[j-1][i-1];
_t_9_ -= in[j-1][i+1];
_t_18_ += in[j-1][i+1];
_t_26_ -= in[j-1][i+1];
_t_33_ += in[j-1][i+1];
_t_9_ -= in[j+1][i-1];
double _t_29_ = in[j+1][i-1];
_t_38_ += -in[j+1][i-1];
_t_9_ += in[j+1][i+1];
_t_29_ -= in[j+1][i+1];
_t_38_ += in[j+1][i+1];
outjc0ic0 += _t_9_ * 0.64;
double _t_1_ = -in[j-4][i-3];
_t_1_ += in[j-4][i+3];
_t_1_ += in[j-3][i+4];
double _t_10_ = -(in[j-3][i+4]);
_t_1_ -= in[j-3][i-4];
_t_10_ += in[j-3][i-4];
_t_1_ += in[j+3][i-4];
double _t_12_ = -(in[j+3][i-4]);
double _t_23_ = in[j+3][i-4];
_t_1_ -= in[j+3][i+4];
_t_12_ += in[j+3][i+4];
_t_23_ -= in[j+3][i+4];
_t_1_ += in[j+4][i-3];
double _t_14_ = -(in[j+4][i-3]);
double _t_25_ = in[j+4][i-3];
double _t_36_ = -(in[j+4][i-3]);
_t_1_ -= in[j+4][i+3];
_t_14_ += in[j+4][i+3];
_t_25_ -= in[j+4][i+3];
_t_36_ += in[j+4][i+3];
outjc0ic0 += _t_1_ * 0.000136017;
double _t_2_ = in[j-4][i-2];
_t_2_ -= in[j-4][i+2];
_t_2_ += in[j-2][i-4];
_t_11_ -= in[j-2][i-4];
double _t_20_ = in[j-2][i-4];
_t_2_ -= in[j-2][i+4];
_t_11_ += in[j-2][i+4];
_t_20_ -= in[j-2][i+4];
_t_2_ -= in[j+2][i-4];
double _t_13_ = in[j+2][i-4];
_t_33_ -= in[j+2][i-4];
_t_2_ += in[j+2][i+4];
_t_13_ -= in[j+2][i+4];
_t_33_ += in[j+2][i+4];
_t_2_ -= in[j+4][i-2];
_t_15_ += in[j+4][i-2];
double _t_27_ = -(in[j+4][i-2]);
_t_38_ += in[j+4][i-2];
_t_2_ += in[j+4][i+2];
_t_15_ -= in[j+4][i+2];
_t_27_ += in[j+4][i+2];
_t_38_ -= in[j+4][i+2];
outjc0ic0 += _t_2_ * 0.000714;
double _t_3_ = -in[j-4][i-1];
_t_3_ += in[j-4][i+1];
_t_3_ -= in[j-1][i-4];
_t_12_ += in[j-1][i-4];
double _t_21_ = -(in[j-1][i-4]);
double _t_30_ = in[j-1][i-4];
_t_3_ += in[j-1][i+4];
_t_12_ -= in[j-1][i+4];
_t_21_ += in[j-1][i+4];
_t_30_ -= in[j-1][i+4];
_t_3_ += in[j+1][i-4];
_t_23_ -= in[j+1][i-4];
double _t_32_ = in[j+1][i-4];
_t_3_ -= in[j+1][i+4];
_t_23_ += in[j+1][i+4];
_t_32_ -= in[j+1][i+4];
_t_3_ += in[j+4][i-1];
double _t_16_ = -(in[j+4][i-1]);
double _t_28_ = in[j+4][i-1];
double _t_39_ = -(in[j+4][i-1]);
_t_3_ -= in[j+4][i+1];
_t_16_ += in[j+4][i+1];
_t_28_ -= in[j+4][i+1];
_t_39_ += in[j+4][i+1];
outjc0ic0 += _t_3_ * 0.002856;
double _t_5_ = -in[j-3][i-2];
_t_12_ += in[j-3][i-2];
_t_5_ += in[j-3][i+2];
_t_12_ -= in[j-3][i+2];
_t_5_ -= in[j-2][i-3];
_t_14_ += in[j-2][i-3];
_t_21_ += -in[j-2][i-3];
_t_5_ += in[j-2][i+3];
_t_14_ -= in[j-2][i+3];
_t_21_ += in[j-2][i+3];
_t_5_ += in[j+2][i-3];
_t_16_ -= in[j+2][i-3];
_t_36_ += in[j+2][i-3];
_t_5_ -= in[j+2][i+3];
_t_16_ += in[j+2][i+3];
_t_36_ -= in[j+2][i+3];
_t_5_ += in[j+3][i-2];
double _t_17_ = -(in[j+3][i-2]);
_t_28_ += in[j+3][i-2];
_t_5_ -= in[j+3][i+2];
_t_17_ += in[j+3][i+2];
_t_28_ -= in[j+3][i+2];
outjc0ic0 += _t_5_ * 0.00762;
double _t_6_ = in[j-3][i-1];
_t_13_ += -in[j-3][i-1];
_t_6_ -= in[j-3][i+1];
_t_13_ += in[j-3][i+1];
_t_6_ += in[j-1][i-3];
_t_15_ -= in[j-1][i-3];
double _t_24_ = in[j-1][i-3];
double _t_31_ = -in[j-1][i-3];
_t_6_ -= in[j-1][i+3];
_t_15_ += in[j-1][i+3];
_t_24_ -= in[j-1][i+3];
_t_31_ += in[j-1][i+3];
_t_6_ -= in[j+1][i-3];
_t_26_ += in[j+1][i-3];
double _t_35_ = -(in[j+1][i-3]);
_t_6_ += in[j+1][i+3];
_t_26_ -= in[j+1][i+3];
_t_35_ += in[j+1][i+3];
_t_6_ -= in[j+3][i-1];
_t_18_ += in[j+3][i-1];
_t_29_ -= in[j+3][i-1];
_t_6_ += in[j+3][i+1];
_t_18_ -= in[j+3][i+1];
_t_29_ += in[j+3][i+1];
outjc0ic0 += _t_6_ * 0.03048;
double _t_8_ = -in[j-2][i-1];
_t_16_ += in[j-2][i-1];
_t_23_ += -in[j-2][i-1];
_t_8_ += in[j-2][i+1];
_t_16_ -= in[j-2][i+1];
_t_23_ += in[j-2][i+1];
_t_8_ -= in[j-1][i-2];
_t_17_ += in[j-1][i-2];
_t_25_ += -in[j-1][i-2];
_t_32_ += in[j-1][i-2];
_t_8_ += in[j-1][i+2];
_t_17_ -= in[j-1][i+2];
_t_25_ += in[j-1][i+2];
_t_32_ -= in[j-1][i+2];
_t_8_ += in[j+1][i-2];
_t_28_ -= in[j+1][i-2];
double _t_37_ = in[j+1][i-2];
_t_8_ -= in[j+1][i+2];
_t_28_ += in[j+1][i+2];
_t_37_ -= in[j+1][i+2];
_t_8_ += in[j+2][i-1];
double _t_19_ = -(in[j+2][i-1]);
_t_39_ += in[j+2][i-1];
_t_8_ -= in[j+2][i+1];
_t_19_ += in[j+2][i+1];
_t_39_ -= in[j+2][i+1];
outjc0ic0 += _t_8_ * 0.16;

_t_10_ -= in[j+5][i-4];
_t_21_ += in[j+5][i-4];
_t_33_ += in[j+7][i-1];
_t_33_ -= in[j+7][i+1];
double outjp3ic0 = _t_33_ * 0.002856;
_t_30_ -= in[j+7][i-4];
_t_30_ += in[j+7][i+4];
outjp3ic0 += _t_30_ * 1.27449;
_t_10_ += in[j+5][i+4];
_t_21_ -= in[j+5][i+4];
double outjp1ic0 = _t_10_ * 1.27449;
_t_32_ -= in[j+5][i-4];
_t_32_ += in[j+5][i+4];
_t_32_ -= in[j+7][i-2];
_t_32_ += in[j+7][i+2];
outjp3ic0 += _t_32_ * 0.000714;
_t_11_ += in[j+5][i-3];
_t_35_ += in[j+5][i-3];
_t_11_ -= in[j+5][i+3];
_t_35_ -= in[j+5][i+3];
outjp1ic0 += _t_11_ * 0.000136017;
_t_24_ -= in[j+5][i-3];
_t_24_ += in[j+5][i+3];
double outjp2ic0 = _t_24_ * 0.00145161;
_t_12_ -= in[j+5][i-2];
_t_12_ += in[j+5][i+2];
outjp1ic0 += _t_12_ * 0.000714;
_t_37_ -= in[j+5][i-2];
_t_37_ += in[j+5][i+2];
outjp3ic0 += _t_37_ * 0.04;
outjp1ic0 += _t_14_ * 0.00145161;
outjp1ic0 += _t_15_ * 0.00762;
outjp1ic0 += _t_17_ * 0.04;
_t_13_ -= in[j][i-4];
_t_22_ += in[j][i-4];
_t_31_ -= in[j][i-4];
_t_13_ += in[j][i+4];
_t_22_ -= in[j][i+4];
_t_31_ += in[j][i+4];
_t_13_ += in[j+5][i-1];
_t_13_ -= in[j+5][i+1];
outjp1ic0 += _t_13_ * 0.002856;
_t_26_ -= in[j+5][i-1];
_t_26_ += in[j+5][i+1];
outjp2ic0 += _t_26_ * 0.03048;
_t_38_ += in[j+5][i-1];
_t_38_ -= in[j+5][i+1];
outjp3ic0 += _t_38_ * 0.16;
_t_16_ += in[j][i-3];
double _t_34_ = in[j][i-3];
_t_16_ -= in[j][i+3];
_t_34_ -= in[j][i+3];
outjp1ic0 += _t_16_ * 0.03048;
_t_18_ -= in[j][i-2];
_t_35_ += -in[j][i-2];
_t_18_ += in[j][i+2];
_t_35_ += in[j][i+2];
outjp1ic0 += _t_18_ * 0.16;
_t_27_ += in[j][i-2];
_t_27_ -= in[j][i+2];
outjp2ic0 += _t_27_ * 0.04;
_t_19_ += in[j][i-1];
_t_36_ += in[j][i-1];
_t_19_ -= in[j][i+1];
_t_36_ -= in[j][i+1];
outjp1ic0 += _t_19_ * 0.64;
_t_28_ += -in[j][i-1];
_t_28_ += in[j][i+1];
outjp2ic0 += _t_28_ * 0.16;
_t_25_ -= in[j][i-3];
_t_25_ += in[j][i+3];
_t_25_ += in[j+5][i-2];
_t_25_ -= in[j+5][i+2];
outjp2ic0 += _t_25_ * 0.00762;

_t_23_ += in[j+6][i-1];
_t_23_ -= in[j+6][i+1];
outjp2ic0 += _t_23_ * 0.002856;
_t_36_ -= in[j+6][i-1];
_t_36_ += in[j+6][i+1];
outjp3ic0 += _t_36_ * 0.03048;
outjp2ic0 += _t_29_ * 0.64;
_t_20_ -= in[j+6][i-4];
_t_20_ += in[j+6][i+4];
outjp2ic0 += _t_20_ * 1.27449;
_t_31_ += in[j+6][i-4];
_t_31_ -= in[j+6][i+4];
_t_31_ += in[j+7][i-3];
_t_31_ -= in[j+7][i+3];
outjp3ic0 += _t_31_ * 0.000136017;
_t_21_ += in[j+6][i-3];
_t_21_ -= in[j+6][i+3];
outjp2ic0 += _t_21_ * 0.000136017;
_t_34_ -= in[j+6][i-3];
_t_34_ += in[j+6][i+3];
outjp3ic0 += _t_34_ * 0.00145161;
_t_22_ -= in[j+6][i-2];
_t_22_ += in[j+6][i+2];
outjp2ic0 += _t_22_ * 0.000714;
_t_35_ += in[j+6][i-2];
_t_35_ -= in[j+6][i+2];
outjp3ic0 += _t_35_ * 0.00762;

outjp3ic0 += _t_39_ * 0.64;

out[j][i] = outjc0ic0;
out[j+1][i] = outjp1ic0;
out[j+2][i] = outjp2ic0;
out[j+3][i] = outjp3ic0;
	} 
}

extern "C" void host_code (double *h_in, double *h_out, int N) {
	double *in;
	hipMalloc (&in, sizeof(double)*N*N);
	check_error ("Failed to allocate device memory for in\n");
	hipMemcpy (in, h_in, sizeof(double)*N*N, hipMemcpyHostToDevice);
	double *out;
	hipMalloc (&out, sizeof(double)*N*N);
	check_error ("Failed to allocate device memory for out\n");

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, 4*blockconfig.y));

	j2d64pt<<<gridconfig, blockconfig>>> (in, out, N);

	hipMemcpy (h_out, out, sizeof(double)*N*N, hipMemcpyDeviceToHost);
	hipFree (in); 
	hipFree (out);
}
